#include "hip/hip_runtime.h"
/* ---------------------------------------------------------------------
 *
 * Copyright (C) 2019 - 2020 by the deal.II authors
 *
 * This file is part of the deal.II library.
 *
 * The deal.II library is free software; you can use it, redistribute
 * it, and/or modify it under the terms of the GNU Lesser General
 * Public License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 * The full text of the license can be found in the file LICENSE.md at
 * the top level directory of deal.II.
 *
 * ---------------------------------------------------------------------

 *
 * Authors: Bruno Turcksin, Daniel Arndt, Oak Ridge National Laboratory, 2019
 */

// First include the necessary files from the deal.II library known from the
// previous tutorials.
#include <deal.II/base/conditional_ostream.h>
#include <deal.II/base/quadrature_lib.h>

#include <deal.II/dofs/dof_tools.h>

#include <deal.II/fe/fe_q.h>

#include <deal.II/grid/grid_generator.h>
#include <deal.II/grid/tria.h>

#include <deal.II/lac/affine_constraints.h>
#include <deal.II/lac/la_parallel_vector.h>
#include <deal.II/lac/precondition.h>
#include <deal.II/lac/solver_cg.h>

#include <deal.II/numerics/data_out.h>
#include <deal.II/numerics/vector_tools.h>

// The following ones include the data structures for the
// implementation of matrix-free methods on GPU:
#include <deal.II/base/hip/hip_runtime.h>

#include <deal.II/matrix_free/cuda_fe_evaluation.h>
#include <deal.II/matrix_free/cuda_matrix_free.h>
#include <deal.II/matrix_free/operators.h>

#include <fstream>


// As usual, we enclose everything into a namespace of its own:
namespace Step64
{
  using namespace dealii;


  // @sect3{Class <code>VaryingCoefficientFunctor</code>}

  // Next, we define a class that implements the varying coefficients
  // we want to use in the Helmholtz operator. Later, we want to pass
  // an object of this type to a CUDAWrappers::MatrixFree
  // object that expects the class to have an `operator()` that fills the
  // values provided in the constructor for a given cell. This operator
  // needs to run on the device, so it needs to be marked as `__device__`
  // for the compiler.
  template <int dim, int fe_degree>
  class VaryingCoefficientFunctor
  {
  public:
    VaryingCoefficientFunctor(double *coefficient)
      : coef(coefficient)
    {}

    __device__ void operator()(
      const unsigned int                                          cell,
      const typename CUDAWrappers::MatrixFree<dim, double>::Data *gpu_data);

    // Since CUDAWrappers::MatrixFree::Data doesn't know about the size of its
    // arrays, we need to store the number of quadrature points and the numbers
    // of degrees of freedom in this class to do necessary index conversions.
    static const unsigned int n_dofs_1d = fe_degree + 1;
    static const unsigned int n_local_dofs =
      dealii::Utilities::pow(n_dofs_1d, dim);
    static const unsigned int n_q_points =
      dealii::Utilities::pow(n_dofs_1d, dim);

  private:
    double *coef;
  };



  // The following function implements this coefficient. Recall from
  // the introduction that we have defined it as $a(\mathbf
  // x)=\frac{10}{0.05 + 2\|\mathbf x\|^2}$
  template <int dim, int fe_degree>
  __device__ void VaryingCoefficientFunctor<dim, fe_degree>::operator()(
    const unsigned int                                          cell,
    const typename CUDAWrappers::MatrixFree<dim, double>::Data *gpu_data)
  {
    const unsigned int pos = CUDAWrappers::local_q_point_id<dim, double>(
      cell, gpu_data, n_dofs_1d, n_q_points);
    const Point<dim> q_point =
      CUDAWrappers::get_quadrature_point<dim, double>(cell,
                                                      gpu_data,
                                                      n_dofs_1d);

    double p_square = 0.;
    for (unsigned int i = 0; i < dim; ++i)
      {
        const double coord = q_point[i];
        p_square += coord * coord;
      }
    coef[pos] = 10. / (0.05 + 2. * p_square);
  }


  // @sect3{Class <code>HelmholtzOperatorQuad</code>}

  // The class `HelmholtzOperatorQuad` implements the evaluation of
  // the Helmholtz operator at each quadrature point. It uses a
  // similar mechanism as the MatrixFree framework introduced in
  // step-37. In contrast to there, the actual quadrature point
  // index is treated implicitly by converting the current thread
  // index. As before, the functions of this class need to run on
  // the device, so need to be marked as `__device__` for the
  // compiler.
  template <int dim, int fe_degree>
  class HelmholtzOperatorQuad
  {
  public:
    __device__ HelmholtzOperatorQuad(double coef)
      : coef(coef)
    {}

    __device__ void
    operator()(CUDAWrappers::FEEvaluation<dim, fe_degree> *fe_eval) const;

  private:
    double coef;
  };


  // The Helmholtz problem we want to solve here reads in weak form as follows:
  // @f{eqnarray*}
  //   (\nabla v, \nabla u)+ (v, a(\mathbf x) u) &=&(v,1) \quad \forall v.
  // @f}
  // If you have seen step-37, then it will be obvious that
  // the two terms on the left-hand side correspond to the two function calls
  // here:
  template <int dim, int fe_degree>
  __device__ void HelmholtzOperatorQuad<dim, fe_degree>::
                  operator()(CUDAWrappers::FEEvaluation<dim, fe_degree> *fe_eval) const
  {
    fe_eval->submit_value(coef * fe_eval->get_value());
    fe_eval->submit_gradient(fe_eval->get_gradient());
  }


  // @sect3{Class <code>LocalHelmholtzOperator</code>}

  // Finally, we need to define a class that implements the whole operator
  // evaluation that corresponds to a matrix-vector product in matrix-based
  // approaches.
  template <int dim, int fe_degree>
  class LocalHelmholtzOperator
  {
  public:
    LocalHelmholtzOperator(double *coefficient)
      : coef(coefficient)
    {}

    __device__ void operator()(
      const unsigned int                                          cell,
      const typename CUDAWrappers::MatrixFree<dim, double>::Data *gpu_data,
      CUDAWrappers::SharedData<dim, double> *                     shared_data,
      const double *                                              src,
      double *                                                    dst) const;

    // Again, the CUDAWrappers::MatrixFree object doesn't know about the number
    // of degrees of freedom and the number of quadrature points so we need
    // to store these for index calculations in the call operator.
    static const unsigned int n_dofs_1d    = fe_degree + 1;
    static const unsigned int n_local_dofs = Utilities::pow(fe_degree + 1, dim);
    static const unsigned int n_q_points   = Utilities::pow(fe_degree + 1, dim);

  private:
    double *coef;
  };


  // This is the call operator that performs the Helmholtz operator evaluation
  // on a given cell similar to the MatrixFree framework on the CPU.
  // In particular, we need access to both values and gradients of the source
  // vector and we write value and gradient information to the destination
  // vector.
  template <int dim, int fe_degree>
  __device__ void LocalHelmholtzOperator<dim, fe_degree>::operator()(
    const unsigned int                                          cell,
    const typename CUDAWrappers::MatrixFree<dim, double>::Data *gpu_data,
    CUDAWrappers::SharedData<dim, double> *                     shared_data,
    const double *                                              src,
    double *                                                    dst) const
  {
    const unsigned int pos = CUDAWrappers::local_q_point_id<dim, double>(
      cell, gpu_data, n_dofs_1d, n_q_points);

    CUDAWrappers::FEEvaluation<dim, fe_degree, fe_degree + 1, 1, double>
      fe_eval(cell, gpu_data, shared_data);
    fe_eval.read_dof_values(src);
    fe_eval.evaluate(true, true);
    fe_eval.apply_for_each_quad_point(
      HelmholtzOperatorQuad<dim, fe_degree>(coef[pos]));
    fe_eval.integrate(true, true);
    fe_eval.distribute_local_to_global(dst);
  }


  // @sect3{Class <code>HelmholtzOperator</code>}

  // The `HelmholtzOperator` class acts as wrapper for
  // `LocalHelmholtzOperator` defining an interface that can be used
  // with linear solvers like SolverCG. In particular, like every
  // class that implements the interface of a linear operator, it
  // needs to have a `vmult()` function that performs the action of
  // the linear operator on a source vector.
  template <int dim, int fe_degree>
  class HelmholtzOperator
  {
  public:
    HelmholtzOperator(const DoFHandler<dim> &          dof_handler,
                      const AffineConstraints<double> &constraints);

    void
    vmult(LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> &dst,
          const LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA>
            &src) const;

    void initialize_dof_vector(
      LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> &vec) const;

  private:
    CUDAWrappers::MatrixFree<dim, double>       mf_data;
    LinearAlgebra::CUDAWrappers::Vector<double> coef;
  };



  // The following is the implementation of the constructor of this
  // class. In the first part, we initialize the `mf_data` member
  // variable that is going to provide us with the necessary
  // information when evaluating the operator.
  //
  // In the second half, we need to store the value of the coefficient
  // for each quadrature point in every active, locally owned cell.
  // We can ask the parallel triangulation for the number of active, locally
  // owned cells but only have a DoFHandler object at hand. Since
  // DoFHandler::get_triangulation() returns a Triangulation object, not a
  // parallel::TriangulationBase object, we have to downcast the return value.
  // This is safe to do here because we know that the triangulation is a
  // parallel:distributed::Triangulation object in fact.
  template <int dim, int fe_degree>
  HelmholtzOperator<dim, fe_degree>::HelmholtzOperator(
    const DoFHandler<dim> &          dof_handler,
    const AffineConstraints<double> &constraints)
  {
    MappingQGeneric<dim> mapping(fe_degree);
    typename CUDAWrappers::MatrixFree<dim, double>::AdditionalData
      additional_data;
    additional_data.mapping_update_flags = update_values | update_gradients |
                                           update_JxW_values |
                                           update_quadrature_points;
    const QGauss<1> quad(fe_degree + 1);
    mf_data.reinit(mapping, dof_handler, constraints, quad, additional_data);


    const unsigned int n_owned_cells =
      dynamic_cast<const parallel::TriangulationBase<dim> *>(
        &dof_handler.get_triangulation())
        ->n_locally_owned_active_cells();
    coef.reinit(Utilities::pow(fe_degree + 1, dim) * n_owned_cells);

    const VaryingCoefficientFunctor<dim, fe_degree> functor(coef.get_values());
    mf_data.evaluate_coefficients(functor);
  }


  // The key step then is to use all of the previous classes to loop over
  // all cells to perform the matrix-vector product. We implement this
  // in the next function.
  //
  // When applying the Helmholtz operator, we have to be careful to handle
  // boundary conditions correctly. Since the local operator doesn't know about
  // constraints, we have to copy the correct values from the source to the
  // destination vector afterwards.
  template <int dim, int fe_degree>
  void HelmholtzOperator<dim, fe_degree>::vmult(
    LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> &      dst,
    const LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> &src)
    const
  {
    dst = 0.;
    LocalHelmholtzOperator<dim, fe_degree> helmholtz_operator(
      coef.get_values());
    mf_data.cell_loop(helmholtz_operator, src, dst);
    mf_data.copy_constrained_values(src, dst);
  }



  template <int dim, int fe_degree>
  void HelmholtzOperator<dim, fe_degree>::initialize_dof_vector(
    LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> &vec) const
  {
    mf_data.initialize_dof_vector(vec);
  }


  // @sect3{Class <code>HelmholtzProblem</code>}

  // This is the main class of this program. It defines the usual
  // framework we use for tutorial programs. The only point worth
  // commenting on is the `solve()` function and the choice of vector
  // types.
  template <int dim, int fe_degree>
  class HelmholtzProblem
  {
  public:
    HelmholtzProblem();

    void run();

  private:
    void setup_system();

    void assemble_rhs();

    void solve();

    void output_results(const unsigned int cycle) const;

    MPI_Comm mpi_communicator;

    parallel::distributed::Triangulation<dim> triangulation;

    FE_Q<dim>       fe;
    DoFHandler<dim> dof_handler;

    IndexSet locally_owned_dofs;
    IndexSet locally_relevant_dofs;

    AffineConstraints<double>                          constraints;
    std::unique_ptr<HelmholtzOperator<dim, fe_degree>> system_matrix_dev;

    // Since all the operations in the `solve()` function are executed on the
    // graphics card, it is necessary for the vectors used to store their values
    // on the GPU as well. LinearAlgebra::distributed::Vector can be told which
    // memory space to use. There is also LinearAlgebra::CUDAWrappers::Vector
    // that always uses GPU memory storage but doesn't work with MPI. It might
    // be worth noticing that the communication between different MPI processes
    // can be improved if the MPI implementation is CUDA-aware and the configure
    // flag `DEAL_II_MPI_WITH_CUDA_SUPPORT` is enabled. (The value of this
    // flag needs to be set at the time you call `cmake` when installing
    // deal.II.)
    //
    // In addition, we also keep a solution vector with CPU storage such that we
    // can view and display the solution as usual.
    LinearAlgebra::distributed::Vector<double, MemorySpace::Host>
                                                                  ghost_solution_host;
    LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> solution_dev;
    LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA>
      system_rhs_dev;

    ConditionalOStream pcout;
  };


  // The implementation of all the remaining functions of this class apart from
  // `Helmholtzproblem::solve()` doesn't contain anything new and we won't
  // further comment much on the overall approach.
  template <int dim, int fe_degree>
  HelmholtzProblem<dim, fe_degree>::HelmholtzProblem()
    : mpi_communicator(MPI_COMM_WORLD)
    , triangulation(mpi_communicator)
    , fe(fe_degree)
    , dof_handler(triangulation)
    , pcout(std::cout, Utilities::MPI::this_mpi_process(mpi_communicator) == 0)
  {}



  template <int dim, int fe_degree>
  void HelmholtzProblem<dim, fe_degree>::setup_system()
  {
    dof_handler.distribute_dofs(fe);

    locally_owned_dofs = dof_handler.locally_owned_dofs();
    DoFTools::extract_locally_relevant_dofs(dof_handler, locally_relevant_dofs);
    system_rhs_dev.reinit(locally_owned_dofs, mpi_communicator);

    constraints.clear();
    constraints.reinit(locally_relevant_dofs);
    DoFTools::make_hanging_node_constraints(dof_handler, constraints);
    VectorTools::interpolate_boundary_values(dof_handler,
                                             0,
                                             Functions::ZeroFunction<dim>(),
                                             constraints);
    constraints.close();

    system_matrix_dev.reset(
      new HelmholtzOperator<dim, fe_degree>(dof_handler, constraints));

    ghost_solution_host.reinit(locally_owned_dofs,
                               locally_relevant_dofs,
                               mpi_communicator);
    system_matrix_dev->initialize_dof_vector(solution_dev);
    system_rhs_dev.reinit(solution_dev);
  }



  // Unlike programs such as step-4 or step-6, we will not have to
  // assemble the whole linear system but only the right hand side
  // vector. This looks in essence like we did in step-4, for example,
  // but we have to pay attention to using the right constraints
  // object when copying local contributions into the global
  // vector. In particular, we need to make sure the entries that
  // correspond to boundary nodes are properly zeroed out. This is
  // necessary for CG to converge.  (Another solution would be to
  // modify the `vmult()` function above in such a way that we pretend
  // the source vector has zero entries by just not taking them into
  // account in matrix-vector products. But the approach used here is
  // simpler.)
  //
  // At the end of the function, we can't directly copy the values
  // from the host to the device but need to use an intermediate
  // object of type LinearAlgebra::ReadWriteVector to construct the
  // correct communication pattern necessary.
  template <int dim, int fe_degree>
  void HelmholtzProblem<dim, fe_degree>::assemble_rhs()
  {
    LinearAlgebra::distributed::Vector<double, MemorySpace::Host>
                      system_rhs_host(locally_owned_dofs,
                      locally_relevant_dofs,
                      mpi_communicator);
    const QGauss<dim> quadrature_formula(fe_degree + 1);

    FEValues<dim> fe_values(fe,
                            quadrature_formula,
                            update_values | update_quadrature_points |
                              update_JxW_values);

    const unsigned int dofs_per_cell = fe.n_dofs_per_cell();
    const unsigned int n_q_points    = quadrature_formula.size();

    Vector<double> cell_rhs(dofs_per_cell);

    std::vector<types::global_dof_index> local_dof_indices(dofs_per_cell);

    for (const auto &cell : dof_handler.active_cell_iterators())
      if (cell->is_locally_owned())
        {
          cell_rhs = 0;

          fe_values.reinit(cell);

          for (unsigned int q_index = 0; q_index < n_q_points; ++q_index)
            {
              for (unsigned int i = 0; i < dofs_per_cell; ++i)
                cell_rhs(i) += (fe_values.shape_value(i, q_index) * 1.0 *
                                fe_values.JxW(q_index));
            }

          cell->get_dof_indices(local_dof_indices);
          constraints.distribute_local_to_global(cell_rhs,
                                                 local_dof_indices,
                                                 system_rhs_host);
        }
    system_rhs_host.compress(VectorOperation::add);

    LinearAlgebra::ReadWriteVector<double> rw_vector(locally_owned_dofs);
    rw_vector.import(system_rhs_host, VectorOperation::insert);
    system_rhs_dev.import(rw_vector, VectorOperation::insert);
  }



  // This solve() function finally contains the calls to the new classes
  // previously discussed. Here we don't use any preconditioner, i.e.,
  // precondition by the identity matrix, to focus just on the peculiarities of
  // the CUDAWrappers::MatrixFree framework. Of course, in a real application
  // the choice of a suitable preconditioner is crucial but we have at least the
  // same restrictions as in step-37 since matrix entries are computed on the
  // fly and not stored.
  //
  // After solving the linear system in the first part of the function, we
  // copy the solution from the device to the host to be able to view its
  // values and display it in `output_results()`. This transfer works the same
  // as at the end of the previous function.
  template <int dim, int fe_degree>
  void HelmholtzProblem<dim, fe_degree>::solve()
  {
    PreconditionIdentity preconditioner;

    SolverControl solver_control(system_rhs_dev.size(),
                                 1e-12 * system_rhs_dev.l2_norm());
    SolverCG<LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA>> cg(
      solver_control);
    cg.solve(*system_matrix_dev, solution_dev, system_rhs_dev, preconditioner);

    pcout << "  Solved in " << solver_control.last_step() << " iterations."
          << std::endl;

    LinearAlgebra::ReadWriteVector<double> rw_vector(locally_owned_dofs);
    rw_vector.import(solution_dev, VectorOperation::insert);
    ghost_solution_host.import(rw_vector, VectorOperation::insert);

    constraints.distribute(ghost_solution_host);

    ghost_solution_host.update_ghost_values();
  }

  // The output results function is as usual since we have already copied the
  // values back from the GPU to the CPU.
  //
  // While we're already doing something with the function, we might
  // as well compute the $L_2$ norm of the solution. We do this by
  // calling VectorTools::integrate_difference(). That function is
  // meant to compute the error by evaluating the difference between
  // the numerical solution (given by a vector of values for the
  // degrees of freedom) and an object representing the exact
  // solution. But we can easily compute the $L_2$ norm of the
  // solution by passing in a zero function instead. That is, instead
  // of evaluating the error $\|u_h-u\|_{L_2(\Omega)}$, we are just
  // evaluating $\|u_h-0\|_{L_2(\Omega)}=\|u_h\|_{L_2(\Omega)}$
  // instead.
  template <int dim, int fe_degree>
  void HelmholtzProblem<dim, fe_degree>::output_results(
    const unsigned int cycle) const
  {
    DataOut<dim> data_out;

    data_out.attach_dof_handler(dof_handler);
    data_out.add_data_vector(ghost_solution_host, "solution");
    data_out.build_patches();

    DataOutBase::VtkFlags flags;
    flags.compression_level = DataOutBase::VtkFlags::best_speed;
    data_out.set_flags(flags);
    data_out.write_vtu_with_pvtu_record(
      "./", "solution", cycle, mpi_communicator, 2);

    Vector<float> cellwise_norm(triangulation.n_active_cells());
    VectorTools::integrate_difference(dof_handler,
                                      ghost_solution_host,
                                      Functions::ZeroFunction<dim>(),
                                      cellwise_norm,
                                      QGauss<dim>(fe.degree + 2),
                                      VectorTools::L2_norm);
    const double global_norm =
      VectorTools::compute_global_error(triangulation,
                                        cellwise_norm,
                                        VectorTools::L2_norm);
    pcout << "  solution norm: " << global_norm << std::endl;
  }


  // There is nothing surprising in the `run()` function either. We simply
  // compute the solution on a series of (globally) refined meshes.
  template <int dim, int fe_degree>
  void HelmholtzProblem<dim, fe_degree>::run()
  {
    for (unsigned int cycle = 0; cycle < 7 - dim; ++cycle)
      {
        pcout << "Cycle " << cycle << std::endl;

        if (cycle == 0)
          GridGenerator::hyper_cube(triangulation, 0., 1.);
        triangulation.refine_global(1);

        setup_system();

        pcout << "   Number of active cells:       "
              << triangulation.n_global_active_cells() << std::endl
              << "   Number of degrees of freedom: " << dof_handler.n_dofs()
              << std::endl;

        assemble_rhs();
        solve();
        output_results(cycle);
        pcout << std::endl;
      }
  }
} // namespace Step64


// @sect3{The <code>main()</code> function}

// Finally for the `main()` function.  By default, all the MPI ranks
// will try to access the device with number 0, which we assume to be
// the GPU device associated with the CPU on which a particular MPI
// rank runs. This works, but if we are running with MPI support it
// may be that multiple MPI processes are running on the same machine
// (for example, one per CPU core) and then they would all want to
// access the same GPU on that machine. If there is only one GPU in
// the machine, there is nothing we can do about it: All MPI ranks on
// that machine need to share it. But if there are more than one GPU,
// then it is better to address different graphic cards for different
// processes. The choice below is based on the MPI process id by
// assigning GPUs round robin to GPU ranks. (To work correctly, this
// scheme assumes that the MPI ranks on one machine are
// consecutive. If that were not the case, then the rank-GPU
// association may just not be optimal.) To make this work, MPI needs
// to be initialized before using this function.
int main(int argc, char *argv[])
{
  try
    {
      using namespace Step64;

      Utilities::MPI::MPI_InitFinalize mpi_init(argc, argv, 1);

      int         n_devices       = 0;
      hipError_t cuda_error_code = hipGetDeviceCount(&n_devices);
      AssertCuda(cuda_error_code);
      const unsigned int my_mpi_id =
        Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
      const int device_id = my_mpi_id % n_devices;
      cuda_error_code     = hipSetDevice(device_id);
      AssertCuda(cuda_error_code);

      HelmholtzProblem<3, 3> helmholtz_problem;
      helmholtz_problem.run();
    }
  catch (std::exception &exc)
    {
      std::cerr << std::endl
                << std::endl
                << "----------------------------------------------------"
                << std::endl;
      std::cerr << "Exception on processing: " << std::endl
                << exc.what() << std::endl
                << "Aborting!" << std::endl
                << "----------------------------------------------------"
                << std::endl;
      return 1;
    }
  catch (...)
    {
      std::cerr << std::endl
                << std::endl
                << "----------------------------------------------------"
                << std::endl;
      std::cerr << "Unknown exception!" << std::endl
                << "Aborting!" << std::endl
                << "----------------------------------------------------"
                << std::endl;
      return 1;
    }

  return 0;
}

#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2016 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Test operator[] and norm_square of cuda_tensor.

#include <deal.II/base/tensor.h>

#include "../tests.h"

template <int rank, int dim, typename Number>
__global__ void
miscellaneous_kernel(Number *check_1,
                     Number *check_2,
                     Number *check_3,
                     Number *check_4,
                     Number *check_5)
{
  // constructors
  typename Tensor<rank, dim, Number>::array_type array{};
  Tensor<rank, dim, Number>                      dummy_1(array);
  *check_1 = dummy_1.norm_square();
  Tensor<rank, dim, Number> dummy_2;
  *check_2                          = dummy_2.norm_square();
  Tensor<rank, dim, Number> dummy_3 = dummy_2;
  *check_3                          = dummy_3.norm_square();

  // access
  Tensor<rank + 1, dim, Number>   initializer_1;
  const Tensor<rank, dim, Number> dummy_5 = initializer_1[0];
  *check_4                                = dummy_5.norm_square();

  // assignment
  dummy_2  = dummy_3;
  *check_5 = dummy_2.norm_square();
}

template <int rank, int dim, typename Number>
__global__ void
summation_kernel(Tensor<rank, dim, Number> *t,
                 Tensor<rank, dim, Number> *t1,
                 Tensor<rank, dim, Number> *t2)
{
  *t2 += *t;
  *t1 = *t1 + *t;
}

template <int rank, int dim, typename Number>
__global__ void
subtraction_kernel(Tensor<rank, dim, Number> *t,
                   Tensor<rank, dim, Number> *t1,
                   Tensor<rank, dim, Number> *t2)
{
  *t2 -= *t;
  *t1 = *t1 - *t;
}

template <int rank, int dim, typename Number>
__global__ void
multiplication_kernel(Tensor<rank, dim, Number> *t,
                      Tensor<rank, dim, Number> *t1,
                      Tensor<rank, dim, Number> *t2)
{
  *t1 = *t * Number(2.);
  *t2 = Number(2.) * *t;
  *t *= 2.;
}

template <int rank, int dim, typename Number>
__global__ void
division_kernel(Tensor<rank, dim, Number> *t,
                Tensor<rank, dim, Number> *t1,
                Tensor<rank, dim, Number> *t2)
{
  *t1 = *t / Number(2.);
  *t /= 2.;
  *t2 = *t1;
}

template <int dim, typename Number>
__global__ void init_kernel(Tensor<0, dim, Number> *t)
{
  if (threadIdx.x == 0)
    *t = 1.;
}

template <int dim, typename Number>
__global__ void init_kernel(Tensor<1, dim, Number> *t)
{
  const unsigned int i = threadIdx.x;
  if (i < dim)
    (*t)[i] = i + 1.;
}

template <int dim, typename Number>
__global__ void init_kernel(Tensor<2, dim, Number> *t)
{
  const unsigned int i = threadIdx.y;
  const unsigned int j = threadIdx.x;
  if ((i < dim) && (j < dim))
    (*t)[i][j] = j + i * dim + 1.;
}


template <int rank, int dim, typename Number>
void
test_gpu()
{
  const double tolerance = 1.e-8;

  Tensor<rank, dim, Number> *t_dev;
  Tensor<rank, dim, Number> *t1_dev;
  Tensor<rank, dim, Number> *t2_dev;

  Tensor<rank, dim, Number> t_host;
  Tensor<rank, dim, Number> t1_host;
  Tensor<rank, dim, Number> t2_host;

  Tensor<rank, dim, Number> reference_host;

  // Allocate objects on the device
  hipError_t cuda_error =
    hipMalloc(&t_dev, sizeof(Tensor<rank, dim, Number>));
  AssertCuda(cuda_error);
  cuda_error = hipMalloc(&t1_dev, sizeof(Tensor<rank, dim, Number>));
  AssertCuda(cuda_error);
  cuda_error = hipMalloc(&t2_dev, sizeof(Tensor<rank, dim, Number>));
  AssertCuda(cuda_error);

  // Initialize
  dim3 block_dim(dim, dim);
  init_kernel<<<1, block_dim>>>(t_dev);
  cuda_error = hipMemcpy(&reference_host,
                          t_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);

  // Test multiplication.
  multiplication_kernel<<<1, 1>>>(t_dev, t1_dev, t2_dev);

  cuda_error = hipMemcpy(&t_host,
                          t_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);
  cuda_error = hipMemcpy(&t1_host,
                          t1_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);
  cuda_error = hipMemcpy(&t2_host,
                          t2_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);

  reference_host *= 2;
  AssertThrow((t_host - reference_host).norm() < tolerance, ExcInternalError());
  AssertThrow((t1_host - reference_host).norm() < tolerance,
              ExcInternalError());
  AssertThrow((t2_host - reference_host).norm() < tolerance,
              ExcInternalError());

  deallog << "multiplication OK" << std::endl;

  // Test division.
  division_kernel<<<1, 1>>>(t_dev, t1_dev, t2_dev);
  cuda_error = hipMemcpy(&t_host,
                          t_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);
  cuda_error = hipMemcpy(&t1_host,
                          t1_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);

  reference_host /= 2.;
  AssertThrow((t_host - reference_host).norm() < tolerance, ExcInternalError());
  AssertThrow((t1_host - reference_host).norm() < tolerance,
              ExcInternalError());

  deallog << "division OK" << std::endl;

  // Test summation
  summation_kernel<<<1, 1>>>(t_dev, t1_dev, t2_dev);
  cuda_error = hipMemcpy(&t1_host,
                          t1_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);
  cuda_error = hipMemcpy(&t2_host,
                          t2_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);

  reference_host *= 2.;
  AssertThrow((t1_host - reference_host).norm() < tolerance,
              ExcInternalError());
  AssertThrow((t2_host - reference_host).norm() < tolerance,
              ExcInternalError());


  // Test subtraction
  subtraction_kernel<<<1, 1>>>(t_dev, t1_dev, t2_dev);
  cuda_error = hipMemcpy(&t1_host,
                          t1_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);
  cuda_error = hipMemcpy(&t2_host,
                          t2_dev,
                          sizeof(Tensor<rank, dim, Number>),
                          hipMemcpyDeviceToHost);

  reference_host /= 2.;
  AssertThrow((t1_host - reference_host).norm() < tolerance,
              ExcInternalError());
  AssertThrow((t2_host - reference_host).norm() < tolerance,
              ExcInternalError());

  // Free memory
  cuda_error = hipFree(t_dev);
  AssertCuda(cuda_error);
  cuda_error = hipFree(t1_dev);
  AssertCuda(cuda_error);
  cuda_error = hipFree(t2_dev);
  AssertCuda(cuda_error);

  // Miscellaneous
  {
    Number *check_1;
    Number *check_2;
    Number *check_3;
    Number *check_4;
    Number *check_5;

    cuda_error = hipMalloc(&check_1, sizeof(Number));
    AssertCuda(cuda_error);
    cuda_error = hipMalloc(&check_2, sizeof(Number));
    AssertCuda(cuda_error);
    cuda_error = hipMalloc(&check_3, sizeof(Number));
    AssertCuda(cuda_error);
    cuda_error = hipMalloc(&check_4, sizeof(Number));
    AssertCuda(cuda_error);
    cuda_error = hipMalloc(&check_5, sizeof(Number));
    AssertCuda(cuda_error);

    miscellaneous_kernel<rank, dim, Number>
      <<<1, 1>>>(check_1, check_2, check_3, check_4, check_5);

    Number check_1_host, check_2_host, check_3_host, check_4_host, check_5_host;

    cuda_error = hipMemcpy(&check_1_host,
                            check_1,
                            sizeof(Number),
                            hipMemcpyDeviceToHost);
    AssertCuda(cuda_error);
    cuda_error = hipMemcpy(&check_2_host,
                            check_2,
                            sizeof(Number),
                            hipMemcpyDeviceToHost);
    AssertCuda(cuda_error);
    cuda_error = hipMemcpy(&check_3_host,
                            check_3,
                            sizeof(Number),
                            hipMemcpyDeviceToHost);
    AssertCuda(cuda_error);
    cuda_error = hipMemcpy(&check_4_host,
                            check_4,
                            sizeof(Number),
                            hipMemcpyDeviceToHost);
    AssertCuda(cuda_error);
    cuda_error = hipMemcpy(&check_5_host,
                            check_5,
                            sizeof(Number),
                            hipMemcpyDeviceToHost);
    AssertCuda(cuda_error);

    AssertThrow(std::abs(check_1_host) < tolerance, ExcInternalError());
    AssertThrow(std::abs(check_2_host) < tolerance, ExcInternalError());
    AssertThrow(std::abs(check_3_host) < tolerance, ExcInternalError());
    AssertThrow(std::abs(check_4_host) < tolerance, ExcInternalError());
    AssertThrow(std::abs(check_5_host) < tolerance, ExcInternalError());

    cuda_error = hipFree(check_1);
    AssertCuda(cuda_error);
    cuda_error = hipFree(check_2);
    AssertCuda(cuda_error);
    cuda_error = hipFree(check_3);
    AssertCuda(cuda_error);
    cuda_error = hipFree(check_4);
    AssertCuda(cuda_error);
    cuda_error = hipFree(check_5);
    AssertCuda(cuda_error);
  }
}

int
main()
{
  initlog();

  init_cuda();

  test_gpu<0, 3, double>();
  test_gpu<1, 3, double>();
  test_gpu<2, 3, double>();
  test_gpu<0, 3, float>();
  test_gpu<1, 3, float>();
  test_gpu<2, 3, float>();
}

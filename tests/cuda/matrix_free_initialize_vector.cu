// ---------------------------------------------------------------------
//
// Copyright (C) 2019 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------



// Test CUDAWrappers::MatrixFree::initialize_dof_vector.

#include <deal.II/distributed/tria.h>

#include <deal.II/dofs/dof_handler.h>
#include <deal.II/dofs/dof_tools.h>

#include <deal.II/fe/fe_q.h>

#include <deal.II/grid/grid_generator.h>

#include <deal.II/lac/affine_constraints.h>
#include <deal.II/lac/cuda_vector.h>

#include <deal.II/matrix_free/cuda_matrix_free.h>

#include <deal.II/numerics/vector_tools.h>

#include "../tests.h"


template <typename Number>
void
check(
  const LinearAlgebra::distributed::Vector<Number, MemorySpace::CUDA> &vector,
  const Utilities::MPI::Partitioner &reference_partitioner)
{
  Assert(vector.get_partitioner()->locally_owned_range() ==
           reference_partitioner.locally_owned_range(),
         ExcInternalError());
  Assert(vector.get_partitioner()->ghost_indices() ==
           reference_partitioner.ghost_indices(),
         ExcInternalError());
}

template <typename Number>
void
check(const LinearAlgebra::CUDAWrappers::Vector<Number> &vector,
      const Utilities::MPI::Partitioner &                reference_partitioner)
{
  AssertDimension(vector.size(), reference_partitioner.size());
}

template <int dim, int fe_degree, typename VectorType>
void
test()
{
  using Number = double;

  parallel::distributed::Triangulation<dim> tria(MPI_COMM_WORLD);
  GridGenerator::hyper_cube(tria);
  tria.refine_global(2);

  FE_Q<dim>       fe(fe_degree);
  DoFHandler<dim> dof(tria);
  dof.distribute_dofs(fe);

  IndexSet owned_set = dof.locally_owned_dofs();
  IndexSet relevant_set;
  DoFTools::extract_locally_relevant_dofs(dof, relevant_set);

  deallog << "locally owned dofs :" << std::endl;
  owned_set.print(deallog.get_file_stream());

  deallog << "locally relevant dofs :" << std::endl;
  relevant_set.print(deallog.get_file_stream());

  AffineConstraints<double> constraints(relevant_set);
  constraints.close();

  MappingQGeneric<dim>                  mapping(fe_degree);
  CUDAWrappers::MatrixFree<dim, Number> mf_data;
  const QGauss<1>                       quad(fe_degree + 1);
  typename CUDAWrappers::MatrixFree<dim, Number>::AdditionalData
    additional_data;
  mf_data.reinit(mapping, dof, constraints, quad, additional_data);

  VectorType vector;
  mf_data.initialize_dof_vector(vector);

  Utilities::MPI::Partitioner reference_partitioner(owned_set,
                                                    relevant_set,
                                                    MPI_COMM_WORLD);
  check(vector, reference_partitioner);

  deallog << "OK" << std::endl;
}


int
main(int argc, char **argv)
{
  Utilities::MPI::MPI_InitFinalize mpi_initialization(
    argc, argv, testing_max_num_threads());

  unsigned int myid = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  deallog.push(Utilities::int_to_string(myid));

  init_cuda(true);
  MPILogInitAll mpi_inilog;

  test<2, 1, LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA>>();
  if (Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD) == 1)
    test<2, 1, LinearAlgebra::CUDAWrappers::Vector<double>>();
}

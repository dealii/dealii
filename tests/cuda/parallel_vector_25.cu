#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------


// check import from LA::d::Vector to another LA::d::Vector

#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/index_set.h>
#include <deal.II/base/utilities.h>

#include <deal.II/lac/la_parallel_vector.h>

#include "../tests.h"

void
test()
{
  unsigned int       rank = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  const unsigned int size = 100;
  const unsigned int local_size = 50;
  const unsigned int ghost_size = 75;

  IndexSet locally_owned(size);
  if (rank == 0)
    locally_owned.add_range(0, local_size);
  else
    locally_owned.add_range(size - local_size, size);
  locally_owned.compress();

  IndexSet ghost_indices(size);
  if (rank == 0)
    ghost_indices.add_range(0, ghost_size);
  else
    ghost_indices.add_range(size - ghost_size, size);
  ghost_indices.size();

  LinearAlgebra::distributed::Vector<double, MemorySpace::Host> vec_ref(
    locally_owned, ghost_indices, MPI_COMM_WORLD);
  for (unsigned int i = 0; i < local_size; ++i)
    vec_ref.local_element(i) = i;
  vec_ref.compress(VectorOperation::insert);

  auto partitioner = vec_ref.get_partitioner();
  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> vec_dev(
    partitioner);
  LinearAlgebra::distributed::Vector<double, MemorySpace::Host> vec_host(
    partitioner);

  // Assignment from Host to CUDA
  vec_dev.import(vec_ref, VectorOperation::insert);

  // Assignment from CUDA to Host
  vec_host.import(vec_dev, VectorOperation::insert);

  for (unsigned int i = 0; i < ghost_size; ++i)
    {
      AssertThrow(std::fabs(vec_ref.local_element(i) -
                            vec_host.local_element(i)) < 1e-12,
                  ExcInternalError());
    }

  if (rank == 0)
    deallog << "OK" << std::endl;
}

int
main(int argc, char **argv)
{
  Utilities::MPI::MPI_InitFinalize mpi_initialization(
    argc, argv, testing_max_num_threads());

  unsigned int myid = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  deallog.push(Utilities::int_to_string(myid));

  init_cuda(true);

  if (myid == 0)
    {
      initlog();

      test();
    }
  else
    test();
}

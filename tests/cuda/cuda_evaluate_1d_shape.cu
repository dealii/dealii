#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2016 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------



// this function tests the correctness of the 1d evaluation functions used in
// CUDAWrappers::FEEvaluation. These functions are marked 'internal' but it is
// much easier to check their correctness directly rather than from the results
// in dependent functions

#include <deal.II/lac/cuda_vector.h>
#include <deal.II/lac/read_write_vector.h>

#include <deal.II/matrix_free/cuda_fe_evaluation.h>

#include <fstream>
#include <iostream>

#include "../tests.h"

namespace CUDA = LinearAlgebra::CUDAWrappers;

template <int M, int N, int type, bool add, bool dof_to_quad>
__global__ void
evaluate_tensor_product(double *dst, double *src)
{
  CUDAWrappers::internal::EvaluatorTensorProduct<
    CUDAWrappers::internal::evaluate_general,
    1,
    M - 1,
    N,
    double>
    evaluator(0);

  if (type == 0)
    evaluator.template values<0, dof_to_quad, add, false>(
      CUDAWrappers::internal::get_global_shape_values<double>(0), src, dst);
  if (type == 1)
    evaluator.template gradients<0, dof_to_quad, add, false>(
      CUDAWrappers::internal::get_global_shape_values<double>(0), src, dst);
}

template <int M, int N, int type, bool add>
void
test()
{
  deallog << "Test " << M << " x " << N << std::endl;
  LinearAlgebra::ReadWriteVector<double> shape_host(M * N);
  for (unsigned int i = 0; i < (M + 1) / 2; ++i)
    for (unsigned int j = 0; j < N; ++j)
      {
        shape_host[i * N + j] =
          -1. + 2. * static_cast<double>(Testing::rand()) / RAND_MAX;
        if (type == 1)
          shape_host[(M - 1 - i) * N + N - 1 - j] = -shape_host[i * N + j];
        else
          shape_host[(M - 1 - i) * N + N - 1 - j] = shape_host[i * N + j];
      }
  if (type == 0 && M % 2 == 1 && N % 2 == 1)
    {
      for (unsigned int i = 0; i < M; ++i)
        shape_host[i * N + N / 2] = 0.;
      shape_host[M / 2 * N + N / 2] = 1.;
    }
  if (type == 1 && M % 2 == 1 && N % 2 == 1)
    shape_host[M / 2 * N + N / 2] = 0.;

  LinearAlgebra::ReadWriteVector<double> x_host(N), x_ref(N), y_host(M),
    y_ref(M);
  for (unsigned int i = 0; i < N; ++i)
    x_host[i] = static_cast<double>(Testing::rand()) / RAND_MAX;

  // Compute reference
  for (unsigned int i = 0; i < M; ++i)
    {
      y_host[i] = 1.;
      y_ref[i]  = add ? y_host[i] : 0.;
      for (unsigned int j = 0; j < N; ++j)
        y_ref[i] += shape_host[i * N + j] * x_host[j];
    }

  // Copy data to the GPU.
  CUDA::Vector<double> x_dev(N), y_dev(M);
  x_dev.import(x_host, VectorOperation::insert);
  y_dev.import(y_host, VectorOperation::insert);

  unsigned int size_shape_values = M * N * sizeof(double);

  hipError_t cuda_error =
    hipMemcpyToSymbol(CUDAWrappers::internal::get_global_shape_values<double>(
                         0),
                       shape_host.begin(),
                       size_shape_values,
                       0,
                       hipMemcpyHostToDevice);
  AssertCuda(cuda_error);

  cuda_error = hipMemcpyToSymbol(
    CUDAWrappers::internal::get_global_shape_gradients<double>(0),
    shape_host.begin(),
    size_shape_values,
    0,
    hipMemcpyHostToDevice);
  AssertCuda(cuda_error);

  // Launch the kernel
  evaluate_tensor_product<M, N, type, add, false>
    <<<1, M>>>(y_dev.get_values(), x_dev.get_values());

  // Check the results on the host
  y_host.import(y_dev, VectorOperation::insert);
  deallog << "Errors no transpose: ";
  for (unsigned int i = 0; i < M; ++i)
    deallog << y_host[i] - y_ref[i] << " ";
  deallog << std::endl;

  for (unsigned int i = 0; i < M; ++i)
    y_host[i] = static_cast<double>(Testing::rand()) / RAND_MAX;

  // Copy y_host to the device
  y_dev.import(y_host, VectorOperation::insert);

  // Compute reference
  for (unsigned int i = 0; i < N; ++i)
    {
      x_host[i] = 2.;
      x_ref[i]  = add ? x_host[i] : 0.;
      for (unsigned int j = 0; j < M; ++j)
        x_ref[i] += shape_host[j * N + i] * y_host[j];
    }

  // Copy x_host to the device
  x_dev.import(x_host, VectorOperation::insert);

  // Launch the kernel
  evaluate_tensor_product<M, N, type, add, true>
    <<<1, M>>>(x_dev.get_values(), y_dev.get_values());

  // Check the results on the host
  x_host.import(x_dev, VectorOperation::insert);
  deallog << "Errors transpose:    ";
  for (unsigned int i = 0; i < N; ++i)
    deallog << x_host[i] - x_ref[i] << " ";
  deallog << std::endl;
}

int
main()
{
  std::ofstream logfile("output");
  deallog.attach(logfile);

  init_cuda();

  deallog.push("values");
  test<4, 4, 0, false>();
  test<3, 3, 0, false>();
  // Not supported right now
  // test<4,3,0,false>();
  // test<3,4,0,false>();
  // test<3,5,0,false>();
  deallog.pop();

  deallog.push("gradients");
  test<4, 4, 1, false>();
  test<3, 3, 1, false>();
  // Not supported right now
  // test<4,3,1,false>();
  // test<3,4,1,false>();
  // test<3,5,1,false>();
  deallog.pop();

  deallog.push("add");

  deallog.push("values");
  test<4, 4, 0, true>();
  test<3, 3, 0, true>();
  // Not supported right now
  // test<4,3,0,true>();
  // test<3,4,0,true>();
  // test<3,5,0,true>();
  deallog.pop();

  deallog.push("gradients");
  test<4, 4, 1, true>();
  test<3, 3, 1, true>();
  // Not supported right now
  // test<4,3,1,true>();
  // test<3,4,1,true>();
  // test<3,5,1,true>();
  deallog.pop();

  deallog.pop();

  return 0;
}

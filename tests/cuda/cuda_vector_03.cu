// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the deal.II distribution.
//
// ---------------------------------------------------------------------

#include "../tests.h"
#include <deal.II/lac/cuda_vector.h>
#include <deal.II/lac/read_write_vector.h>

// Check that reinit correctly set all the entries of the vector to zero

void
test()
{
  const unsigned int size = 100;
  LinearAlgebra::CUDAWrappers::Vector<double> a(size);
  LinearAlgebra::ReadWriteVector<double> read_write(size);
  for (unsigned int i=0; i<size; ++i)
    read_write[i] = i;
  a.import(read_write, VectorOperation::insert);

  a.reinit(size/2);
  AssertThrow(a.l1_norm()==0., ExcMessage("reinit did not zero the entry"));
}


int
main(int argc, char **argv)
{
  initlog();
  deallog.depth_console(0);

  test();

  deallog << "OK" <<std::endl;

  return 0;
}

#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2017 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------



// this function tests the correctness of the 2d evaluation functions used in
// CUDAWrappers::FEEvaluation. These functions are marked 'internal' but it is
// much easier to check their correctness directly rather than from the results
// in dependent functions

#include <deal.II/lac/cuda_vector.h>
#include <deal.II/lac/read_write_vector.h>

#include <deal.II/matrix_free/cuda_fe_evaluation.h>

#include <fstream>
#include <iostream>

#include "../tests.h"

namespace CUDA = LinearAlgebra::CUDAWrappers;

template <int M, int N, int type, bool add, bool dof_to_quad>
__global__ void
evaluate_tensor_product(double *dst, double *src)
{
  CUDAWrappers::internal::EvaluatorTensorProduct<
    CUDAWrappers::internal::evaluate_general,
    2,
    M - 1,
    N,
    double>
    evaluator(0);

  if (type == 0)
    {
      evaluator.template values<0, dof_to_quad, false, false>(
        CUDAWrappers::internal::get_global_shape_values<double>(0), src, src);
      __syncthreads();
      evaluator.template values<1, dof_to_quad, add, false>(
        CUDAWrappers::internal::get_global_shape_values<double>(0), src, dst);
    }
  if (type == 1)
    {
      evaluator.template gradients<0, dof_to_quad, false, false>(
        CUDAWrappers::internal::get_global_shape_values<double>(0), src, src);
      __syncthreads();
      evaluator.template gradients<1, dof_to_quad, add, false>(
        CUDAWrappers::internal::get_global_shape_values<double>(0), src, dst);
    }
}

template <int M, int N, int type, bool add>
void
test()
{
  deallog << "Test " << M << " x " << N << std::endl;
  LinearAlgebra::ReadWriteVector<double> shape_host(M * N);
  for (unsigned int i = 0; i < (M + 1) / 2; ++i)
    for (unsigned int j = 0; j < N; ++j)
      {
        shape_host[i * N + j] =
          -1. + 2. * static_cast<double>(Testing::rand()) / RAND_MAX;
        if (type == 1)
          shape_host[(M - 1 - i) * N + N - 1 - j] = -shape_host[i * N + j];
        else
          shape_host[(M - 1 - i) * N + N - 1 - j] = shape_host[i * N + j];
      }
  if (type == 0 && M % 2 == 1 && N % 2 == 1)
    {
      for (unsigned int i = 0; i < M; ++i)
        shape_host[i * N + N / 2] = 0.;
      shape_host[M / 2 * N + N / 2] = 1.;
    }
  if (type == 1 && M % 2 == 1 && N % 2 == 1)
    shape_host[M / 2 * N + N / 2] = 0.;

  constexpr int                          M_2d = M * M;
  constexpr int                          N_2d = N * N;
  LinearAlgebra::ReadWriteVector<double> x_host(N_2d), x_ref(N_2d),
    y_host(M_2d), y_ref(M_2d);
  for (unsigned int i = 0; i < N_2d; ++i)
    x_host[i] = static_cast<double>(Testing::rand()) / RAND_MAX;

  FullMatrix<double> shape_2d(M_2d, N_2d);
  for (unsigned int i = 0; i < M; ++i)
    {
      for (unsigned int j = 0; j < N; ++j)
        {
          const double shape_val = shape_host[i * N + j];
          for (unsigned int m = 0; m < M; ++m)
            for (unsigned int n = 0; n < N; ++n)
              shape_2d(i * M + m, j * N + n) =
                shape_val * shape_host[m * N + n];
        }
    }

  // Compute reference
  for (unsigned int i = 0; i < M_2d; ++i)
    {
      y_host[i] = 1.;
      y_ref[i]  = add ? y_host[i] : 0.;
      for (unsigned int j = 0; j < N_2d; ++j)
        y_ref[i] += shape_2d(i, j) * x_host[j];
    }

  // Copy data to the GPU.
  CUDA::Vector<double> x_dev(N_2d), y_dev(M_2d);
  x_dev.import(x_host, VectorOperation::insert);
  y_dev.import(y_host, VectorOperation::insert);

  unsigned int size_shape_values = M * N * sizeof(double);

  hipError_t cuda_error =
    hipMemcpyToSymbol(CUDAWrappers::internal::get_global_shape_values<double>(
                         0),
                       shape_host.begin(),
                       size_shape_values,
                       0,
                       hipMemcpyHostToDevice);
  AssertCuda(cuda_error);

  cuda_error = hipMemcpyToSymbol(
    CUDAWrappers::internal::get_global_shape_gradients<double>(0),
    shape_host.begin(),
    size_shape_values,
    0,
    hipMemcpyHostToDevice);
  AssertCuda(cuda_error);

  // Launch the kernel
  dim3 block_dim(M, N);
  evaluate_tensor_product<M, N, type, add, false>
    <<<1, block_dim>>>(y_dev.get_values(), x_dev.get_values());

  // Check the results on the host
  y_host.import(y_dev, VectorOperation::insert);
  deallog << "Errors no transpose: ";

  for (unsigned int i = 0; i < M_2d; ++i)
    deallog << y_host[i] - y_ref[i] << " ";
  deallog << std::endl;

  for (unsigned int i = 0; i < M_2d; ++i)
    y_host[i] = static_cast<double>(Testing::rand()) / RAND_MAX;

  // Copy y_host to the device
  y_dev.import(y_host, VectorOperation::insert);

  // Compute reference
  for (unsigned int i = 0; i < N_2d; ++i)
    {
      x_host[i] = 2.;
      x_ref[i]  = add ? x_host[i] : 0.;
      for (unsigned int j = 0; j < M_2d; ++j)
        x_ref[i] += shape_2d(j, i) * y_host[j];
    }

  // Copy x_host to the device
  x_dev.import(x_host, VectorOperation::insert);

  // Launch the kernel
  evaluate_tensor_product<M, N, type, add, true>
    <<<1, block_dim>>>(x_dev.get_values(), y_dev.get_values());

  // Check the results on the host
  x_host.import(x_dev, VectorOperation::insert);
  deallog << "Errors transpose:    ";
  for (unsigned int i = 0; i < N_2d; ++i)
    deallog << x_host[i] - x_ref[i] << " ";
  deallog << std::endl;
}

int
main()
{
  std::ofstream logfile("output");
  deallog.attach(logfile);

  init_cuda();

  deallog.push("values");
  test<4, 4, 0, false>();
  test<3, 3, 0, false>();
  deallog.pop();

  deallog.push("gradients");
  test<4, 4, 1, false>();
  test<3, 3, 1, false>();
  deallog.pop();

  deallog.push("add");

  deallog.push("values");
  test<4, 4, 0, true>();
  test<3, 3, 0, true>();
  deallog.pop();

  deallog.push("gradients");
  test<4, 4, 1, true>();
  test<3, 3, 1, true>();
  deallog.pop();

  deallog.pop();

  return 0;
}

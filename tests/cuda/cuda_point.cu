#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2019 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Test that Point operations on a CUDA device can be used.

#include <deal.II/base/point.h>

#include "../tests.h"

template <int dim, typename Number>
__global__ void
miscellaneous_kernel(Number check[16])
{
  Point<dim, Number> p_1;
  check[0] = p_1.norm_square();
  Point<dim, Number> p_2(Tensor<1, dim, Number>{});
  check[1] = p_2.norm_square();
  if (dim == 1)
    {
      Point<dim, Number> p(1.);
      check[2] = p.norm_square();
    }
  if (dim == 2)
    {
      Point<dim, Number> p(.6, .8);
      check[2] = p.norm_square();
    }
  if (dim == 3)
    {
      Point<dim, Number> p(.48, .64, .6);
      check[2] = p.norm_square();
    }

  auto p_3 = Point<dim, Number>::unit_vector(0);
  check[3] = p_3.norm_square();

  auto entry_1 = p_1(0);
  check[4]     = entry_1;
  p_1(0)       = Number{1.};
  check[5]     = p_1.norm_square();
  auto p_4     = p_1 + Tensor<1, dim, Number>{};
  check[6]     = p_4.norm_square();
  auto p_5     = p_1 - Tensor<1, dim, Number>{};
  check[7]     = p_5.norm_square();
  auto t_1     = p_1 - p_2;
  check[8]     = t_1.norm_square();
  auto p_6     = -p_3;
  check[9]     = p_6.norm_square();
  auto p_7     = p_4 / 2.;
  check[10]    = p_7.norm_square();
  auto p_8     = p_7 * 5.;
  check[11]    = p_8.norm_square();

  auto s_1  = p_1 * t_1;
  check[12] = s_1;
  auto s_2  = p_2.square();
  check[13] = s_2;
  auto s_3  = p_3.distance(p_5);
  check[14] = s_3;
  auto s_4  = p_4.distance_square(p_1);
  check[15] = s_4;
}

template <int dim, typename Number>
void
test_gpu()
{
  Number *           check;
  const unsigned int n_tests = 16;

  auto cuda_error = hipMalloc(&check, n_tests * sizeof(Number));
  AssertCuda(cuda_error);

  // Miscellaneous
  miscellaneous_kernel<dim, Number><<<1, 1>>>(check);
  // Check that the kernel was launched correctly
  AssertCuda(hipPeekAtLastError());
  // Check that there was no problem during the execution of the kernel
  AssertCuda(hipDeviceSynchronize());

  std::vector<Number> check_host(n_tests);

  cuda_error = hipMemcpy(check_host.data(),
                          check,
                          n_tests * sizeof(Number),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);

  const double tolerance = 1.e-8;
  AssertThrow(std::abs(check_host[0] - 0.) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[1] - 0.) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[2] - 1.) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[3] - 1.) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[4] - 0.) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[5] - 1.) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[6] - 1.) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[7] - 1.) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[8] - 1.) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[9] - 1.) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[10] - .25) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[11] - 6.25) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[12] - 1.) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[13] - 0.) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[14] - 0.) < tolerance, ExcInternalError());
  AssertThrow(std::abs(check_host[15] - 0.) < tolerance, ExcInternalError());

  cuda_error = hipFree(check);
  AssertCuda(cuda_error);

  deallog << "OK" << std::endl;
}

int
main()
{
  initlog();

  init_cuda();

  test_gpu<1, double>();
  test_gpu<2, double>();
  test_gpu<3, double>();
  test_gpu<1, float>();
  test_gpu<2, float>();
  test_gpu<3, float>();
}

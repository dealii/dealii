// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Check that dealii::SolverCG works with CUDAWrappers::SparseMatrix
// and PreconditionIC

#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/exceptions.h>

#include <deal.II/lac/cuda_precondition.h>
#include <deal.II/lac/cuda_sparse_matrix.h>
#include <deal.II/lac/read_write_vector.h>
#include <deal.II/lac/solver_cg.h>
#include <deal.II/lac/solver_control.h>

#include "../tests.h"

#include "../testmatrix.h"

template <typename Number>
void
test(Utilities::CUDA::Handle &cuda_handle)
{
  // Build the sparse matrix on the host
  const unsigned int   problem_size = 10;
  unsigned int         size         = (problem_size - 1) * (problem_size - 1);
  FDMatrix             testproblem(problem_size, problem_size);
  SparsityPattern      structure(size, size, 5);
  SparseMatrix<Number> A;
  testproblem.five_point_structure(structure);
  structure.compress();
  A.reinit(structure);
  testproblem.five_point(A);

  // Solve on the device
  CUDAWrappers::SparseMatrix<Number>          A_dev(cuda_handle, A);
  LinearAlgebra::CUDAWrappers::Vector<Number> sol_dev(size);
  LinearAlgebra::CUDAWrappers::Vector<Number> rhs_dev(size);
  LinearAlgebra::ReadWriteVector<Number>      rw_vector(size);
  for (unsigned int i = 0; i < size; ++i)
    rw_vector[i] = static_cast<Number>(i);
  rhs_dev.import(rw_vector, VectorOperation::insert);
  const Number  tolerance = 1000. * std::numeric_limits<Number>::epsilon();
  SolverControl control(100, tolerance);
  SolverCG<LinearAlgebra::CUDAWrappers::Vector<Number>> cg_dev(control);

  CUDAWrappers::PreconditionIC<Number> prec_ic(cuda_handle);
  prec_ic.initialize(A_dev);

  cg_dev.solve(A_dev, sol_dev, rhs_dev, prec_ic);

  // Check the result
  LinearAlgebra::CUDAWrappers::Vector<Number> residual(size);
  A_dev.residual(residual, sol_dev, rhs_dev);
  Assert(residual.l2_norm() < 20 * tolerance, ExcInternalError());
  deallog << "OK" << std::endl;
}

int
main()
{
  initlog();
  deallog << std::setprecision(10);
  deallog.depth_console(0);

  init_cuda();

  Utilities::CUDA::Handle cuda_handle;
  deallog << "Testing float" << std::endl;
  test<float>(cuda_handle);
  deallog << "Testing double" << std::endl;
  test<double>(cuda_handle);

  deallog << "OK" << std::endl;

  return 0;
}

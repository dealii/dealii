#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2016 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the deal.II distribution.
//
// ---------------------------------------------------------------------

// Test operator[] and norm_square of cuda_tensor.

#include "../tests.h"
#include <deal.II/base/tensor.h>
#include <deal.II/base/logstream.h>
#include <fstream>
#include <iomanip>

void
test_cpu()
{
  double a[3][3] = {{1, 2, 3}, {4, 5, 6}, {7, 8, 9}};
  const unsigned int dim=3;
  Tensor<2,dim> t;
  for (unsigned int i=0; i<dim; ++i)
    for (unsigned int j=0; j<dim; ++j)
      t[i][j] = a[i][j];


  deallog.push("values");
  for (unsigned int i=0; i<dim; ++i)
    for (unsigned int j=0; j<dim; ++j)
      deallog << t[i][j] << std::endl;
  deallog.pop();

  deallog.push("norm_square");
  deallog << t.norm_square() << std::endl;
  deallog.pop();
}

__global__ void
init_kernel(Tensor<2,3> *t,
            const unsigned int N)
{
  const unsigned int i = threadIdx.y;
  const unsigned int j = threadIdx.x;
  if ((i < N) && (j < N))
    (*t)[i][j] = j + i*N + 1.;
}

__global__ void
norm_kernel(Tensor<2,3> *t, double *norm)
{
  if (threadIdx.x == 0)
    *norm = t->norm_square();
}

void
test_gpu()
{
  const unsigned int dim=3;
  double *norm_dev;
  double norm_host;
  Tensor<2,dim> *t_dev;

  // Allocate objects on the device
  hipError_t cuda_error = hipMalloc(&t_dev, sizeof(Tensor<2,dim>));
  AssertCuda(cuda_error);
  cuda_error = hipMalloc(&norm_dev, sizeof(double));
  AssertCuda(cuda_error);

  // Launch the kernels.
  dim3 block_dim(dim, dim);
  init_kernel<<<1,block_dim>>>(t_dev, dim);
  norm_kernel<<<1,1>>>(t_dev, norm_dev);

  // Copy the result to the device
  cuda_error = hipMemcpy(&norm_host, norm_dev, sizeof(double),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);

  // Free memory
  cuda_error = hipFree(t_dev);
  AssertCuda(cuda_error);
  cuda_error = hipFree(norm_dev);
  AssertCuda(cuda_error);

  // Output result
  deallog.push("norm_square GPU");
  deallog << norm_host << std::endl;
}

int
main ()
{
  std::ofstream logfile("output");
  deallog << std::setprecision(5);
  deallog.attach(logfile);

  test_cpu();

  test_gpu();
}

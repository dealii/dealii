#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2016 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Test operator[], norm and norm_square of cuda_tensor.

#include <deal.II/base/tensor.h>

#include "../tests.h"

void
test_cpu()
{
  double             a[3][3] = {{1, 2, 3}, {4, 5, 6}, {7, 8, 9}};
  const unsigned int dim     = 3;
  Tensor<2, dim>     t;
  for (unsigned int i = 0; i < dim; ++i)
    for (unsigned int j = 0; j < dim; ++j)
      t[i][j] = a[i][j];

  deallog.push("values");
  for (unsigned int i = 0; i < dim; ++i)
    for (unsigned int j = 0; j < dim; ++j)
      deallog << t[i][j] << std::endl;
  deallog.pop();

  deallog << "norm: " << t.norm() << std::endl;
  deallog << "norm_square: " << t.norm_square() << std::endl;
}

__global__ void init_kernel(Tensor<2, 3> *t, const unsigned int N)
{
  const unsigned int i = threadIdx.y;
  const unsigned int j = threadIdx.x;
  if ((i < N) && (j < N))
    (*t)[i][j] = j + i * N + 1.;
}

__global__ void norm_kernel(Tensor<2, 3> *t, double *norm, double *norm_square)
{
  if (threadIdx.x == 0)
    {
      *norm        = t->norm();
      *norm_square = t->norm_square();
    }
}

void
test_gpu()
{
  const unsigned int dim = 3;
  double *           norm_dev;
  double             norm_host;
  double *           norm_square_dev;
  double             norm_square_host;
  Tensor<2, dim> *   t_dev;

  // Allocate objects on the device
  hipError_t cuda_error = hipMalloc(&t_dev, sizeof(Tensor<2, dim>));
  AssertCuda(cuda_error);
  cuda_error = hipMalloc(&norm_dev, sizeof(double));
  AssertCuda(cuda_error);
  cuda_error = hipMalloc(&norm_square_dev, sizeof(double));
  AssertCuda(cuda_error);

  // Launch the kernels.
  dim3 block_dim(dim, dim);
  init_kernel<<<1, block_dim>>>(t_dev, dim);
  norm_kernel<<<1, 1>>>(t_dev, norm_dev, norm_square_dev);

  // Copy the result to the device
  cuda_error =
    hipMemcpy(&norm_host, norm_dev, sizeof(double), hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);
  cuda_error = hipMemcpy(&norm_square_host,
                          norm_square_dev,
                          sizeof(double),
                          hipMemcpyDeviceToHost);
  AssertCuda(cuda_error);

  // Free memory
  cuda_error = hipFree(t_dev);
  AssertCuda(cuda_error);
  cuda_error = hipFree(norm_dev);
  AssertCuda(cuda_error);
  cuda_error = hipFree(norm_square_dev);
  AssertCuda(cuda_error);

  // Output result
  deallog << "norm GPU: " << norm_host << std::endl;
  deallog << "norm_square GPU: " << norm_square_host << std::endl;
}

int
main()
{
  initlog();

  init_cuda();

  test_cpu();

  test_gpu();
}

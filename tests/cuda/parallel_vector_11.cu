#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------


// check that add, sadd, equ, scale work correctly on a vector where some
// processor do not own any degrees of freedom

#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/index_set.h>
#include <deal.II/base/utilities.h>

#include <deal.II/lac/la_parallel_vector.h>
#include <deal.II/lac/read_write_vector.h>

#include <iostream>
#include <vector>

#include "../tests.h"


void
test()
{
  unsigned int myid    = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  unsigned int numproc = Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD);

  if (myid == 0)
    deallog << "numproc=" << numproc << std::endl;

  // global size: 20, local_size: 3 as long as
  // less than 20
  const unsigned int local_size  = 3;
  const unsigned int global_size = std::min(20U, local_size * numproc);
  const int          my_start    = std::min(local_size * myid, global_size);
  const int          my_end = std::min(local_size * (myid + 1), global_size);
  const int          actual_local_size = my_end - my_start;

  IndexSet local_owned(global_size);
  if (my_end > my_start)
    local_owned.add_range(static_cast<unsigned int>(my_start),
                          static_cast<unsigned int>(my_end));
  IndexSet local_relevant(global_size);
  local_relevant = local_owned;
  local_relevant.add_index(2);

  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> v(
    local_owned, local_relevant, MPI_COMM_WORLD);
  AssertDimension(static_cast<unsigned int>(actual_local_size), v.local_size());
  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> w(v), x(v),
    y(v);

  // set local elements
  LinearAlgebra::ReadWriteVector<double> v_rw(local_owned);
  LinearAlgebra::ReadWriteVector<double> w_rw(local_owned);
  LinearAlgebra::ReadWriteVector<double> x_rw(local_owned);
  for (int i = 0; i < actual_local_size; ++i)
    {
      v_rw.local_element(i) = i + my_start;
      w_rw.local_element(i) = 1000 + 2 * (my_start + i);
      x_rw.local_element(i) = 10000;
    }
  v.import(v_rw, VectorOperation::insert);
  w.import(w_rw, VectorOperation::insert);
  x.import(x_rw, VectorOperation::insert);

  y = v;
  LinearAlgebra::ReadWriteVector<double> y_rw(local_owned);
  y_rw.import(y, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    AssertThrow(y_rw.local_element(i) == i + my_start, ExcInternalError());

  if (myid == 0)
    deallog << "Check add (scalar): ";
  y.add(42);
  y_rw.import(y, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    AssertThrow(y_rw.local_element(i) == i + my_start + 42, ExcInternalError());
  if (myid == 0)
    deallog << "OK" << std::endl;

  if (myid == 0)
    deallog << "Check add (vector): ";
  y.add(1., w);
  y_rw.import(y, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    AssertThrow(y_rw.local_element(i) == 3 * (i + my_start) + 1042,
                ExcInternalError());
  if (myid == 0)
    deallog << "OK" << std::endl;

  if (myid == 0)
    deallog << "Check add (factor, vector): ";
  y.add(-1., w);
  y_rw.import(y, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    AssertThrow(y_rw.local_element(i) == i + my_start + 42, ExcInternalError());
  if (myid == 0)
    deallog << "OK" << std::endl;

  if (myid == 0)
    deallog << "Check add (factor, vector, factor, vector): ";
  y.add(2., w, -0.5, x);
  y_rw.import(y, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    AssertThrow(y_rw.local_element(i) == 5 * (i + my_start) + 2042 - 5000,
                ExcInternalError());
  if (myid == 0)
    deallog << "OK" << std::endl;

  if (myid == 0)
    deallog << "Check sadd (factor, factor, vector): ";
  y = v;
  y.sadd(-3., 2., v);
  y_rw.import(y, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    AssertThrow(y_rw.local_element(i) == (-i - my_start), ExcInternalError());
  if (myid == 0)
    deallog << "OK" << std::endl;

  if (myid == 0)
    deallog << "Check sadd (factor, factor, vector, factor, vector): ";
  y.sadd(2., 3., v);
  y.add(2., w);
  y_rw.import(y, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    {
      AssertThrow(y_rw.local_element(i) == 5 * (i + my_start) + 2000,
                  ExcInternalError());
    }
  if (myid == 0)
    deallog << "OK" << std::endl;

  if (myid == 0)
    deallog
      << "Check sadd (factor, factor, vector, factor, vector, factor, vector): ";
  y.sadd(-1., 1., v);
  y.add(2., w);
  y.add(2., x);
  y_rw.import(y, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    AssertThrow(y_rw.local_element(i) == 20000, ExcInternalError());
  if (myid == 0)
    deallog << "OK" << std::endl;

  if (myid == 0)
    deallog << "Check add (factor, vector_1, factor, vector_1): ";
  y = 0;
  y.add(1., v, 3., v);
  y_rw.import(y, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    AssertThrow(y_rw.local_element(i) == 4 * (i + my_start),
                ExcInternalError());
  if (myid == 0)
    deallog << "OK" << std::endl;

  if (myid == 0)
    deallog << "Check operator * (scalar): ";
  x *= 2.;
  x_rw.import(x, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    AssertThrow(x_rw.local_element(i) == 20000., ExcInternalError());
  if (myid == 0)
    deallog << "OK" << std::endl;

  if (myid == 0)
    deallog << "Check operator / (scalar): ";
  x /= 2.;
  x_rw.import(x, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    AssertThrow(x_rw.local_element(i) == 10000., ExcInternalError());
  if (myid == 0)
    deallog << "OK" << std::endl;

  if (myid == 0)
    deallog << "Check scale (vector): ";
  y.scale(x);
  y_rw.import(y, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    AssertThrow(y_rw.local_element(i) == 40000. * (i + my_start),
                ExcInternalError());
  if (myid == 0)
    deallog << "OK" << std::endl;

  if (myid == 0)
    deallog << "Check equ (factor, vector): ";
  y.equ(10., x);
  y_rw.import(y, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    AssertThrow(y_rw.local_element(i) == 100000., ExcInternalError());
  if (myid == 0)
    deallog << "OK" << std::endl;

  if (myid == 0)
    deallog << "Check equ (factor, vector, factor, vector): ";
  y.equ(10., v);
  y.add(-2., w);
  y_rw.import(y, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    AssertThrow(y_rw.local_element(i) == 6. * (i + my_start) - 2000,
                ExcInternalError());
  if (myid == 0)
    deallog << "OK" << std::endl;

  if (myid == 0)
    deallog << "Check equ (factor, vector, factor, vector, factor, vector): ";
  y.equ(10., v);
  y.add(-2., w);
  y.add(3., x);
  y_rw.import(y, VectorOperation::insert);
  for (int i = 0; i < actual_local_size; ++i)
    AssertThrow(y_rw.local_element(i) == 6. * (i + my_start) + 28000,
                ExcInternalError());
  if (myid == 0)
    deallog << "OK" << std::endl;
}



int
main(int argc, char **argv)
{
  Utilities::MPI::MPI_InitFinalize mpi_initialization(
    argc, argv, testing_max_num_threads());

  unsigned int myid = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  deallog.push(Utilities::int_to_string(myid));

  init_cuda(true);

  if (myid == 0)
    {
      initlog();
      deallog << std::setprecision(4);

      test();
    }
  else
    test();
}

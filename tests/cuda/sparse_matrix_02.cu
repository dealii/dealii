// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Check print and print_format

#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/exceptions.h>

#include <deal.II/lac/cuda_sparse_matrix.h>

#include "../tests.h"

#include "../testmatrix.h"


void
test(Utilities::CUDA::Handle &cuda_handle)
{
  // Build the sparse matrix on the host
  const unsigned int size = 3;
  unsigned int       dim  = (size - 1) * (size - 1);

  FDMatrix        testproblem(size, size);
  SparsityPattern structure(dim, dim, 5);
  testproblem.five_point_structure(structure);
  structure.compress();
  SparseMatrix<double> A(structure);
  testproblem.upwind(A, true);
  A.print(deallog.get_file_stream());
  A.print_formatted(deallog.get_file_stream());

  // Create the sparse matrix on the device
  CUDAWrappers::SparseMatrix<double> A_dev(cuda_handle, A);
  A_dev.print(deallog.get_file_stream());
  A_dev.print_formatted(deallog.get_file_stream());
}

int
main()
{
  initlog();
  deallog.depth_console(0);

  init_cuda();

  Utilities::CUDA::Handle cuda_handle;

  test(cuda_handle);

  deallog << "OK" << std::endl;

  return 0;
}

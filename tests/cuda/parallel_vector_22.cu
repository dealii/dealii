#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the deal.II distribution.
//
// ---------------------------------------------------------------------


// check LA::Vector::compress(VectorOperation::min/max) from ghosts

#include <deal.II/base/index_set.h>
#include <deal.II/base/utilities.h>

#include <deal.II/lac/la_parallel_vector.h>

#include <iostream>
#include <vector>

#include "../tests.h"


__global__ void
set_value(double *values_dev, unsigned int index, double val)
{
  values_dev[index] = val;
}


template <typename Number>
double
print_value(Number *values_dev, unsigned int index)
{
  static std::vector<Number> cpu_value(1);
  Utilities::CUDA::copy_to_host(values_dev + index, cpu_value);
  return cpu_value[0];
}



void
test()
{
  unsigned int myid    = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  unsigned int numproc = Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD);

  if (myid == 0)
    deallog << "numproc=" << numproc << std::endl;


  // each processor owns 2 indices and all
  // are ghosting element 1 (the second)
  IndexSet local_owned(numproc * 2);
  local_owned.add_range(myid * 2, myid * 2 + 2);
  IndexSet local_relevant(numproc * 2);
  local_relevant = local_owned;
  local_relevant.add_range(1, 2);

  // create vector
  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> v(
    local_owned, local_relevant, MPI_COMM_WORLD);
  const auto &partitioner = v.get_partitioner();

  // set local values
  set_value<<<1, 1>>>(v.get_values(),
                      partitioner->global_to_local(myid * 2),
                      myid * 2.0);
  set_value<<<1, 1>>>(v.get_values(),
                      partitioner->global_to_local(myid * 2 + 1),
                      myid * 2.0 + 1.0);
  v.compress(VectorOperation::add);
  v *= 2.0;

  // check setup of vectors
  deallog << myid << ":"
          << "first owned entry: "
          << print_value(v.get_values(), partitioner->global_to_local(myid * 2))
          << std::endl;
  deallog << myid << ":"
          << "second owned entry: "
          << print_value(v.get_values(),
                         partitioner->global_to_local(myid * 2 + 1))
          << std::endl;

  // set ghost dof on owning processor and maximize
  if (myid != 0)
    set_value<<<1, 1>>>(v.get_values(),
                        partitioner->global_to_local(1),
                        7. * myid);
  v.compress(VectorOperation::max);

  // import ghosts onto all procs
  v.update_ghost_values();

  // check
  deallog << myid << ":"
          << "ghost entry after max from owner: "
          << print_value(v.get_values(), partitioner->global_to_local(1))
          << std::endl;

  // ghosts are set to zero
  v.zero_out_ghost_values();

  // minimize
  v.compress(VectorOperation::min);
  v.update_ghost_values();

  // check
  deallog << myid << ":"
          << "ghost entry after min from zero: "
          << print_value(v.get_values(), partitioner->global_to_local(1))
          << std::endl;

  // set ghost dof on non-owning processors and minimize
  v.zero_out_ghost_values();
  if (myid == 0)
    set_value<<<1, 1>>>(v.get_values(), partitioner->global_to_local(1), -1.);
  v.compress(VectorOperation::min);
  v.update_ghost_values();

  // check
  deallog << myid << ":"
          << "ghost entry after min from : "
          << print_value(v.get_values(), partitioner->global_to_local(1))
          << std::endl;

  // set vector to 1, zeros in ghosts except on owner where -1. is set
  v.zero_out_ghost_values();
  v = 1.0;
  if (myid == 0)
    set_value<<<1, 1>>>(v.get_values(), partitioner->global_to_local(1), -1.);

  // maximize
  v.compress(VectorOperation::max);
  v.update_ghost_values();

  // even if only one value is set (-1. on owner), the other values
  // contribute a "0" and maximization receives zero and returns it
  deallog << myid << ":"
          << "ghost entry after max and partly init: "
          << print_value(v.get_values(), partitioner->global_to_local(1))
          << std::endl;

  // however, if the ghost value is set on all processors, the
  // maximum is -1:
  v.zero_out_ghost_values();
  v = 1.0;
  set_value<<<1, 1>>>(v.get_values(), partitioner->global_to_local(1), -1.);
  v.compress(VectorOperation::max);
  v.update_ghost_values();
  deallog << myid << ":"
          << "ghost entry after max and full init: "
          << print_value(v.get_values(), partitioner->global_to_local(1))
          << std::endl;

  // what happens in case max is called two times and all values were smaller
  // than zero
  v.zero_out_ghost_values();
  v = -1.0;
  set_value<<<1, 1>>>(v.get_values(), partitioner->global_to_local(1), -1.);
  v.compress(VectorOperation::max);
  deallog << myid << ":"
          << "ghost entry after first max: "
          << print_value(v.get_values(), partitioner->global_to_local(1))
          << std::endl;
  v.compress(VectorOperation::max);
  deallog << myid << ":"
          << "ghost entry after second max: "
          << print_value(v.get_values(), partitioner->global_to_local(1))
          << std::endl;

  if (myid == 0)
    deallog << "OK" << std::endl;
}



int
main(int argc, char **argv)
{
  Utilities::MPI::MPI_InitFinalize mpi_initialization(
    argc, argv, testing_max_num_threads());

  MPILogInitAll log;

  init_cuda(true);

  test();
}

#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2019 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------


// Test that we can assign LA::d::Vector from one MemorySpace to another one


#include <deal.II/base/exceptions.h>

#include <deal.II/lac/la_parallel_vector.h>

#include <cmath>

#include "../tests.h"


using namespace dealii;

template <typename Number>
void
test()
{
  const unsigned int                                            size = 100;
  LinearAlgebra::distributed::Vector<Number, MemorySpace::Host> vec_ref(size);
  for (unsigned int i = 0; i < size; ++i)
    vec_ref[i] = i;

  // Assignment from Host to CUDA
  LinearAlgebra::distributed::Vector<Number, MemorySpace::CUDA> vec_dev;
  vec_dev = vec_ref;
  vec_dev *= 2;

  // Assignment from CUDA to HOST
  LinearAlgebra::distributed::Vector<Number, MemorySpace::Host> vec_host;
  vec_host = vec_dev;

  vec_ref *= 2;

  for (unsigned int i = 0; i < size; ++i)
    AssertThrow(std::fabs(vec_ref[i] - vec_host[i]) < 1e-12,
                ExcInternalError());
}


int
main(int argc, char *argv[])
{
  initlog();
  Utilities::MPI::MPI_InitFinalize mpi_init(argc, argv, 1);

  test<float>();
  test<double>();

  deallog << "OK" << std::endl;

  return 0;
}

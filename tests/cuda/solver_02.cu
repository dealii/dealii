#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Check that CUDA direct solvers work

#include <deal.II/base/hip/hip_runtime.h>

#include <deal.II/lac/cuda_solver_direct.h>
#include <deal.II/lac/cuda_sparse_matrix.h>
#include <deal.II/lac/cuda_vector.h>
#include <deal.II/lac/read_write_vector.h>
#include <deal.II/lac/solver_control.h>
#include <deal.II/lac/vector.h>

#include "../tests.h"

#include "../testmatrix.h"


void
test(Utilities::CUDA::Handle &cuda_handle)
{
  // Create the matrix on the host.
  dealii::SparsityPattern                sparsity_pattern;
  dealii::SparseMatrix<double>           matrix;
  unsigned int const                     size = 30;
  std::vector<std::vector<unsigned int>> column_indices(size);
  for (unsigned int i = 0; i < size; ++i)
    {
      unsigned int j_max = std::min(size, i + 2);
      unsigned int j_min = (i == 0) ? 0 : i - 1;
      for (unsigned int j = j_min; j < j_max; ++j)
        column_indices[i].emplace_back(j);
    }
  sparsity_pattern.copy_from(size,
                             size,
                             column_indices.begin(),
                             column_indices.end());
  matrix.reinit(sparsity_pattern);
  for (unsigned int i = 0; i < size; ++i)
    {
      unsigned int j_max = std::min(size - 1, i + 1);
      unsigned int j_min = (i == 0) ? 0 : i - 1;
      matrix.set(i, j_min, -1.);
      matrix.set(i, j_max, -1.);
      matrix.set(i, i, 4.);
    }

  // Generate a random solution and then compute the rhs
  dealii::Vector<double> sol_ref(size);
  for (auto &val : sol_ref)
    val = random_value(5., 15.);

  dealii::Vector<double> rhs(size);
  matrix.vmult(rhs, sol_ref);

  // Move the matrix and the rhs to the host
  CUDAWrappers::SparseMatrix<double> matrix_dev(cuda_handle, matrix);

  LinearAlgebra::CUDAWrappers::Vector<double> rhs_dev(size);
  LinearAlgebra::ReadWriteVector<double>      rhs_host(size);
  std::copy(rhs.begin(), rhs.end(), rhs_host.begin());
  rhs_dev.import(rhs_host, VectorOperation::insert);

  LinearAlgebra::CUDAWrappers::Vector<double> solution_dev(size);
  const std::array<std::string, 3>            solver_names{"Cholesky",
                                                "LU_dense",
                                                "LU_host"};

  for (auto solver_type : solver_names)
    {
      // Solve on the device
      CUDAWrappers::SolverDirect<double>::AdditionalData data(solver_type);
      SolverControl                                      solver_control;

      CUDAWrappers::SolverDirect<double> solver(cuda_handle,
                                                solver_control,
                                                data);
      solver.solve(matrix_dev, solution_dev, rhs_dev);

      // Move the result back to the host
      LinearAlgebra::ReadWriteVector<double> solution_host(size);
      solution_host.import(solution_dev, VectorOperation::insert);

      // Check the result
      for (unsigned int i = 0; i < size; ++i)
        AssertThrow(std::abs(solution_host[i] - sol_ref[i]) < 1e-12,
                    ExcInternalError());
      deallog << solver_type << std::endl;
    }
}

int
main()
{
  initlog();
  deallog.depth_console(0);

  init_cuda();

  Utilities::CUDA::Handle cuda_handle;
  test(cuda_handle);

  deallog << "OK" << std::endl;

  return 0;
}

#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------


// check correct initialization of parallel vector without any ghosts

#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/index_set.h>
#include <deal.II/base/utilities.h>

#include <deal.II/lac/la_parallel_vector.h>
#include <deal.II/lac/read_write_vector.h>

#include <iostream>
#include <vector>

#include "../tests.h"


void
test()
{
  unsigned int myid    = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  unsigned int numproc = Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD);

  if (myid == 0)
    deallog << "numproc=" << numproc << std::endl;


  // each processor owns 2 indices and all
  // are ghosting element 1 (the second)
  IndexSet local_owned(numproc * 2);
  local_owned.add_range(myid * 2, myid * 2 + 2);

  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> v(
    local_owned, local_owned, MPI_COMM_WORLD);

  // set local values
  LinearAlgebra::ReadWriteVector<double> rw_vector(local_owned);
  rw_vector(myid * 2)     = myid * 2.0;
  rw_vector(myid * 2 + 1) = myid * 2.0 + 1.0;
  v.import(rw_vector, VectorOperation::insert);

  v *= 2.0;

  rw_vector.import(v, VectorOperation::insert);
  if (myid == 0)
    {
      deallog << myid * 2 << ":" << rw_vector(myid * 2) << std::endl;
      deallog << myid * 2 + 1 << ":" << rw_vector(myid * 2 + 1) << std::endl;
    }

  DEAL_II_Assert(rw_vector(myid * 2) == myid * 4.0, ExcInternalError());
  DEAL_II_Assert(rw_vector(myid * 2 + 1) == myid * 4.0 + 2.0,
                 ExcInternalError());

  // check l2 norm
  const double l2_norm = v.l2_norm();
  if (myid == 0)
    deallog << "L2 norm: " << l2_norm << std::endl;

  if (myid == 0)
    deallog << "OK" << std::endl;
}



int
main(int argc, char **argv)
{
  Utilities::MPI::MPI_InitFinalize mpi_initialization(
    argc, argv, testing_max_num_threads());

  unsigned int myid = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  deallog.push(Utilities::int_to_string(myid));

  Utilities::CUDA::Handle cuda_handle;
  // By default, all the ranks will try to access the device 0. This is fine if
  // we have one rank per node _and_ one gpu per node. If we have multiple GPUs
  // on one node, we need each process to access a different GPU. We assume that
  // each node has the same number of GPUs.
  int         n_devices       = 0;
  hipError_t cuda_error_code = hipGetDeviceCount(&n_devices);
  DEAL_II_AssertCuda(cuda_error_code);
  int device_id   = myid % n_devices;
  cuda_error_code = hipSetDevice(device_id);
  DEAL_II_AssertCuda(cuda_error_code);


  if (myid == 0)
    {
      initlog();
      deallog << std::setprecision(4);

      test();
    }
  else
    test();
}

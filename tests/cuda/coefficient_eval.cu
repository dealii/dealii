#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2020 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Test that we execute the loop in the same order on the CPU and the GPU

#include <deal.II/fe/fe_q.h>

#include <deal.II/grid/grid_generator.h>
#include <deal.II/grid/manifold_lib.h>
#include <deal.II/grid/tria.h>

#include <deal.II/lac/cuda_vector.h>
#include <deal.II/lac/la_parallel_vector.h>

#include <deal.II/matrix_free/cuda_matrix_free.templates.h>

#include "../tests.h"


template <int dim, int fe_degree>
class DummyOperator
{
public:
  DummyOperator() = default;

  __device__ void
  operator()(
    const unsigned int                                          cell,
    const typename CUDAWrappers::MatrixFree<dim, double>::Data *gpu_data,
    CUDAWrappers::SharedData<dim, double> *                     shared_data,
    const double *                                              src,
    double *                                                    dst) const;

  static const unsigned int n_dofs_1d = fe_degree + 1;
  static const unsigned int n_local_dofs =
    dealii::Utilities::pow(fe_degree + 1, dim);
  static const unsigned int n_q_points =
    dealii::Utilities::pow(fe_degree + 1, dim);
};



template <int dim, int fe_degree>
__device__ void
DummyOperator<dim, fe_degree>::
operator()(const unsigned int                                          cell,
           const typename CUDAWrappers::MatrixFree<dim, double>::Data *gpu_data,
           CUDAWrappers::SharedData<dim, double> *,
           const double *,
           double *dst) const
{
  const unsigned int pos = CUDAWrappers::local_q_point_id<dim, double>(
    cell, gpu_data, n_dofs_1d, n_q_points);
  auto point = CUDAWrappers::get_quadrature_point<dim, double>(cell,
                                                               gpu_data,
                                                               fe_degree + 1);
  dst[pos]   = dim == 2 ? point(0) + point(1) : point(0) + point(1) + point(2);
}



template <int dim, int fe_degree>
class DummyMatrixFree : public Subscriptor
{
public:
  DummyMatrixFree(const CUDAWrappers::MatrixFree<dim, double> &data_in,
                  const unsigned int                           size);
  void
  eval(LinearAlgebra::CUDAWrappers::Vector<double> &dst) const;

private:
  const CUDAWrappers::MatrixFree<dim, double> &data;
};

template <int dim, int fe_degree>
DummyMatrixFree<dim, fe_degree>::DummyMatrixFree(
  const CUDAWrappers::MatrixFree<dim, double> &data_in,
  const unsigned int                           size)
  : data(data_in)
{}


template <int dim, int fe_degree>
void
DummyMatrixFree<dim, fe_degree>::eval(
  LinearAlgebra::CUDAWrappers::Vector<double> &dst) const
{
  LinearAlgebra::CUDAWrappers::Vector<double> src(dst);
  DummyOperator<dim, fe_degree>               dummy_operator;
  data.cell_loop(dummy_operator, src, dst);
}

template <int dim, int fe_degree>
void
test()
{
  Triangulation<dim> tria;
  GridGenerator::hyper_cube(tria);
  tria.refine_global(5 - dim);

  FE_Q<dim>       fe(fe_degree);
  DoFHandler<dim> dof(tria);
  dof.distribute_dofs(fe);
  AffineConstraints<double> constraints;
  constraints.close();

  // Computation on the device
  MappingQGeneric<dim>                  mapping(fe_degree);
  CUDAWrappers::MatrixFree<dim, double> mf_data;
  typename CUDAWrappers::MatrixFree<dim, double>::AdditionalData
    additional_data;
  additional_data.mapping_update_flags = update_values | update_gradients |
                                         update_JxW_values |
                                         update_quadrature_points;
  const QGauss<1> quad(fe_degree + 1);
  mf_data.reinit(mapping, dof, constraints, quad, additional_data);
  constexpr unsigned int n_q_points_per_cell =
    dealii::Utilities::pow(fe_degree + 1, dim);

  const unsigned int              n_dofs = dof.n_dofs();
  DummyMatrixFree<dim, fe_degree> mf(mf_data,
                                     tria.n_active_cells() *
                                       n_q_points_per_cell);
  const unsigned int size = tria.n_active_cells() * n_q_points_per_cell;
  LinearAlgebra::ReadWriteVector<double>      coef(size);
  LinearAlgebra::CUDAWrappers::Vector<double> coef_device(size);

  mf.eval(coef_device);
  hipDeviceSynchronize();
  coef.import(coef_device, VectorOperation::insert);

  // Computation the host
  auto               graph    = mf_data.get_colored_graph();
  unsigned int const n_colors = graph.size();
  for (unsigned int color = 0; color < n_colors; ++color)
    {
      typename CUDAWrappers::MatrixFree<dim, double>::Data gpu_data =
        mf_data.get_data(color);
      unsigned int const n_cells = gpu_data.n_cells;
      auto gpu_data_host = CUDAWrappers::copy_mf_data_to_host<dim, double>(
        gpu_data, additional_data.mapping_update_flags);
      for (unsigned int cell_id = 0; cell_id < n_cells; ++cell_id)
        {
          for (unsigned int i = 0; i < n_q_points_per_cell; ++i)
            {
              unsigned int const pos =
                CUDAWrappers::local_q_point_id_host<dim, double>(
                  cell_id, gpu_data_host, n_q_points_per_cell, i);
              auto p = CUDAWrappers::get_quadrature_point_host<dim, double>(
                cell_id, gpu_data_host, i);
              const double p_val = dim == 2 ? p(0) + p(1) : p(0) + p(1) + p(2);
              AssertThrow(std::abs(coef[pos] - p_val) < 1e-12,
                          ExcInternalError());
            }
        }
    }
}

int
main()
{
  initlog();
  init_cuda();

  test<2, 3>();
  test<3, 3>();

  deallog << "OK" << std::endl;
  return 0;
}

#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2011 - 2017 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------


// build a vector whose elements exceed the size of unsigned int in case of 64
// bit indices. To avoid excessive memory consumption, let the vector start at
// a number close to the maximum of unsigned int but extend past the last
// index

#include <deal.II/base/index_set.h>
#include <deal.II/base/utilities.h>

#include <deal.II/lac/la_parallel_vector.h>

#include <iostream>
#include <vector>

#include "../tests.h"


__global__ void
set_value(double *values_dev, unsigned int index, double val)
{
  values_dev[index] = val;
}

void
test()
{
  unsigned int myid    = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  unsigned int numproc = Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD);

  if (myid == 0)
    deallog << "numproc=" << numproc << std::endl;

  types::global_dof_index min_index  = 0xffffffffU - 39;
  types::global_dof_index local_size = 42;
  IndexSet                local_owned(min_index + numproc * local_size);
  local_owned.add_range(min_index + myid * local_size,
                        min_index + (myid + 1) * local_size);

  // all processors ghost some entries around invalid_unsigned_int and on the
  // border between two processors
  IndexSet local_relevant(local_owned.size());
  local_relevant = local_owned;
  local_relevant.add_range(min_index + 38, min_index + 40);
  local_relevant.add_range(min_index + 41, min_index + 43);

  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> v(
    local_owned, local_relevant, MPI_COMM_WORLD);

  deallog << "Local range of proc 0: " << v.local_range().first << " "
          << v.local_range().second << std::endl;

  // set local values
  for (types::global_dof_index i = 0; i < local_size; ++i)
    {
      double *values_dev = v.get_values();
      set_value<<<1, 1>>>(values_dev, i, min_index + myid * local_size + i);
    }

  deallog << "vector norm: " << v.l2_norm() << std::endl;

  // check ghost values
  v.print(deallog.get_file_stream(), 12, false, false);
  v.update_ghost_values();
  v.print(deallog.get_file_stream(), 12, false, false);

  v.zero_out_ghost_values();
  double *    values_dev  = v.get_values();
  const auto &partitioner = v.get_partitioner();
  set_value<<<1, 1>>>(values_dev,
                      partitioner->global_to_local(min_index + 38),
                      min_index);
  set_value<<<1, 1>>>(values_dev,
                      partitioner->global_to_local(min_index + 39),
                      min_index * 2);
  set_value<<<1, 1>>>(values_dev,
                      partitioner->global_to_local(min_index + 41),
                      min_index + 7);
  set_value<<<1, 1>>>(values_dev,
                      partitioner->global_to_local(min_index + 42),
                      -static_cast<double>(min_index));
  v.compress(VectorOperation::add);
  v.update_ghost_values();
  v.print(deallog.get_file_stream(), 12, false, false);

  deallog << "OK" << std::endl;
}



int
main(int argc, char **argv)
{
  Utilities::MPI::MPI_InitFinalize mpi_initialization(
    argc, argv, testing_max_num_threads());

  MPILogInitAll mpilog;

  init_cuda(true);

  deallog << std::setprecision(12);

  test();
}

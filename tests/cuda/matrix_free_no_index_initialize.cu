#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2019 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------


// check that CUDAWrappers::FEEvaluation::submit_dof_value/get_dof_value
// works correctly.

#include <deal.II/grid/grid_generator.h>
#include <deal.II/grid/manifold_lib.h>

#include <deal.II/matrix_free/cuda_fe_evaluation.h>

#include "../tests.h"

template <int dim,
          int fe_degree,
          int n_q_points_1d = fe_degree + 1,
          typename Number   = double>
class MatrixFreeTest
{
public:
  static const unsigned int n_dofs_1d    = fe_degree + 1;
  static const unsigned int n_local_dofs = Utilities::pow(n_dofs_1d, dim);
  static const unsigned int n_q_points   = Utilities::pow(n_q_points_1d, dim);

  MatrixFreeTest(const CUDAWrappers::MatrixFree<dim, Number> &data_in)
    : data(data_in){};

  __device__ void
  operator()(
    const unsigned int                                          cell,
    const typename CUDAWrappers::MatrixFree<dim, Number>::Data *gpu_data,
    CUDAWrappers::SharedData<dim, Number> *                     shared_data,
    const Number *                                              src,
    Number *                                                    dst) const
  {
    CUDAWrappers::FEEvaluation<dim, fe_degree, n_q_points_1d, 1, Number>
      fe_eval(cell, gpu_data, shared_data);

    // set to unit vector
    fe_eval.submit_dof_value(1.);
    __syncthreads();
    fe_eval.evaluate(/*evaluate_values =*/true, /*evaluate_gradients=*/true);

#ifndef __APPLE__
    // values should evaluate to one, derivatives to zero
    assert(fe_eval.get_value() == 1.);
    for (unsigned int e = 0; e < dim; ++e)
      assert(fe_eval.get_gradient()[e] == 0.);

    fe_eval.integrate(/*integrate_values = */ true,
                      /*integrate_gradients=*/true);
    assert(fe_eval.get_dof_value() == 1.);
#endif
  }



  void
  test() const
  {
    LinearAlgebra::distributed::Vector<Number, MemorySpace::CUDA> dst_dummy;
    LinearAlgebra::distributed::Vector<Number, MemorySpace::CUDA> src_dummy;

    data.cell_loop(*this, src_dummy, dst_dummy);

    // Check that the kernel was launched correctly
    AssertCuda(hipPeekAtLastError());
    // Check that there was no problem during the execution of the kernel
    AssertCuda(hipDeviceSynchronize());

    deallog << "OK" << std::endl;
  };

protected:
  const CUDAWrappers::MatrixFree<dim, Number> &data;
};



template <int dim, int fe_degree, typename number>
void
do_test(const DoFHandler<dim> &          dof,
        const AffineConstraints<double> &constraints)
{
  CUDAWrappers::MatrixFree<dim, number> mf_data;
  {
    const QGauss<1> quad(fe_degree + 1);
    typename CUDAWrappers::MatrixFree<dim, number>::AdditionalData data;
    data.mapping_update_flags = update_values | update_gradients |
                                update_JxW_values | update_quadrature_points;
    mf_data.reinit(dof, constraints, quad, data);
  }

  deallog << "Testing " << dof.get_fe().get_name() << std::endl;
  MatrixFreeTest<dim, fe_degree, fe_degree + 1, number> mf(mf_data);
  mf.test();
}


template <int dim, int fe_degree>
void
test()
{
  const SphericalManifold<dim> manifold;
  Triangulation<dim>           tria;
  GridGenerator::hyper_ball(tria);
  for (const auto &cell : tria.active_cell_iterators())
    for (const unsigned int f : GeometryInfo<dim>::face_indices())
      if (cell->at_boundary(f))
        cell->face(f)->set_all_manifold_ids(0);
  tria.set_manifold(0, manifold);

  // refine first and last cell
  tria.begin(tria.n_levels() - 1)->set_refine_flag();
  tria.last()->set_refine_flag();
  tria.execute_coarsening_and_refinement();
  tria.refine_global(4 - dim);

  FE_Q<dim>       fe(fe_degree);
  DoFHandler<dim> dof(tria);
  dof.distribute_dofs(fe);

  AffineConstraints<double> constraints;
  DoFTools::make_hanging_node_constraints(dof, constraints);
  constraints.close();

  do_test<dim, fe_degree, double>(dof, constraints);
}



int
main()
{
  initlog();

  init_cuda();

  test<2, 1>();
}

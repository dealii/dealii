// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------


// check operator= when we do some operations with ghosts

#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/index_set.h>
#include <deal.II/base/utilities.h>

#include <deal.II/lac/la_parallel_vector.h>
#include <deal.II/lac/read_write_vector.h>

#include <iostream>
#include <vector>

#include "../tests.h"


void
test()
{
  unsigned int myid    = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  unsigned int numproc = Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD);

  if (myid == 0)
    deallog << "numproc=" << numproc << std::endl;


  // each processor owns 2 indices and all
  // are ghosting element 1 (the second)
  IndexSet local_owned(numproc * 2);
  local_owned.add_range(myid * 2, myid * 2 + 2);
  IndexSet local_relevant(numproc * 2);
  local_relevant = local_owned;
  local_relevant.add_range(1, 2);

  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> v(
    local_owned, local_relevant, MPI_COMM_WORLD);
  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> w(v);

  // set local values and check them
  LinearAlgebra::ReadWriteVector<double> rw_vector(local_owned);
  rw_vector(myid * 2)     = myid * 2.0;
  rw_vector(myid * 2 + 1) = myid * 2.0 + 1.0;
  v.import(rw_vector, VectorOperation::insert);

  v.update_ghost_values();

  // check that the value of the ghost is 1.0
  IndexSet ghost_set(numproc * 2);
  ghost_set.add_index(1);
  LinearAlgebra::ReadWriteVector<double> ghost_vector(ghost_set);
  ghost_vector.import(v, VectorOperation::insert);
  DEAL_II_AssertThrow(ghost_vector(1) == 1., ExcInternalError());

  // copy vector
  w = v;
  v *= 2.0;

  v.update_ghost_values();
  w.update_ghost_values();
  ghost_vector.import(v, VectorOperation::insert);
  DEAL_II_AssertThrow(ghost_vector(1) == 2., ExcInternalError());
  ghost_vector.import(w, VectorOperation::insert);
  DEAL_II_AssertThrow(ghost_vector(1) == 1., ExcInternalError());

  if (myid == 0)
    deallog << "OK" << std::endl;
}



int
main(int argc, char **argv)
{
  Utilities::MPI::MPI_InitFinalize mpi_initialization(
    argc, argv, testing_max_num_threads());

  unsigned int myid = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  deallog.push(Utilities::int_to_string(myid));

  Utilities::CUDA::Handle cuda_handle;
  // By default, all the ranks will try to access the device 0. This is fine if
  // we have one rank per node _and_ one gpu per node. If we have multiple GPUs
  // on one node, we need each process to access a different GPU. We assume that
  // each node has the same number of GPUs.
  int         n_devices       = 0;
  hipError_t cuda_error_code = hipGetDeviceCount(&n_devices);
  DEAL_II_AssertCuda(cuda_error_code);
  int device_id   = myid % n_devices;
  cuda_error_code = hipSetDevice(device_id);
  DEAL_II_AssertCuda(cuda_error_code);

  if (myid == 0)
    {
      initlog();
      deallog << std::setprecision(4);

      test();
    }
  else
    test();
}

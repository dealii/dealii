// ---------------------------------------------------------------------
//
// Copyright (C) 2015 - 2016 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the deal.II distribution.
//
// ---------------------------------------------------------------------


// Check LinearAlgebra::CUDAWrappers::Vector assignment and import

#include "../tests.h"
#include <deal.II/base/utilities.h>
#include <deal.II/lac/cuda_vector.h>
#include <deal.II/lac/read_write_vector.h>
#include <fstream>
#include <iostream>

void
test()
{
  const unsigned int size = 100;
  LinearAlgebra::CUDAWrappers::Vector<double> a;
  LinearAlgebra::CUDAWrappers::Vector<double> b(size);
  LinearAlgebra::CUDAWrappers::Vector<double> c(b);
  LinearAlgebra::CUDAWrappers::Vector<double> d;
  d.reinit(c);

  AssertThrow(a.size()==0, ExcMessage("Vector has the wrong size."));
  AssertThrow(b.size()==size, ExcMessage("Vector has the wrong size."));
  AssertThrow(c.size()==size, ExcMessage("Vector has the wrong size."));
  AssertThrow(d.size()==size, ExcMessage("Vector has the wrong size."));

  a.reinit(size);
  AssertThrow(a.size()==size, ExcMessage("Vector has the wrong size."));


  LinearAlgebra::ReadWriteVector<double> read_write_1(size);
  LinearAlgebra::ReadWriteVector<double> read_write_2(size);
  LinearAlgebra::ReadWriteVector<double> read_write_3(size);
  for (unsigned int i=0; i<size; ++i)
    {
      read_write_1[i] = i;
      read_write_2[i] = 5.+i;
    }

  a.import(read_write_2, VectorOperation::insert);
  b.import(read_write_1, VectorOperation::insert);
  c.import(read_write_2, VectorOperation::insert);


  read_write_3.import(a, VectorOperation::insert);
  for (unsigned int i=0; i<size; ++i)
    AssertThrow(read_write_2[i] == read_write_3[i],
                ExcMessage("Vector a has been modified."));

  read_write_3.import(b, VectorOperation::insert);
  for (unsigned int i=0; i<size; ++i)
    AssertThrow(read_write_1[i] == read_write_3[i],
                ExcMessage("Vector b has been modified."));

  read_write_3.import(c, VectorOperation::insert);
  for (unsigned int i=0; i<size; ++i)
    AssertThrow(read_write_2[i] == read_write_3[i],
                ExcMessage("Vector c has been modified."));

  a *= 2.;
  read_write_3.import(a, VectorOperation::insert);
  for (unsigned int i=0; i<size; ++i)
    AssertThrow(2.*read_write_2[i]==read_write_3[i],
                ExcMessage("Problem in operator *=."));

  c /= 2.;
  read_write_3.import(c, VectorOperation::insert);
  for (unsigned int i=0; i<size; ++i)
    AssertThrow(0.5*read_write_2[i]==read_write_3[i],
                ExcMessage("Problem in operator /=."));

  b += a;
  read_write_3.import(b, VectorOperation::insert);
  for (unsigned int i=0; i<size; ++i)
    AssertThrow(2.*read_write_2[i]+read_write_1[i]==read_write_3[i],
                ExcMessage("Problem in operator +=."));

  b -= c;
  read_write_3.import(b, VectorOperation::insert);
  for (unsigned int i=0; i<size; ++i)
    AssertThrow(1.5*read_write_2[i]+read_write_1[i]==read_write_3[i],
                ExcMessage("Problem in operator -=."));

  b.import(read_write_1, VectorOperation::insert);
  c.import(read_write_1, VectorOperation::insert);
  const double val = b*c;
  AssertThrow(val==328350., ExcMessage("Problem in operator *."));

  b = 0.;
  read_write_3.import(b, VectorOperation::insert);
  for (unsigned int i=0; i<size; ++i)
    AssertThrow(read_write_3[i] == 0.,ExcMessage("Problem in operator =."));
}

int
main(int argc, char **argv)
{
  initlog();
  deallog.depth_console(0);

  test();

  deallog << "OK" <<std::endl;

  return 0;
}

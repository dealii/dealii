// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------


// check parallel_vector::copy_from to update ghost values. Same vector layout
// as in parallel_vector_07.cc

#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/index_set.h>
#include <deal.II/base/utilities.h>

#include <deal.II/lac/la_parallel_vector.h>
#include <deal.II/lac/read_write_vector.h>

#include <iostream>
#include <vector>

#include "../tests.h"


void
test()
{
  unsigned int myid    = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  unsigned int numproc = Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD);

  if (myid == 0)
    deallog << "numproc=" << numproc << std::endl;

  const unsigned int set = 200;
  DEAL_II_AssertIndexRange(numproc, set - 2);
  const unsigned int local_size  = set - myid;
  unsigned int       global_size = 0;
  unsigned int       my_start    = 0;
  for (unsigned int i = 0; i < numproc; ++i)
    {
      global_size += set - i;
      if (i < myid)
        my_start += set - i;
    }
  // each processor owns some indices and all
  // are ghosting elements from three
  // processors (the second). some entries
  // are right around the border between two
  // processors
  IndexSet local_owned(global_size);
  local_owned.add_range(my_start, my_start + local_size);
  IndexSet local_relevant(global_size);
  local_relevant                 = local_owned;
  unsigned int ghost_indices[10] = {
    1, 2, 13, set - 3, set - 2, set - 1, set, set + 1, set + 2, set + 3};
  local_relevant.add_indices(&ghost_indices[0], &ghost_indices[0] + 10);

  // v has ghosts, w has none. set some entries
  // on w, copy into v and check if they are
  // there
  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> v(
    local_owned, local_relevant, MPI_COMM_WORLD);
  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> w(
    local_owned, local_owned, MPI_COMM_WORLD);

  // set a few of the local elements
  LinearAlgebra::ReadWriteVector<double> rw_vector(local_owned);
  for (unsigned i = 0; i < local_size; ++i)
    rw_vector.local_element(i) = 2.0 * (i + my_start);
  w.import(rw_vector, VectorOperation::insert);

  v = w;
  v.update_ghost_values();

  // check local values for correctness
  rw_vector.import(v, VectorOperation::insert);
  for (unsigned int i = 0; i < local_size; ++i)
    DEAL_II_AssertThrow(rw_vector.local_element(i) == 2.0 * (i + my_start),
                        ExcInternalError());

  // check non-local entries on all processors
  LinearAlgebra::ReadWriteVector<double> ghost_vector(local_relevant);
  ghost_vector.import(v, VectorOperation::insert);
  for (unsigned int i = 0; i < 10; ++i)
    DEAL_II_AssertThrow(ghost_vector(ghost_indices[i]) == 2. * ghost_indices[i],
                        ExcInternalError());

  // now the same again, but import ghosts automatically because v had ghosts
  // set before calling operator =
  v.reinit(local_owned, local_relevant, MPI_COMM_WORLD);
  v.update_ghost_values();
  v = w;

  // check local values for correctness
  rw_vector.import(v, VectorOperation::insert);
  for (unsigned int i = 0; i < local_size; ++i)
    DEAL_II_AssertThrow(rw_vector.local_element(i) == 2.0 * (i + my_start),
                        ExcInternalError());

  // check non-local entries on all processors
  ghost_vector.import(v, VectorOperation::insert);
  for (unsigned int i = 0; i < 10; ++i)
    DEAL_II_AssertThrow(ghost_vector(ghost_indices[i]) == 2. * ghost_indices[i],
                        ExcInternalError());

  if (myid == 0)
    deallog << "OK" << std::endl;
}



int
main(int argc, char **argv)
{
  Utilities::MPI::MPI_InitFinalize mpi_initialization(
    argc, argv, testing_max_num_threads());

  unsigned int myid = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  deallog.push(Utilities::int_to_string(myid));

  Utilities::CUDA::Handle cuda_handle;
  // By default, all the ranks will try to access the device 0. This is fine if
  // we have one rank per node _and_ one gpu per node. If we have multiple GPUs
  // on one node, we need each process to access a different GPU. We assume that
  // each node has the same number of GPUs.
  int         n_devices       = 0;
  hipError_t cuda_error_code = hipGetDeviceCount(&n_devices);
  DEAL_II_AssertCuda(cuda_error_code);
  int device_id   = myid % n_devices;
  cuda_error_code = hipSetDevice(device_id);
  DEAL_II_AssertCuda(cuda_error_code);

  if (myid == 0)
    {
      initlog();
      deallog << std::setprecision(4);

      test();
    }
  else
    test();
}

#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2019 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------



// check AffineConstraints<double>::set_zero(Vector) for parallel distributed
// vectors

#include <deal.II/base/cuda_size.h>

#include <deal.II/dofs/dof_tools.h>

#include <deal.II/grid/grid_generator.h>

#include <deal.II/lac/la_parallel_vector.h>

#include "../tests.h"


__global__ void
initialize_vector(double *vector, int local_size, int offset)
{
  const int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < local_size)
    vector[index] = 1.0 + index + offset;
}


void
test()
{
  unsigned int myid    = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  unsigned int numproc = Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD);

  IndexSet local_active;
  local_active.set_size(2 * numproc);
  local_active.add_range(myid * numproc, (myid + 1) * numproc);

  AffineConstraints<double> cm;
  cm.add_line(1);
  cm.add_line(2);
  cm.close();

  deallog << "CM:" << std::endl;
  cm.print(deallog.get_file_stream());

  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> ghosted;
  {
    ghosted.reinit(local_active,
                   complete_index_set(2 * numproc),
                   MPI_COMM_WORLD);

    const int n_blocks = 1 + ghosted.size() / CUDAWrappers::block_size;
    initialize_vector<<<n_blocks, CUDAWrappers::block_size>>>(
      ghosted.get_values(), numproc, myid * numproc);
    ghosted.compress(VectorOperation::insert);

    deallog << "ghosted vector before:" << std::endl;
    ghosted.print(deallog.get_file_stream());

    cm.set_zero(ghosted);

    deallog << "ghosted vector after:" << std::endl;
    ghosted.print(deallog.get_file_stream());
  }

  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> distributed;
  {
    distributed.reinit(local_active,
                       complete_index_set(2 * numproc),
                       MPI_COMM_WORLD);

    const int n_blocks = 1 + distributed.size() / CUDAWrappers::block_size;
    initialize_vector<<<n_blocks, CUDAWrappers::block_size>>>(
      distributed.get_values(), numproc, myid * numproc);
    distributed.compress(VectorOperation::insert);

    deallog << "distributed vector before:" << std::endl;
    distributed.print(deallog.get_file_stream());

    cm.set_zero(distributed);

    deallog << "distributed vector after:" << std::endl;
    distributed.print(deallog.get_file_stream());
  }

  deallog << "OK" << std::endl;
}


int
main(int argc, char *argv[])
{
  Utilities::MPI::MPI_InitFinalize mpi_initialization(argc, argv, 1);
  MPILogInitAll                    log;

  init_cuda();

  test();
  return 0;
}

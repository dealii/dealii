#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Check that dealii::SolverRelaxation works with CUDAWrappers::SparseMatrix

#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/exceptions.h>

#include <deal.II/lac/cuda_sparse_matrix.h>
#include <deal.II/lac/precondition.h>
#include <deal.II/lac/read_write_vector.h>
#include <deal.II/lac/solver_control.h>
#include <deal.II/lac/solver_relaxation.h>
#include <deal.II/lac/vector.h>

#include "../tests.h"

#include "../testmatrix.h"


template <typename MatrixType>
class RelaxationOperator
{
public:
  RelaxationOperator(const MatrixType &system_matrix_,
                     const MatrixType &inverse_diagonal_matrix_)
    : system_matrix(system_matrix_)
    , inverse_diagonal_matrix(inverse_diagonal_matrix_)
  {}

  template <typename VectorType>
  void
  step(VectorType &u, const VectorType &v) const
  {
    // u = u - omega*inverse_diagonal_matrix*(system_matrix*u-v)
    const double omega = 1.;
    VectorType   tmp_1(v.size());
    system_matrix.vmult(tmp_1, u);
    tmp_1 -= v;
    VectorType tmp_2(u.size());
    inverse_diagonal_matrix.vmult(tmp_2, tmp_1);
    tmp_2 *= omega;
    u -= tmp_2;
  }

  template <typename VectorType>
  void
  Tstep(VectorType &u, const VectorType &v) const
  {
    AssertThrow(false, ExcNotImplemented());
  }

private:
  const MatrixType &system_matrix;
  const MatrixType &inverse_diagonal_matrix;
};


void
test(Utilities::CUDA::Handle &cuda_handle)
{
  // Build the sparse matrix on the host
  const unsigned int   problem_size = 10;
  unsigned int         size         = (problem_size - 1) * (problem_size - 1);
  FDMatrix             testproblem(problem_size, problem_size);
  SparsityPattern      structure(size, size, 5);
  SparseMatrix<double> A;
  testproblem.five_point_structure(structure);
  structure.compress();
  A.reinit(structure);
  testproblem.five_point(A);
  SparseMatrix<double> A_diagonal_inverse;
  A_diagonal_inverse.reinit(structure);
  for (unsigned int i = 0; i < size; ++i)
    A_diagonal_inverse(i, i) = 1. / A(i, i);

  // Solve on the host
  RelaxationOperator<SparseMatrix<double>> relaxation_operator(
    A, A_diagonal_inverse);
  SolverControl      control(1000, 1.e-3);
  SolverRelaxation<> relaxation_host(control);
  Vector<double>     sol_host(size);
  Vector<double>     rhs_host(size);
  for (unsigned int i = 0; i < size; ++i)
    rhs_host[i] = static_cast<double>(i);
  relaxation_host.solve(A, sol_host, rhs_host, relaxation_operator);

  // Solve on the device
  CUDAWrappers::SparseMatrix<double> A_dev(cuda_handle, A);
  CUDAWrappers::SparseMatrix<double> A_diagonal_inverse_dev(cuda_handle,
                                                            A_diagonal_inverse);
  RelaxationOperator<CUDAWrappers::SparseMatrix<double>>
                                              relaxation_operator_dev(A_dev, A_diagonal_inverse_dev);
  LinearAlgebra::CUDAWrappers::Vector<double> sol_dev(size);
  LinearAlgebra::CUDAWrappers::Vector<double> rhs_dev(size);
  LinearAlgebra::ReadWriteVector<double>      rw_vector(size);
  for (unsigned int i = 0; i < size; ++i)
    rw_vector[i] = static_cast<double>(i);
  rhs_dev.import(rw_vector, VectorOperation::insert);
  SolverRelaxation<LinearAlgebra::CUDAWrappers::Vector<double>> relaxation_dev(
    control);
  relaxation_dev.solve(A_dev, sol_dev, rhs_dev, relaxation_operator_dev);

  // Check the result
  rw_vector.import(sol_dev, VectorOperation::insert);
  for (unsigned int i = 0; i < size; ++i)
    AssertThrow(std::fabs(rw_vector[i] - sol_host[i]) < 1e-8,
                ExcInternalError());
}

int
main()
{
  initlog();
  deallog.depth_console(10);

  init_cuda();

  Utilities::CUDA::Handle cuda_handle;
  test(cuda_handle);

  deallog << "OK" << std::endl;

  return 0;
}

#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Check multiplications and norms

#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/exceptions.h>

#include <deal.II/lac/cuda_sparse_matrix.h>
#include <deal.II/lac/read_write_vector.h>
#include <deal.II/lac/vector.h>

#include "../tests.h"

#include "../testmatrix.h"


void
check_matrix(SparseMatrix<double> const &        A,
             CUDAWrappers::SparseMatrix<double> &A_dev)
{
  hipError_t cuda_error_code;
  double *    val_dev          = nullptr;
  int *       column_index_dev = nullptr;
  int *       row_ptr_dev      = nullptr;
  std::tie(val_dev, column_index_dev, row_ptr_dev, std::ignore, std::ignore) =
    A_dev.get_cusparse_matrix();

  int                 nnz = A_dev.n_nonzero_elements();
  std::vector<double> val_host(nnz);
  cuda_error_code = hipMemcpy(&val_host[0],
                               val_dev,
                               nnz * sizeof(double),
                               hipMemcpyDeviceToHost);
  AssertCuda(cuda_error_code);

  std::vector<int> column_index_host(nnz);
  cuda_error_code = hipMemcpy(&column_index_host[0],
                               column_index_dev,
                               nnz * sizeof(int),
                               hipMemcpyDeviceToHost);
  AssertCuda(cuda_error_code);

  int const        n_rows = A_dev.m() + 1;
  std::vector<int> row_ptr_host(n_rows + 1);
  cuda_error_code = hipMemcpy(&row_ptr_host[0],
                               row_ptr_dev,
                               (A_dev.m() + 1) * sizeof(int),
                               hipMemcpyDeviceToHost);
  AssertCuda(cuda_error_code);

  for (int i = 0; i < n_rows; ++i)
    for (int j = row_ptr_host[i]; j < row_ptr_host[i + 1]; ++j)
      AssertThrow(std::abs(val_host[j] - A(i, column_index_host[j])) < 1e-15,
                  ExcInternalError());
}

void
check_vector(Vector<double> const &                        a,
             LinearAlgebra::ReadWriteVector<double> const &b)
{
  unsigned int size = a.size();
  for (unsigned int i = 0; i < size; ++i)
    AssertThrow(std::abs(a[i] - b[i]) < 1e-15, ExcInternalError());
}

void
test(Utilities::CUDA::Handle &cuda_handle)
{
  // Build the sparse matrix on the host
  const unsigned int   size = 10;
  unsigned int         dim  = (size - 1) * (size - 1);
  FDMatrix             testproblem(size, size);
  SparsityPattern      structure(dim, dim, 5);
  SparseMatrix<double> A;
  testproblem.five_point_structure(structure);
  structure.compress();
  A.reinit(structure);
  testproblem.five_point(A, true);

  // Create the sparse matrix on the device
  CUDAWrappers::SparseMatrix<double> A_dev(cuda_handle, A);
  check_matrix(A, A_dev);

  AssertDimension(A.m(), A_dev.m());
  AssertDimension(A.n(), A_dev.n());

  // Multiply by a constant
  A *= 2.;
  A_dev *= 2.;
  check_matrix(A, A_dev);

  // Divide by a constant
  A /= 2.;
  A_dev /= 2.;
  check_matrix(A, A_dev);

  // Matrix-vector multiplication
  const unsigned int vector_size = A.n();
  Vector<double>     dst(vector_size);
  Vector<double>     src(vector_size);
  for (unsigned int i = 0; i < vector_size; ++i)
    src[i] = i;
  A.vmult(dst, src);
  LinearAlgebra::CUDAWrappers::Vector<double> dst_dev(vector_size);
  LinearAlgebra::CUDAWrappers::Vector<double> src_dev(vector_size);
  LinearAlgebra::ReadWriteVector<double>      read_write(vector_size);
  for (unsigned int i = 0; i < vector_size; ++i)
    read_write[i] = i;
  src_dev.import(read_write, VectorOperation::insert);
  A_dev.vmult(dst_dev, src_dev);
  read_write.import(dst_dev, VectorOperation::insert);
  check_vector(dst, read_write);

  // Transpose matrix-vector multiplication
  A.Tvmult(dst, src);
  A_dev.Tvmult(dst_dev, src_dev);
  read_write.import(dst_dev, VectorOperation::insert);
  check_vector(dst, read_write);

  // Matrix-vector multiplication and add
  A.vmult_add(dst, src);
  A_dev.vmult_add(dst_dev, src_dev);
  read_write.import(dst_dev, VectorOperation::insert);
  check_vector(dst, read_write);

  // Transpose matrix-vector multiplication and add
  A.Tvmult_add(dst, src);
  A_dev.Tvmult_add(dst_dev, src_dev);
  read_write.import(dst_dev, VectorOperation::insert);
  check_vector(dst, read_write);

  // Matrix norm square
  double value      = A.matrix_norm_square(src);
  double value_host = A_dev.matrix_norm_square(src_dev);
  AssertThrow(std::abs(value - value_host) < 1e-15, ExcInternalError());

  // Matrix scalar product (reuse dst and src but they are both input)
  value      = A.matrix_scalar_product(dst, src);
  value_host = A_dev.matrix_scalar_product(dst_dev, src_dev);
  AssertThrow(std::abs(value - value_host) < 1e-15, ExcInternalError());

  // Compute the residual
  Vector<double> b(src);
  for (unsigned int i = 0; i < vector_size; ++i)
    {
      b[i]          = i;
      src[i]        = i;
      read_write[i] = i;
    }
  LinearAlgebra::CUDAWrappers::Vector<double> b_dev(vector_size);
  b_dev.import(read_write, VectorOperation::insert);
  src_dev.import(read_write, VectorOperation::insert);
  value      = A.residual(dst, src, b);
  value_host = A_dev.residual(dst_dev, src_dev, b_dev);
  AssertThrow(std::abs(value - value_host) < 1e-15, ExcInternalError());
  read_write.import(dst_dev, VectorOperation::insert);
  check_vector(dst, read_write);

  // Compute L1 norm
  value      = A.l1_norm();
  value_host = A_dev.l1_norm();
  AssertThrow(std::abs(value - value_host) < 1e-15, ExcInternalError());

  // Compute Linfty norm
  value      = A.linfty_norm();
  value_host = A_dev.linfty_norm();
  AssertThrow(std::abs(value - value_host) < 1e-15, ExcInternalError());

  // Compute Frobenius norm
  value      = A.frobenius_norm();
  value_host = A_dev.frobenius_norm();
  AssertThrow(std::abs(value - value_host) < 1e-15, ExcInternalError());

  // Compute L1 norm second test
  SparsityPattern sparsity_pattern(vector_size, vector_size, 3);
  for (unsigned int i = 0; i < vector_size; ++i)
    {
      sparsity_pattern.add(i, 0);
      sparsity_pattern.add(i, i);
      if (i < vector_size - 1)
        sparsity_pattern.add(i, i + 1);
    }
  sparsity_pattern.compress();
  SparseMatrix<double> B(sparsity_pattern);
  for (unsigned int i = 0; i < vector_size; ++i)
    {
      B.set(i, 0, 1);
      B.set(i, i, 1);
      if (i < vector_size - 1)
        B.set(i, i + 1, 1);
    }
  CUDAWrappers::SparseMatrix<double> B_dev(cuda_handle, B);
  value      = B.l1_norm();
  value_host = B_dev.l1_norm();
  AssertThrow(std::abs(value - value_host) < 1e-15, ExcInternalError());
}

int
main()
{
  initlog();
  deallog.depth_console(0);

  init_cuda();

  Utilities::CUDA::Handle cuda_handle;

  test(cuda_handle);

  deallog << "OK" << std::endl;

  return 0;
}

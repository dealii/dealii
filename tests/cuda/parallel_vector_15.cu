#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------


// check that handling of ghost elements in parallel distributed vectors works
// appropriately when creating a vector from a non-ghosted source vector using
// the assignment operator

#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/index_set.h>
#include <deal.II/base/utilities.h>

#include <deal.II/lac/la_parallel_vector.h>
#include <deal.II/lac/read_write_vector.h>

#include <iostream>
#include <vector>

#include "../tests.h"


void
test()
{
  unsigned int myid    = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  unsigned int numproc = Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD);

  if (myid == 0)
    deallog << "numproc=" << numproc << std::endl;

  // processor 0 and 1 own 2 indices each, higher processors nothing, all are
  // ghosting global elements 1 and 3
  IndexSet local_owned(std::min(numproc * 2, 4U));
  if (myid < 2)
    local_owned.add_range(myid * 2, myid * 2 + 2);
  IndexSet local_relevant(local_owned.size());
  local_relevant = local_owned;
  local_relevant.add_range(1, 2);
  if (numproc > 1)
    local_relevant.add_range(3, 4);

  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> v(
    local_owned, local_relevant, MPI_COMM_WORLD);

  // set local values
  LinearAlgebra::ReadWriteVector<double> rw_vector(local_owned);
  if (myid < 2)
    {
      rw_vector(myid * 2)     = myid * 2.0;
      rw_vector(myid * 2 + 1) = myid * 2.0 + 1.0;
    }

  v.import(rw_vector, VectorOperation::insert);

  if (myid == 0)
    deallog << "v has ghost elements: " << v.has_ghost_elements() << std::endl;

  LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA> w, x;
  w = v;
  if (myid == 0)
    deallog << "w has ghost elements: " << w.has_ghost_elements() << std::endl;

  v.update_ghost_values();
  w = v;
  if (myid == 0)
    deallog << "w has ghost elements: " << w.has_ghost_elements() << std::endl;

  v.zero_out_ghost_values();
  w = v;
  if (myid == 0)
    deallog << "w has ghost elements: " << w.has_ghost_elements() << std::endl;

  w.zero_out_ghost_values();
  w = v;
  if (myid == 0)
    deallog << "w has ghost elements: " << w.has_ghost_elements() << std::endl;

  v.update_ghost_values();
  x = v;
  if (myid == 0)
    deallog << "x has ghost elements: " << x.has_ghost_elements() << std::endl;

  x.zero_out_ghost_values();
  if (myid == 0)
    deallog << "x has ghost elements: " << x.has_ghost_elements() << std::endl;

  if (myid == 0)
    deallog << "OK" << std::endl;
}



int
main(int argc, char **argv)
{
  Utilities::MPI::MPI_InitFinalize mpi_initialization(
    argc, argv, testing_max_num_threads());

  unsigned int myid = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  deallog.push(Utilities::int_to_string(myid));

  init_cuda(true);

  if (myid == 0)
    {
      initlog();
      deallog << std::setprecision(4);

      test();
    }
  else
    test();
}

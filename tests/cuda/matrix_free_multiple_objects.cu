#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2017 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

// Reproduce a bug where only one matrix-free object is valid

#include <deal.II/base/logstream.h>
#include <deal.II/base/point.h>
#include <deal.II/base/utilities.h>

#include <deal.II/dofs/dof_handler.h>
#include <deal.II/dofs/dof_tools.h>

#include <deal.II/fe/fe_q.h>
#include <deal.II/fe/fe_values.h>

#include <deal.II/grid/grid_generator.h>
#include <deal.II/grid/manifold_lib.h>
#include <deal.II/grid/tria.h>

#include <deal.II/lac/affine_constraints.h>
#include <deal.II/lac/cuda_vector.h>
#include <deal.II/lac/dynamic_sparsity_pattern.h>
#include <deal.II/lac/la_parallel_vector.h>
#include <deal.II/lac/read_write_vector.h>
#include <deal.II/lac/sparse_matrix.h>
#include <deal.II/lac/vector.h>

#include <deal.II/numerics/vector_tools.h>

#include <iostream>

#include "../tests.h"

#include "matrix_vector_mf.h"

template <int fe_degree, int n_q_points_1d>
void
do_test(const DoFHandler<2> &            dof,
        MatrixFreeTest<2,
                       fe_degree,
                       double,
                       LinearAlgebra::CUDAWrappers::Vector<double>,
                       n_q_points_1d> &  mf,
        unsigned int                     n_dofs,
        MappingQGeneric<2> &             mapping,
        const AffineConstraints<double> &constraints)
{
  Vector<double>                              in_host(n_dofs), out_host(n_dofs);
  LinearAlgebra::ReadWriteVector<double>      in(n_dofs), out(n_dofs);
  LinearAlgebra::CUDAWrappers::Vector<double> in_device(n_dofs);
  LinearAlgebra::CUDAWrappers::Vector<double> out_device(n_dofs);

  for (unsigned int i = 0; i < n_dofs; ++i)
    {
      if (constraints.is_constrained(i))
        continue;
      const double entry = Testing::rand() / (double)RAND_MAX;
      in(i)              = entry;
    }

  in_device.import(in, VectorOperation::insert);
  mf.vmult(out_device, in_device);
  hipDeviceSynchronize();
  out.import(out_device, VectorOperation::insert);

  // assemble sparse matrix with (\nabla v, \nabla u) + (v, 10 * u)
  SparsityPattern sparsity;
  {
    DynamicSparsityPattern csp(n_dofs, n_dofs);
    DoFTools::make_sparsity_pattern(dof, csp, constraints, true);
    sparsity.copy_from(csp);
  }
  SparseMatrix<double> sparse_matrix(sparsity);
  {
    QGauss<2> quadrature_formula(n_q_points_1d);

    FEValues<2> fe_values(mapping,
                          dof.get_fe(),
                          quadrature_formula,
                          update_values | update_gradients |
                            update_quadrature_points | update_JxW_values);

    const unsigned int dofs_per_cell = dof.get_fe().dofs_per_cell;
    const unsigned int n_q_points    = quadrature_formula.size();

    FullMatrix<double> cell_matrix(dofs_per_cell, dofs_per_cell);
    std::vector<types::global_dof_index> local_dof_indices(dofs_per_cell);

    typename DoFHandler<2>::active_cell_iterator cell = dof.begin_active(),
                                                 endc = dof.end();
    for (; cell != endc; ++cell)
      {
        cell_matrix = 0;
        fe_values.reinit(cell);

        for (unsigned int q_point = 0; q_point < n_q_points; ++q_point)
          {
            const auto coef = 10.;
            for (unsigned int i = 0; i < dofs_per_cell; ++i)
              {
                for (unsigned int j = 0; j < dofs_per_cell; ++j)
                  cell_matrix(i, j) +=
                    ((fe_values.shape_grad(i, q_point) *
                        fe_values.shape_grad(j, q_point) +
                      coef * fe_values.shape_value(i, q_point) *
                        fe_values.shape_value(j, q_point)) *
                     fe_values.JxW(q_point));
              }
          }

        cell->get_dof_indices(local_dof_indices);
        constraints.distribute_local_to_global(cell_matrix,
                                               local_dof_indices,
                                               sparse_matrix);
      }
  }
  for (unsigned i = 0; i < n_dofs; ++i)
    in_host[i] = in[i];
  sparse_matrix.vmult(out_host, in_host);

  double out_norm = 0.;
  for (unsigned i = 0; i < n_dofs; ++i)
    out_norm += std::pow(out[i] - out_host[i], 2);
  const double diff_norm = std::sqrt(out_norm) / out_host.linfty_norm();

  deallog << "Norm of difference: " << diff_norm << std::endl << std::endl;
}


int
main()
{
  initlog();
  deallog.depth_console(0);

  deallog << std::setprecision(3);

  init_cuda();

  Triangulation<2> tria;
  GridGenerator::hyper_cube(tria);
  tria.refine_global(5 - 2);
  AffineConstraints<double> constraints;
  constraints.close();
  bool constant_coefficient = true;

  // Create the first MatrixFree object
  constexpr unsigned int fe_degree_1     = 1;
  constexpr unsigned int n_q_points_1d_1 = fe_degree_1 + 1;
  FE_Q<2>                fe_1(fe_degree_1);
  DoFHandler<2>          dof_1(tria);
  dof_1.distribute_dofs(fe_1);
  MappingQGeneric<2>                                  mapping_1(fe_degree_1);
  CUDAWrappers::MatrixFree<2, double>                 mf_data_1;
  CUDAWrappers::MatrixFree<2, double>::AdditionalData additional_data_1;
  additional_data_1.mapping_update_flags = update_values | update_gradients |
                                           update_JxW_values |
                                           update_quadrature_points;
  const QGauss<1> quad_1(n_q_points_1d_1);
  mf_data_1.reinit(mapping_1, dof_1, constraints, quad_1, additional_data_1);
  const unsigned int n_dofs_1 = dof_1.n_dofs();
  MatrixFreeTest<2,
                 fe_degree_1,
                 double,
                 LinearAlgebra::CUDAWrappers::Vector<double>,
                 n_q_points_1d_1>
    mf_1(mf_data_1,
         n_dofs_1 * std::pow(n_q_points_1d_1, 2),
         constant_coefficient);

  // Create the second MatrixFree object
  constexpr unsigned int fe_degree_2     = 2;
  constexpr unsigned int n_q_points_1d_2 = fe_degree_2 + 1;
  FE_Q<2>                fe_2(fe_degree_2);
  DoFHandler<2>          dof_2(tria);
  dof_2.distribute_dofs(fe_2);
  MappingQGeneric<2>                                  mapping_2(fe_degree_2);
  CUDAWrappers::MatrixFree<2, double>                 mf_data_2;
  CUDAWrappers::MatrixFree<2, double>::AdditionalData additional_data_2;
  additional_data_2.mapping_update_flags = update_values | update_gradients |
                                           update_JxW_values |
                                           update_quadrature_points;
  const QGauss<1> quad_2(n_q_points_1d_2);
  mf_data_2.reinit(mapping_2, dof_2, constraints, quad_2, additional_data_2);
  const unsigned int n_dofs_2 = dof_2.n_dofs();
  MatrixFreeTest<2,
                 fe_degree_2,
                 double,
                 LinearAlgebra::CUDAWrappers::Vector<double>,
                 n_q_points_1d_2>
    mf_2(mf_data_2,
         n_dofs_2 * std::pow(n_q_points_1d_2, 2),
         constant_coefficient);

  // Perform MV with the first object
  do_test<fe_degree_1, n_q_points_1d_1>(
    dof_1, mf_1, n_dofs_1, mapping_1, constraints);

  // Perform MV with the second object
  do_test<fe_degree_2, n_q_points_1d_2>(
    dof_2, mf_2, n_dofs_2, mapping_2, constraints);

  return 0;
}

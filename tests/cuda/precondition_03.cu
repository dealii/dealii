#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------



// this tests the correctness of MPI-parallel matrix free matrix-vector
// products by comparing the result with a Trilinos sparse matrix assembled in
// the usual way. The mesh is distributed among processors (hypercube) and has
// both hanging nodes (by randomly refining some cells, so the mesh is going
// to be different when run with different numbers of processors) and
// Dirichlet boundary conditions

#include <deal.II/base/function.h>
#include <deal.II/base/utilities.h>

#include <deal.II/distributed/tria.h>

#include <deal.II/dofs/dof_handler.h>
#include <deal.II/dofs/dof_tools.h>

#include <deal.II/fe/fe_q.h>
#include <deal.II/fe/fe_values.h>

#include <deal.II/grid/grid_generator.h>
#include <deal.II/grid/manifold_lib.h>

#include <deal.II/lac/affine_constraints.h>
#include <deal.II/lac/precondition.h>
#include <deal.II/lac/trilinos_sparse_matrix.h>
#include <deal.II/lac/trilinos_sparsity_pattern.h>

#include <deal.II/numerics/vector_tools.h>

#include <iostream>

#include "../tests.h"

#include "matrix_vector_mf.h"



template <int dim, int fe_degree>
void
test()
{
  typedef double Number;

  parallel::distributed::Triangulation<dim> tria(MPI_COMM_WORLD);
  GridGenerator::hyper_cube(tria);
  tria.refine_global(1);
  typename Triangulation<dim>::active_cell_iterator cell = tria.begin_active(),
                                                    endc = tria.end();
  for (; cell != endc; ++cell)
    if (cell->is_locally_owned())
      if (cell->center().norm() < 0.2)
        cell->set_refine_flag();
  tria.execute_coarsening_and_refinement();
  if (dim < 3 && fe_degree < 2)
    tria.refine_global(2);
  else
    tria.refine_global(1);
  if (tria.begin(tria.n_levels() - 1)->is_locally_owned())
    tria.begin(tria.n_levels() - 1)->set_refine_flag();
  if (tria.last()->is_locally_owned())
    tria.last()->set_refine_flag();
  tria.execute_coarsening_and_refinement();
  cell = tria.begin_active();
  for (unsigned int i = 0; i < 10 - 3 * dim; ++i)
    {
      cell                 = tria.begin_active();
      unsigned int counter = 0;
      for (; cell != endc; ++cell, ++counter)
        if (cell->is_locally_owned())
          if (counter % (7 - i) == 0)
            cell->set_refine_flag();
      tria.execute_coarsening_and_refinement();
    }

  FE_Q<dim>       fe(fe_degree);
  DoFHandler<dim> dof(tria);
  dof.distribute_dofs(fe);

  IndexSet owned_set = dof.locally_owned_dofs();
  IndexSet relevant_set;
  DoFTools::extract_locally_relevant_dofs(dof, relevant_set);

  AffineConstraints<double> constraints(relevant_set);
  DoFTools::make_hanging_node_constraints(dof, constraints);
  VectorTools::interpolate_boundary_values(dof,
                                           0,
                                           Functions::ZeroFunction<dim>(),
                                           constraints);
  constraints.close();

  deallog << "Testing " << dof.get_fe().get_name() << std::endl;

  MappingQGeneric<dim>                  mapping(fe_degree);
  CUDAWrappers::MatrixFree<dim, Number> mf_data;
  const QGauss<1>                       quad(fe_degree + 1);
  typename CUDAWrappers::MatrixFree<dim, Number>::AdditionalData
    additional_data;
  additional_data.mapping_update_flags = update_values | update_gradients |
                                         update_JxW_values |
                                         update_quadrature_points;
  mf_data.reinit(mapping, dof, constraints, quad, additional_data);

  const unsigned int coef_size =
    tria.n_locally_owned_active_cells() * std::pow(fe_degree + 1, dim);
  MatrixFreeTest<dim,
                 fe_degree,
                 Number,
                 LinearAlgebra::distributed::Vector<Number, MemorySpace::CUDA>>
    mf(mf_data, coef_size);
  mf.internal_m = owned_set.size();
  LinearAlgebra::distributed::Vector<Number, MemorySpace::CUDA> in_dev(
    owned_set, MPI_COMM_WORLD);
  LinearAlgebra::distributed::Vector<Number, MemorySpace::CUDA> out_dev(
    owned_set, MPI_COMM_WORLD);

  LinearAlgebra::ReadWriteVector<Number> rw_in(owned_set);
  for (unsigned int i = 0; i < in_dev.local_size(); ++i)
    {
      const unsigned int glob_index = owned_set.nth_index_in_set(i);
      if (constraints.is_constrained(glob_index))
        continue;
      rw_in.local_element(i) = random_value<double>();
    }
  in_dev.import(rw_in, VectorOperation::insert);

  // assemble trilinos sparse matrix with
  // (\nabla v, \nabla u) + (v, 10 * u) for
  // reference
  LinearAlgebra::distributed::Vector<Number, MemorySpace::Host> in_host(
    owned_set, MPI_COMM_WORLD);
  in_host.import(rw_in, VectorOperation::insert);
  LinearAlgebra::distributed::Vector<Number, MemorySpace::Host> ref(
    owned_set, MPI_COMM_WORLD);
  TrilinosWrappers::SparseMatrix sparse_matrix;
  {
    TrilinosWrappers::SparsityPattern csp(owned_set, MPI_COMM_WORLD);
    DoFTools::make_sparsity_pattern(dof,
                                    csp,
                                    constraints,
                                    true,
                                    Utilities::MPI::this_mpi_process(
                                      MPI_COMM_WORLD));
    csp.compress();
    sparse_matrix.reinit(csp);
  }
  {
    QGauss<dim> quadrature_formula(fe_degree + 1);

    FEValues<dim> fe_values(dof.get_fe(),
                            quadrature_formula,
                            update_values | update_gradients |
                              update_JxW_values);

    const unsigned int dofs_per_cell = dof.get_fe().dofs_per_cell;
    const unsigned int n_q_points    = quadrature_formula.size();

    FullMatrix<double> cell_matrix(dofs_per_cell, dofs_per_cell);
    std::vector<types::global_dof_index> local_dof_indices(dofs_per_cell);

    typename DoFHandler<dim>::active_cell_iterator cell = dof.begin_active(),
                                                   endc = dof.end();
    for (; cell != endc; ++cell)
      if (cell->is_locally_owned())
        {
          cell_matrix = 0;
          fe_values.reinit(cell);

          for (unsigned int q_point = 0; q_point < n_q_points; ++q_point)
            for (unsigned int i = 0; i < dofs_per_cell; ++i)
              {
                for (unsigned int j = 0; j < dofs_per_cell; ++j)
                  cell_matrix(i, j) +=
                    ((fe_values.shape_grad(i, q_point) *
                        fe_values.shape_grad(j, q_point) +
                      10. * fe_values.shape_value(i, q_point) *
                        fe_values.shape_value(j, q_point)) *
                     fe_values.JxW(q_point));
              }

          cell->get_dof_indices(local_dof_indices);
          constraints.distribute_local_to_global(cell_matrix,
                                                 local_dof_indices,
                                                 sparse_matrix);
        }
  }
  sparse_matrix.compress(VectorOperation::add);

  LinearAlgebra::distributed::Vector<Number, MemorySpace::Host> matrix_diagonal(
    ref.get_partitioner());
  for (const auto index : matrix_diagonal.locally_owned_elements())
    matrix_diagonal[index] = sparse_matrix(index, index);

  using HostPreconditionerType = PreconditionChebyshev<
    TrilinosWrappers::SparseMatrix,
    LinearAlgebra::distributed::Vector<Number, MemorySpace::Host>>;
  HostPreconditionerType                          precondition_chebyshev_host;
  typename HostPreconditionerType::AdditionalData host_preconditioner_data;
  host_preconditioner_data.preconditioner = std::make_shared<DiagonalMatrix<
    LinearAlgebra::distributed::Vector<Number, MemorySpace::Host>>>(
    matrix_diagonal);
  host_preconditioner_data.constraints.copy_from(constraints);
  precondition_chebyshev_host.initialize(sparse_matrix,
                                         host_preconditioner_data);

  using DevicePreconditionerType = PreconditionChebyshev<
    MatrixFreeTest<
      dim,
      fe_degree,
      Number,
      LinearAlgebra::distributed::Vector<Number, MemorySpace::CUDA>>,
    LinearAlgebra::distributed::Vector<Number, MemorySpace::CUDA>>;
  DevicePreconditionerType precondition_chebyshev_device;
  typename DevicePreconditionerType::AdditionalData device_preconditioner_data;
  device_preconditioner_data.preconditioner = std::make_shared<DiagonalMatrix<
    LinearAlgebra::distributed::Vector<Number, MemorySpace::CUDA>>>(
    LinearAlgebra::distributed::Vector<Number, MemorySpace::CUDA>(
      ref.get_partitioner()));
  device_preconditioner_data.preconditioner->get_vector().import(
    matrix_diagonal, VectorOperation::insert);
  device_preconditioner_data.constraints.copy_from(constraints);
  precondition_chebyshev_device.initialize(mf, device_preconditioner_data);

  precondition_chebyshev_device.vmult(out_dev, in_dev);

  LinearAlgebra::distributed::Vector<Number, MemorySpace::Host> out_host(
    owned_set, MPI_COMM_WORLD);
  LinearAlgebra::ReadWriteVector<Number> rw_out(owned_set);
  rw_out.import(out_dev, VectorOperation::insert);
  out_host.import(rw_out, VectorOperation::insert);

  precondition_chebyshev_host.vmult(ref, in_host);
  out_host -= ref;

  const double diff_norm = out_host.linfty_norm();

  deallog << "Norm of difference: " << diff_norm << std::endl << std::endl;
}


int
main(int argc, char **argv)
{
  Utilities::MPI::MPI_InitFinalize mpi_initialization(
    argc, argv, testing_max_num_threads());

  unsigned int myid = Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
  deallog.push(Utilities::int_to_string(myid));

  init_cuda(true);

  if (myid == 0)
    {
      initlog();
      deallog << std::setprecision(4);

      deallog.push("2d");
      test<2, 1>();
      deallog.pop();

      deallog.push("3d");
      test<3, 1>();
      test<3, 2>();
      deallog.pop();
    }
  else
    {
      test<2, 1>();
      test<3, 1>();
      test<3, 2>();
    }
}

// ---------------------------------------------------------------------
//
// Copyright (C) 2016 - 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

#include <deal.II/matrix_free/cuda_matrix_free.templates.h>

#ifdef DEAL_II_WITH_CUDA

DEAL_II_NAMESPACE_OPEN



namespace CUDAWrappers
{
  namespace internal
  {
    std::array<std::atomic_bool, mf_n_concurrent_objects> used_objects;
  }

  // Do not instantiate for dim = 1
  template class MatrixFree<2, float>;
  template class MatrixFree<2, double>;
  template class MatrixFree<3, float>;
  template class MatrixFree<3, double>;
} // namespace CUDAWrappers

DEAL_II_NAMESPACE_CLOSE

#endif

#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2016 - 2020 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/cuda_size.h>
#include <deal.II/base/exceptions.h>

#include <deal.II/lac/cuda_atomic.h>
#include <deal.II/lac/cuda_kernels.h>
#include <deal.II/lac/cuda_vector.h>
#include <deal.II/lac/read_write_vector.h>

#include <cmath>

#ifdef DEAL_II_WITH_CUDA

DEAL_II_NAMESPACE_OPEN

namespace LinearAlgebra
{
  namespace CUDAWrappers
  {
    using ::dealii::CUDAWrappers::block_size;
    using ::dealii::CUDAWrappers::chunk_size;



    template <typename Number>
    Vector<Number>::Vector()
      : val(nullptr, Utilities::CUDA::delete_device_data<Number>)
      , n_elements(0)
    {}



    template <typename Number>
    Vector<Number>::Vector(const Vector<Number> &V)
      : val(Utilities::CUDA::allocate_device_data<Number>(V.n_elements),
            Utilities::CUDA::delete_device_data<Number>)
      , n_elements(V.n_elements)
    {
      // Copy the values.
      const hipError_t error_code = hipMemcpy(val.get(),
                                                V.val.get(),
                                                n_elements * sizeof(Number),
                                                hipMemcpyDeviceToDevice);
      AssertCuda(error_code);
    }



    template <typename Number>
    Vector<Number> &
    Vector<Number>::operator=(const Vector<Number> &V)
    {
      if (n_elements < V.n_elements)
        reinit(V.n_elements, true);
      else
        n_elements = V.n_elements;

      // Copy the values.
      const hipError_t error_code = hipMemcpy(val.get(),
                                                V.val.get(),
                                                n_elements * sizeof(Number),
                                                hipMemcpyDeviceToDevice);
      AssertCuda(error_code);

      return *this;
    }



    template <typename Number>
    Vector<Number>::Vector(const size_type n)
      : val(nullptr, Utilities::CUDA::delete_device_data<Number>)
      , n_elements(0)
    {
      reinit(n, false);
    }



    template <typename Number>
    void
    Vector<Number>::reinit(const size_type n, const bool omit_zeroing_entries)
    {
      // Resize the underlying array if necessary
      if (n == 0)
        val.reset();
      else if (n != n_elements)
        val.reset(Utilities::CUDA::allocate_device_data<Number>(n));

      // If necessary set the elements to zero
      if (omit_zeroing_entries == false)
        {
          const hipError_t error_code =
            hipMemset(val.get(), 0, n * sizeof(Number));
          AssertCuda(error_code);
        }
      n_elements = n;
    }



    template <typename Number>
    void
    Vector<Number>::reinit(const VectorSpaceVector<Number> &V,
                           const bool omit_zeroing_entries)
    {
      reinit(V.size(), omit_zeroing_entries);
    }



    template <typename Number>
    void
    Vector<Number>::import(
      const ReadWriteVector<Number> &V,
      VectorOperation::values        operation,
      std::shared_ptr<const Utilities::MPI::CommunicationPatternBase>)
    {
      if (operation == VectorOperation::insert)
        {
          const hipError_t error_code = hipMemcpy(val.get(),
                                                    V.begin(),
                                                    n_elements * sizeof(Number),
                                                    hipMemcpyHostToDevice);
          AssertCuda(error_code);
        }
      else if (operation == VectorOperation::add)
        {
          // Create a temporary vector on the device
          Number *    tmp;
          hipError_t error_code =
            hipMalloc(&tmp, n_elements * sizeof(Number));
          AssertCuda(error_code);

          // Copy the vector from the host to the temporary vector on the device
          error_code = hipMemcpy(tmp,
                                  V.begin(),
                                  n_elements * sizeof(Number),
                                  hipMemcpyHostToDevice);
          AssertCuda(error_code);

          // Add the two vectors
          const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);

          kernel::vector_bin_op<Number, kernel::Binop_Addition>
            <<<n_blocks, block_size>>>(val.get(), tmp, n_elements);
          AssertCudaKernel();

          // Delete the temporary vector
          Utilities::CUDA::free(tmp);
        }
      else
        AssertThrow(false, ExcNotImplemented());
    }



    template <typename Number>
    Vector<Number> &
    Vector<Number>::operator=(const Number s)
    {
      Assert(s == Number(), ExcMessage("Only 0 can be assigned to a vector."));
      (void)s;

      const hipError_t error_code =
        hipMemset(val.get(), 0, n_elements * sizeof(Number));
      AssertCuda(error_code);

      return *this;
    }



    template <typename Number>
    Vector<Number> &
    Vector<Number>::operator*=(const Number factor)
    {
      AssertIsFinite(factor);
      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);
      kernel::vec_scale<Number>
        <<<n_blocks, block_size>>>(val.get(), factor, n_elements);
      AssertCudaKernel();

      return *this;
    }



    template <typename Number>
    Vector<Number> &
    Vector<Number>::operator/=(const Number factor)
    {
      AssertIsFinite(factor);
      Assert(factor != Number(0.), ExcZero());
      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);
      kernel::vec_scale<Number>
        <<<n_blocks, block_size>>>(val.get(), 1. / factor, n_elements);
      AssertCudaKernel();

      return *this;
    }



    template <typename Number>
    Vector<Number> &
    Vector<Number>::operator+=(const VectorSpaceVector<Number> &V)
    {
      // Check that casting will work
      Assert(dynamic_cast<const Vector<Number> *>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If it fails, it throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number> &>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage(
               "Cannot add two vectors with different numbers of elements"));

      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);

      kernel::vector_bin_op<Number, kernel::Binop_Addition>
        <<<n_blocks, block_size>>>(val.get(), down_V.val.get(), n_elements);
      AssertCudaKernel();

      return *this;
    }



    template <typename Number>
    Vector<Number> &
    Vector<Number>::operator-=(const VectorSpaceVector<Number> &V)
    {
      // Check that casting will work
      Assert(dynamic_cast<const Vector<Number> *>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throws an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number> &>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage(
               "Cannot add two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);

      kernel::vector_bin_op<Number, kernel::Binop_Subtraction>
        <<<n_blocks, block_size>>>(val.get(), down_V.val.get(), n_elements);
      AssertCudaKernel();

      return *this;
    }



    template <typename Number>
    Number Vector<Number>::operator*(const VectorSpaceVector<Number> &V) const
    {
      // Check that casting will work
      Assert(dynamic_cast<const Vector<Number> *>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throws an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number> &>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage(
               "Cannot add two vectors with different numbers of elements"));

      Number *    result_device;
      hipError_t error_code =
        hipMalloc(&result_device, n_elements * sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(result_device, 0, sizeof(Number));

      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);
      kernel::double_vector_reduction<Number, kernel::DotProduct<Number>>
        <<<dim3(n_blocks, 1), dim3(block_size)>>>(result_device,
                                                  val.get(),
                                                  down_V.val.get(),
                                                  static_cast<unsigned int>(
                                                    n_elements));

      // Copy the result back to the host
      Number result;
      error_code = hipMemcpy(&result,
                              result_device,
                              sizeof(Number),
                              hipMemcpyDeviceToHost);
      AssertCuda(error_code);
      // Free the memory on the device
      Utilities::CUDA::free(result_device);

      return result;
    }



    template <typename Number>
    void
    Vector<Number>::add(const Number a)
    {
      AssertIsFinite(a);
      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);
      kernel::vec_add<Number>
        <<<n_blocks, block_size>>>(val.get(), a, n_elements);
      AssertCudaKernel();
    }



    template <typename Number>
    void
    Vector<Number>::add(const Number a, const VectorSpaceVector<Number> &V)
    {
      AssertIsFinite(a);

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number> *>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number> &>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage(
               "Cannot add two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);
      kernel::add_aV<Number><<<dim3(n_blocks, 1), dim3(block_size)>>>(
        val.get(), a, down_V.val.get(), n_elements);
      AssertCudaKernel();
    }



    template <typename Number>
    void
    Vector<Number>::add(const Number                     a,
                        const VectorSpaceVector<Number> &V,
                        const Number                     b,
                        const VectorSpaceVector<Number> &W)
    {
      AssertIsFinite(a);
      AssertIsFinite(b);

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number> *>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number> &>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage(
               "Cannot add two vectors with different numbers of elements."));

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number> *>(&W) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_W = dynamic_cast<const Vector<Number> &>(W);
      Assert(down_W.size() == this->size(),
             ExcMessage(
               "Cannot add two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);
      kernel::add_aVbW<Number><<<dim3(n_blocks, 1), dim3(block_size)>>>(
        val.get(), a, down_V.val.get(), b, down_W.val.get(), n_elements);
      AssertCudaKernel();
    }



    template <typename Number>
    void
    Vector<Number>::sadd(const Number                     s,
                         const Number                     a,
                         const VectorSpaceVector<Number> &V)
    {
      AssertIsFinite(s);
      AssertIsFinite(a);

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number> *>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number> &>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage(
               "Cannot add two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);
      kernel::sadd<Number><<<dim3(n_blocks, 1), dim3(block_size)>>>(
        s, val.get(), a, down_V.val.get(), n_elements);
      AssertCudaKernel();
    }



    template <typename Number>
    void
    Vector<Number>::scale(const VectorSpaceVector<Number> &scaling_factors)
    {
      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number> *>(&scaling_factors) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_scaling_factors =
        dynamic_cast<const Vector<Number> &>(scaling_factors);
      Assert(down_scaling_factors.size() == this->size(),
             ExcMessage(
               "Cannot scale two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);
      kernel::scale<Number><<<dim3(n_blocks, 1), dim3(block_size)>>>(
        val.get(), down_scaling_factors.val.get(), n_elements);
      AssertCudaKernel();
    }



    template <typename Number>
    void
    Vector<Number>::equ(const Number a, const VectorSpaceVector<Number> &V)
    {
      AssertIsFinite(a);

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number> *>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number> &>(V);
      Assert(
        down_V.size() == this->size(),
        ExcMessage(
          "Cannot assign two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);
      kernel::equ<Number><<<dim3(n_blocks, 1), dim3(block_size)>>>(
        val.get(), a, down_V.val.get(), n_elements);
      AssertCudaKernel();
    }



    template <typename Number>
    bool
    Vector<Number>::all_zero() const
    {
      return (linfty_norm() == 0) ? true : false;
    }



    template <typename Number>
    typename Vector<Number>::value_type
    Vector<Number>::mean_value() const
    {
      Number *    result_device;
      hipError_t error_code = hipMalloc(&result_device, sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(result_device, 0, sizeof(Number));

      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);
      kernel::reduction<Number, kernel::ElemSum<Number>>
        <<<dim3(n_blocks, 1), dim3(block_size)>>>(result_device,
                                                  val.get(),
                                                  n_elements);

      // Copy the result back to the host
      Number result;
      error_code = hipMemcpy(&result,
                              result_device,
                              sizeof(Number),
                              hipMemcpyDeviceToHost);
      AssertCuda(error_code);
      // Free the memory on the device
      Utilities::CUDA::free(result_device);

      return result /
             static_cast<typename Vector<Number>::value_type>(n_elements);
    }



    template <typename Number>
    typename Vector<Number>::real_type
    Vector<Number>::l1_norm() const
    {
      Number *    result_device;
      hipError_t error_code = hipMalloc(&result_device, sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(result_device, 0, sizeof(Number));

      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);
      kernel::reduction<Number, kernel::L1Norm<Number>>
        <<<dim3(n_blocks, 1), dim3(block_size)>>>(result_device,
                                                  val.get(),
                                                  n_elements);

      // Copy the result back to the host
      Number result;
      error_code = hipMemcpy(&result,
                              result_device,
                              sizeof(Number),
                              hipMemcpyDeviceToHost);
      AssertCuda(error_code);
      // Free the memory on the device
      Utilities::CUDA::free(result_device);

      return result;
    }



    template <typename Number>
    typename Vector<Number>::real_type
    Vector<Number>::l2_norm() const
    {
      return std::sqrt(norm_sqr());
    }



    template <typename Number>
    typename Vector<Number>::real_type
    Vector<Number>::norm_sqr() const
    {
      return (*this) * (*this);
    }



    template <typename Number>
    typename Vector<Number>::real_type
    Vector<Number>::linfty_norm() const
    {
      Number *    result_device;
      hipError_t error_code = hipMalloc(&result_device, sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(result_device, 0, sizeof(Number));

      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);
      kernel::reduction<Number, kernel::LInfty<Number>>
        <<<dim3(n_blocks, 1), dim3(block_size)>>>(result_device,
                                                  val.get(),
                                                  n_elements);

      // Copy the result back to the host
      Number result;
      error_code = hipMemcpy(&result,
                              result_device,
                              sizeof(Number),
                              hipMemcpyDeviceToHost);
      AssertCuda(error_code);
      // Free the memory on the device
      Utilities::CUDA::free(result_device);

      return result;
    }



    template <typename Number>
    Number
    Vector<Number>::add_and_dot(const Number                     a,
                                const VectorSpaceVector<Number> &V,
                                const VectorSpaceVector<Number> &W)
    {
      AssertIsFinite(a);

      // Check that casting will work
      Assert(dynamic_cast<const Vector<Number> *>(&V) != nullptr,
             ExcVectorTypeNotCompatible());
      Assert(dynamic_cast<const Vector<Number> *>(&W) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V and W. If it fails, throw an exceptiion.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number> &>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage("Vector V has the wrong size."));
      const Vector<Number> &down_W = dynamic_cast<const Vector<Number> &>(W);
      Assert(down_W.size() == this->size(),
             ExcMessage("Vector W has the wrong size."));

      Number *    result_device;
      hipError_t error_code = hipMalloc(&result_device, sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(result_device, 0, sizeof(Number));
      AssertCuda(error_code);

      const int n_blocks = 1 + (n_elements - 1) / (chunk_size * block_size);
      kernel::add_and_dot<Number>
        <<<dim3(n_blocks, 1), dim3(block_size)>>>(result_device,
                                                  val.get(),
                                                  down_V.val.get(),
                                                  down_W.val.get(),
                                                  a,
                                                  n_elements);

      Number result;
      error_code = hipMemcpy(&result,
                              result_device,
                              sizeof(Number),
                              hipMemcpyDeviceToHost);
      Utilities::CUDA::free(result_device);

      return result;
    }



    template <typename Number>
    void
    Vector<Number>::print(std::ostream &     out,
                          const unsigned int precision,
                          const bool         scientific,
                          const bool) const
    {
      AssertThrow(out, ExcIO());
      std::ios::fmtflags old_flags     = out.flags();
      unsigned int       old_precision = out.precision(precision);

      out.precision(precision);
      if (scientific)
        out.setf(std::ios::scientific, std::ios::floatfield);
      else
        out.setf(std::ios::fixed, std::ios::floatfield);

      out << "IndexSet: ";
      complete_index_set(n_elements).print(out);
      out << std::endl;

      // Copy the vector to the host
      std::vector<Number> cpu_val(n_elements);
      Utilities::CUDA::copy_to_host(val.get(), cpu_val);
      for (unsigned int i = 0; i < n_elements; ++i)
        out << cpu_val[i] << std::endl;
      out << std::flush;

      AssertThrow(out, ExcIO());
      // reset output format
      out.flags(old_flags);
      out.precision(old_precision);
    }



    template <typename Number>
    std::size_t
    Vector<Number>::memory_consumption() const
    {
      std::size_t memory = sizeof(*this);
      memory += sizeof(Number) * static_cast<std::size_t>(n_elements);

      return memory;
    }



    // Explicit Instanationation
    template class Vector<float>;
    template class Vector<double>;
  } // namespace CUDAWrappers
} // namespace LinearAlgebra

DEAL_II_NAMESPACE_CLOSE

#endif

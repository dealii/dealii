#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2016 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the deal.II distribution.
//
// ---------------------------------------------------------------------

#include <deal.II/lac/cuda_vector.h>
#include <deal.II/lac/cuda_atomic.cuh>
#include <deal.II/lac/read_write_vector.h>
#include <deal.II/base/exceptions.h>
#include <cmath>

#ifdef DEAL_II_WITH_CUDA

DEAL_II_NAMESPACE_OPEN

#define BLOCK_SIZE 512
#define CHUNK_SIZE 8

namespace LinearAlgebra
{
  namespace CUDAWrappers
  {
    namespace internal
    {
      template <typename Number>
      __global__ void vec_scale(Number                                   *val,
                                const Number                              a,
                                const typename Vector<Number>::size_type  N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*CHUNK_SIZE);
        for (unsigned int i=0; i<CHUNK_SIZE; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*BLOCK_SIZE;
            if (idx<N)
              val[idx] *= a;
          }
      }



      struct Binop_Addition
      {
        template <typename Number>
        __device__ static inline Number operation(const Number a,
                                                  const Number b)
        {
          return a+b;
        }
      };



      struct Binop_Subtraction
      {
        template <typename Number>
        __device__ static inline Number operation(const Number a,
                                                  const Number b)
        {
          return a-b;
        }
      };



      template <typename Number, typename Binop>
      __global__ void vector_bin_op(Number                                   *v1,
                                    Number                                   *v2,
                                    const typename Vector<Number>::size_type  N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*CHUNK_SIZE);
        for (unsigned int i=0; i<CHUNK_SIZE; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*BLOCK_SIZE;
            if (idx<N)
              v1[idx] = Binop::operation(v1[idx],v2[idx]);
          }
      }



      template <typename Number>
      struct ElemSum
      {
        __device__ static Number reduction_op(const Number a, const Number b)
        {
          return (a + b);
        }

        __device__ static Number atomic_op(Number *dst, const Number a)
        {
          return atomicAdd_wrapper(dst, a);
        }

        __device__ static Number element_wise_op(const Number a)
        {
          return a;
        }

        __device__ static Number null_value()
        {
          return Number();
        }
      };



      template <typename Number>
      struct L1Norm
      {
        __device__ static Number reduction_op(const Number a, const Number b)
        {
          return (a + b);
        }

        __device__ static Number atomic_op(Number *dst, const Number a)
        {
          return atomicAdd_wrapper(dst, a);
        }

        __device__ static Number element_wise_op(const Number a)
        {
          return std::fabs(a);
        }

        __device__ static Number null_value()
        {
          return Number();
        }
      };



      template <typename Number>
      struct LInfty
      {
        __device__ static Number reduction_op(const Number a, const Number b)
        {
          if  (a > b)
            return a;
          else
            return b;
        }

        __device__ static Number atomic_op(Number *dst, const Number a)
        {
          return atomicMax_wrapper(dst, a);
        }

        __device__ static Number element_wise_op(const Number a)
        {
          return std::fabs(a);
        }

        __device__ static Number null_value()
        {
          return Number();
        }
      };



      template <typename Number, typename Operation>
      __device__ void reduce_within_warp(volatile Number                    *result_buffer,
                                         typename Vector<Number>::size_type  local_idx)
      {
        if (BLOCK_SIZE >= 64)
          result_buffer[local_idx] =
            Operation::reduction_op(result_buffer[local_idx],
                                    result_buffer[local_idx+32]);
        if (BLOCK_SIZE >= 32)
          result_buffer[local_idx] =
            Operation::reduction_op(result_buffer[local_idx],
                                    result_buffer[local_idx+16]);
        if (BLOCK_SIZE >= 16)
          result_buffer[local_idx] =
            Operation::reduction_op(result_buffer[local_idx],
                                    result_buffer[local_idx+8]);
        if (BLOCK_SIZE >= 8)
          result_buffer[local_idx] =
            Operation::reduction_op(result_buffer[local_idx],
                                    result_buffer[local_idx+4]);
        if (BLOCK_SIZE >= 4)
          result_buffer[local_idx] =
            Operation::reduction_op(result_buffer[local_idx],
                                    result_buffer[local_idx+2]);
        if (BLOCK_SIZE >= 2)
          result_buffer[local_idx] =
            Operation::reduction_op(result_buffer[local_idx],
                                    result_buffer[local_idx+1]);
      }



      template <typename Number, typename Operation>
      __device__ void reduce(Number                                   *result,
                             Number                                   *result_buffer,
                             const typename Vector<Number>::size_type  local_idx,
                             const typename Vector<Number>::size_type  global_idx,
                             const typename Vector<Number>::size_type  N)
      {
        for (typename Vector<Number>::size_type s=BLOCK_SIZE/2; s>32; s=s>>1)
          {
            if (local_idx < s)
              result_buffer[local_idx] = Operation::reduction_op(result_buffer[local_idx],
                                                                 result_buffer[local_idx+s]);
            __syncthreads();
          }

        if (local_idx < 32)
          reduce_within_warp<Number,Operation>(result_buffer, local_idx);

        if (local_idx == 0)
          Operation::atomic_op(result, result_buffer[0]);
      }



      template <typename Number, typename Operation>
      __global__ void reduction(Number       *result,
                                const Number *v,
                                const typename Vector<Number>::size_type N)
      {
        __shared__ Number result_buffer[BLOCK_SIZE];

        const typename Vector<Number>::size_type global_idx = threadIdx.x +
                                                              blockIdx.x*(blockDim.x*CHUNK_SIZE);
        const typename Vector<Number>::size_type local_idx = threadIdx.x;

        if (global_idx<N)
          result_buffer[local_idx] = Operation::element_wise_op(v[global_idx]);
        else
          result_buffer[local_idx] = Operation::null_value();

        __syncthreads();

        reduce<Number,Operation> (result, result_buffer, local_idx, global_idx, N);
      }



      template <typename Number>
      struct DotProduct
      {
        __device__ static Number binary_op(const Number a, const Number b)
        {
          return a*b;
        }

        __device__ static Number reduction_op(const Number a, const Number b)
        {
          return a+b;
        }

        __device__ static Number atomic_op(Number *dst, const Number a)
        {
          return atomicAdd_wrapper(dst, a);
        }

        __device__ static Number null_value()
        {
          return Number();
        }
      };



      template <typename Number, typename Operation>
      __global__ void double_vector_reduction(Number       *result,
                                              Number *v1,
                                              Number *v2,
                                              const typename Vector<Number>::size_type N)
      {
        __shared__ Number result_buffer[BLOCK_SIZE];

        const typename Vector<Number>::size_type global_idx = threadIdx.x +
                                                              blockIdx.x*(blockDim.x*CHUNK_SIZE);
        const typename Vector<Number>::size_type local_idx = threadIdx.x;

        if (global_idx<N)
          result_buffer[local_idx] = Operation::binary_op(v1[global_idx],v2[global_idx]);
        else
          result_buffer[local_idx] = Operation::null_value();

        for (unsigned int i=1; i<CHUNK_SIZE; ++i)
          {
            const typename Vector<Number>::size_type idx = global_idx +
                                                           i*BLOCK_SIZE;
            if (idx<N)
              result_buffer[local_idx] =
                Operation::reduction_op(result_buffer[local_idx],
                                        Operation::binary_op(v1[idx], v2[idx]));
          }

        __syncthreads();

        reduce<Number,Operation> (result,result_buffer,local_idx,global_idx,N);
      }



      template <typename Number>
      __global__ void vec_add(Number       *val,
                              const Number  a,
                              const typename Vector<Number>::size_type  N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*CHUNK_SIZE);
        for (unsigned int i=0; i<CHUNK_SIZE; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*BLOCK_SIZE;
            if (idx<N)
              val[idx] += a;
          }
      }



      template <typename Number>
      __global__ void add_aV(Number       *val,
                             const Number  a,
                             Number       *V_val,
                             const typename Vector<Number>::size_type  N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*CHUNK_SIZE);
        for (unsigned int i=0; i<CHUNK_SIZE; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*BLOCK_SIZE;
            if (idx<N)
              val[idx] += a*V_val[idx];
          }
      }



      template <typename Number>
      __global__ void add_aVbW(Number       *val,
                               const Number  a,
                               Number       *V_val,
                               const Number  b,
                               Number       *W_val,
                               const typename Vector<Number>::size_type  N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*CHUNK_SIZE);
        for (unsigned int i=0; i<CHUNK_SIZE; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*BLOCK_SIZE;
            if (idx<N)
              val[idx] += a*V_val[idx] + b*W_val[idx];
          }
      }



      template <typename Number>
      __global__ void sadd(const Number  s,
                           Number       *val,
                           const Number  a,
                           const Number *V_val,
                           const typename Vector<Number>::size_type  N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*CHUNK_SIZE);
        for (unsigned int i=0; i<CHUNK_SIZE; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*BLOCK_SIZE;
            if (idx<N)
              val[idx] = s*val[idx] + a*V_val[idx];
          }
      }



      template <typename Number>
      __global__ void scale(Number       *val,
                            const Number *V_val,
                            const typename Vector<Number>::size_type N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*CHUNK_SIZE);
        for (unsigned int i=0; i<CHUNK_SIZE; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*BLOCK_SIZE;
            if (idx<N)
              val[idx] *= V_val[idx];
          }
      }



      template <typename Number>
      __global__ void equ(Number       *val,
                          const Number a,
                          const Number *V_val,
                          const typename Vector<Number>::size_type N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*CHUNK_SIZE);
        for (unsigned int i=0; i<CHUNK_SIZE; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*BLOCK_SIZE;
            if (idx<N)
              val[idx] = a * V_val[idx];
          }
      }



      template <typename Number>
      __global__ void add_and_dot(Number       *res,
                                  Number       *v1,
                                  const Number *v2,
                                  const Number *v3,
                                  const Number  a,
                                  const typename Vector<Number>::size_type N)
      {
        __shared__ Number res_buf[BLOCK_SIZE];

        const unsigned int global_idx = threadIdx.x + blockIdx.x *
                                        (blockDim.x*CHUNK_SIZE);
        const unsigned int local_idx = threadIdx.x;
        if (global_idx < N)
          {
            v1[global_idx] += a*v2[global_idx];
            res_buf[local_idx] = v1[global_idx]*v3[global_idx];
          }
        else
          res_buf[local_idx] = 0.;

        for (unsigned int i=1; i<BLOCK_SIZE; ++i)
          {
            const unsigned int idx = global_idx + i*BLOCK_SIZE;
            if (idx < N)
              {
                v1[idx] += a*v2[idx];
                res_buf[local_idx] += v1[idx]*v3[idx];
              }
          }

        __syncthreads();

        reduce<Number, DotProduct<Number>> (res, res_buf, local_idx,
                                            global_idx, N);
      }
    }



    template <typename Number>
    Vector<Number>::Vector()
      :
      val(nullptr),
      n_elements(0)
    {}



    template <typename Number>
    Vector<Number>::Vector(const Vector<Number> &V)
      :
      n_elements(V.n_elements)
    {
      // Allocate the memory
      hipError_t error_code = hipMalloc(&val, n_elements*sizeof(Number));
      AssertCuda(error_code);
      // Copy the values.
      error_code = hipMemcpy(val, V.val,n_elements*sizeof(Number),
                              hipMemcpyDeviceToDevice);
      AssertCuda(error_code);
    }



    template <typename Number>
    Vector<Number>::Vector(const size_type n)
      :
      n_elements(n)
    {
      // Allocate the memory
      hipError_t error_code = hipMalloc(&val, n_elements*sizeof(Number));
      AssertCuda(error_code);
    }



    template <typename Number>
    Vector<Number>::~Vector()
    {
      if (val != nullptr)
        {
          hipError_t error_code = hipFree(val);
          AssertCuda(error_code);
          val = nullptr;
          n_elements = 0;
        }
    }



    template <typename Number>
    void Vector<Number>::reinit(const size_type n,
                                const bool      omit_zeroing_entries)
    {
      // Resize the underlying array if necessary
      if (n == 0)
        {
          if (val != nullptr)
            {
              hipError_t error_code = hipFree(val);
              AssertCuda(error_code);
              val = nullptr;
            }
        }
      else
        {
          if (n_elements != n)
            {
              hipError_t error_code = hipFree(val);
              AssertCuda(error_code);
            }

          hipError_t error_code = hipMalloc(&val, n*sizeof(Number));
          AssertCuda(error_code);

          // If necessary set the elements to zero
          if (omit_zeroing_entries == false)
            {
              hipError_t error_code = hipMemset(val, 0,
                                                  n_elements*sizeof(Number));
              AssertCuda(error_code);
            }
        }
      n_elements = n;
    }



    template <typename Number>
    void Vector<Number>::import(const ReadWriteVector<Number> &V,
                                VectorOperation::values operation,
                                std_cxx11::shared_ptr<const CommunicationPatternBase> )
    {
      if (operation == VectorOperation::insert)
        {
          hipError_t error_code = hipMemcpy(val, V.begin(),
                                              n_elements*sizeof(Number),
                                              hipMemcpyHostToDevice);
          AssertCuda(error_code);
        }
      else
        {
          // Create a temporary vector on the device
          Number *tmp;
          hipError_t error_code = hipMalloc(&tmp, n_elements*sizeof(Number));
          AssertCuda(error_code);

          // Copy the vector from the host to the temporary vector on the device
          error_code = hipMemcpy(&tmp[0], V.begin(), n_elements*sizeof(Number),
                                  hipMemcpyHostToDevice);
          AssertCuda(error_code);

          // Add the two vectors
          const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);

          internal::vector_bin_op<Number,internal::Binop_Addition>
          <<<n_blocks,BLOCK_SIZE>>>(val, tmp, n_elements);
          // Check that the kernel was launched correctly
          AssertCuda(hipGetLastError());
          // Check that there was no problem during the execution of the kernel
          AssertCuda(hipDeviceSynchronize());

          // Delete the temporary vector
          error_code = hipFree(tmp);
          AssertCuda(error_code);
        }
    }



    template <typename Number>
    Vector<Number> &Vector<Number>::operator*= (const Number factor)
    {
      AssertIsFinite(factor);
      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);
      internal::vec_scale<Number> <<<n_blocks,BLOCK_SIZE>>>(val,
                                                            factor, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());

      return *this;
    }



    template <typename Number>
    Vector<Number> &Vector<Number>::operator/= (const Number factor)
    {
      AssertIsFinite(factor);
      Assert(factor!=Number(0.), ExcZero());
      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);
      internal::vec_scale<Number> <<<n_blocks,BLOCK_SIZE>>>(val,
                                                            1./factor, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());

      return *this;
    }



    template <typename Number>
    Vector<Number> &Vector<Number>::operator+= (const VectorSpaceVector<Number> &V)
    {
      // Check that casting will work
      Assert(dynamic_cast<const Vector<Number>*>(&V)!=nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If it fails, it throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size()==this->size(),
             ExcMessage("Cannot add two vectors with different numbers of elements"));

      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);

      internal::vector_bin_op<Number,internal::Binop_Addition>
      <<<n_blocks,BLOCK_SIZE>>>(val, down_V.val, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());

      return *this;
    }



    template <typename Number>
    Vector<Number> &Vector<Number>::operator-= (const VectorSpaceVector<Number> &V)
    {
      // Check that casting will work
      Assert(dynamic_cast<const Vector<Number>*>(&V)!=nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throws an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size()==this->size(),
             ExcMessage("Cannot add two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);

      internal::vector_bin_op<Number,internal::Binop_Subtraction>
      <<<n_blocks,BLOCK_SIZE>>>(val, down_V.val, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());

      return *this;
    }



    template <typename Number>
    Number Vector<Number>::operator* (const VectorSpaceVector<Number> &V) const
    {
      // Check that casting will work
      Assert(dynamic_cast<const Vector<Number>*>(&V)!=nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throws an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size()==this->size(),
             ExcMessage("Cannot add two vectors with different numbers of elements"));

      Number *result_device;
      hipError_t error_code = hipMalloc(&result_device, n_elements*sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(result_device, Number(), sizeof(Number));

      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);
      internal::double_vector_reduction<Number, internal::DotProduct<Number>>
          <<<dim3(n_blocks,1),dim3(BLOCK_SIZE)>>> (result_device, val,
                                                   down_V.val,
                                                   static_cast<unsigned int>(n_elements));

      // Copy the result back to the host
      Number result;
      error_code = hipMemcpy(&result, result_device, sizeof(Number),
                              hipMemcpyDeviceToHost);
      AssertCuda(error_code);
      // Free the memory on the device
      error_code = hipFree(result_device);
      AssertCuda(error_code);

      return result;
    }



    template <typename Number>
    void Vector<Number>::add(const Number a)
    {
      AssertIsFinite(a);
      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);
      internal::vec_add<Number> <<<n_blocks,BLOCK_SIZE>>>(val, a,
                                                          n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());
    }



    template <typename Number>
    void Vector<Number>::add(const Number a, const VectorSpaceVector<Number> &V)
    {
      AssertIsFinite(a);

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number>*>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage("Cannot add two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);
      internal::add_aV<Number> <<<dim3(n_blocks,1),dim3(BLOCK_SIZE)>>> (val,
          a, down_V.val, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());
    }



    template <typename Number>
    void Vector<Number>::add(const Number a, const VectorSpaceVector<Number> &V,
                             const Number b, const VectorSpaceVector<Number> &W)
    {
      AssertIsFinite(a);
      AssertIsFinite(b);

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number>*>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage("Cannot add two vectors with different numbers of elements."));

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number>*>(&W) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_W = dynamic_cast<const Vector<Number>&>(W);
      Assert(down_W.size() == this->size(),
             ExcMessage("Cannot add two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);
      internal::add_aVbW<Number> <<<dim3(n_blocks,1),dim3(BLOCK_SIZE)>>> (val,
          a, down_V.val, b, down_W.val, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());
    }



    template <typename Number>
    void Vector<Number>::sadd(const Number s, const Number a,
                              const VectorSpaceVector<Number> &V)
    {
      AssertIsFinite(s);
      AssertIsFinite(a);

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number>*>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage("Cannot add two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);
      internal::sadd<Number> <<<dim3(n_blocks,1),dim3(BLOCK_SIZE)>>> (s, val,
          a, down_V.val, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());
    }



    template <typename Number>
    void Vector<Number>::scale(const VectorSpaceVector<Number> &scaling_factors)
    {
      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number>*>(&scaling_factors) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_scaling_factors =
        dynamic_cast<const Vector<Number>&>(scaling_factors);
      Assert(down_scaling_factors.size() == this->size(),
             ExcMessage("Cannot scale two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);
      internal::scale<Number> <<<dim3(n_blocks,1),dim3(BLOCK_SIZE)>>> (val,
          down_scaling_factors.val, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());
    }



    template <typename Number>
    void Vector<Number>::equ(const Number a, const VectorSpaceVector<Number> &V)
    {
      AssertIsFinite(a);

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number>*>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage("Cannot assign two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);
      internal::equ<Number> <<<dim3(n_blocks,1),dim3(BLOCK_SIZE)>>> (val, a,
          down_V.val, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());
    }



    template <typename Number>
    typename Vector<Number>::value_type Vector<Number>::mean_value() const
    {
      Number *result_device;
      hipError_t error_code = hipMalloc(&result_device, sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(result_device, Number(), sizeof(Number));

      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);
      internal::reduction<Number, internal::ElemSum<Number>>
                                                          <<<dim3(n_blocks,1),dim3(BLOCK_SIZE)>>> (
                                                            result_device, val,
                                                            n_elements);

      // Copy the result back to the host
      Number result;
      error_code = hipMemcpy(&result, result_device, sizeof(Number),
                              hipMemcpyDeviceToHost);
      AssertCuda(error_code);
      // Free the memory on the device
      error_code = hipFree(result_device);
      AssertCuda(error_code);

      return result/static_cast<typename Vector<Number>::value_type>(n_elements);
    }



    template <typename Number>
    typename Vector<Number>::real_type Vector<Number>::l1_norm() const
    {
      Number *result_device;
      hipError_t error_code = hipMalloc(&result_device, sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(result_device, Number(), sizeof(Number));

      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);
      internal::reduction<Number, internal::L1Norm<Number>>
                                                         <<<dim3(n_blocks,1),dim3(BLOCK_SIZE)>>> (
                                                           result_device, val,
                                                           n_elements);

      // Copy the result back to the host
      Number result;
      error_code = hipMemcpy(&result, result_device, sizeof(Number),
                              hipMemcpyDeviceToHost);
      AssertCuda(error_code);
      // Free the memory on the device
      error_code = hipFree(result_device);
      AssertCuda(error_code);

      return result;
    }



    template <typename Number>
    typename Vector<Number>::real_type Vector<Number>::l2_norm() const
    {
      return std::sqrt((*this)*(*this));
    }



    template <typename Number>
    typename Vector<Number>::real_type Vector<Number>::linfty_norm() const
    {
      Number *result_device;
      hipError_t error_code = hipMalloc(&result_device, sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(result_device, Number(), sizeof(Number));

      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);
      internal::reduction<Number, internal::LInfty<Number>>
                                                         <<<dim3(n_blocks,1),dim3(BLOCK_SIZE)>>> (
                                                           result_device, val,
                                                           n_elements);

      // Copy the result back to the host
      Number result;
      error_code = hipMemcpy(&result, result_device, sizeof(Number),
                              hipMemcpyDeviceToHost);
      AssertCuda(error_code);
      // Free the memory on the device
      error_code = hipFree(result_device);
      AssertCuda(error_code);

      return result;
    }



    template <typename Number>
    Number Vector<Number>::add_and_dot(const Number                     a,
                                       const VectorSpaceVector<Number> &V,
                                       const VectorSpaceVector<Number> &W)
    {
      AssertIsFinite(a);

      // Check that casting will work
      Assert(dynamic_cast<const Vector<Number>*>(&V)!=nullptr,
             ExcVectorTypeNotCompatible());
      Assert(dynamic_cast<const Vector<Number>*>(&W)!=nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V and W. If it fails, throw an exceptiion.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage("Vector V has the wrong size."));
      const Vector<Number> &down_W = dynamic_cast<const Vector<Number>&>(W);
      Assert(down_W.size() == this->size(),
             ExcMessage("Vector W has the wrong size."));

      Number *res_d;
      hipError_t error_code = hipMalloc(&res_d, sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(res_d, 0., sizeof(Number));
      AssertCuda(error_code);

      const int n_blocks = 1 + (n_elements-1)/(CHUNK_SIZE*BLOCK_SIZE);
      internal::add_and_dot<Number> <<<dim3(n_blocks,1),dim3(BLOCK_SIZE)>>>(
        res_d, val, down_V.val, down_W.val, a, n_elements);

      Number res;
      error_code = hipMemcpy(&res, res_d, sizeof(Number), hipMemcpyDeviceToHost);
      AssertCuda(error_code);
      error_code = hipFree(res_d);

      return res;
    }



    template <typename Number>
    void Vector<Number>::print(std::ostream       &out,
                               const unsigned int  precision,
                               const bool          scientific,
                               const bool          across) const
    {
      AssertThrow(out, ExcIO());
      std::ios::fmtflags old_flags = out.flags();
      unsigned int old_precision = out.precision (precision);

      out.precision (precision);
      if (scientific)
        out.setf (std::ios::scientific, std::ios::floatfield);
      else
        out.setf (std::ios::fixed, std::ios::floatfield);

      out << "IndexSet: ";
      complete_index_set(n_elements).print(out);
      out << std::endl;

      // Copy the vector to the host
      Number *cpu_val = new Number[n_elements];
      hipError_t error_code = hipMemcpy(cpu_val, val,
                                          n_elements*sizeof(Number),
                                          hipMemcpyHostToDevice);
      AssertCuda(error_code);
      for (unsigned int i=0; i<n_elements; ++i)
        out << cpu_val[i] << std::endl;
      out << std::flush;
      delete [] cpu_val;
      cpu_val = nullptr;


      AssertThrow (out, ExcIO());
      // reset output format
      out.flags (old_flags);
      out.precision(old_precision);
    }



    template <typename Number>
    std::size_t Vector<Number>::memory_consumption() const
    {
      std::size_t memory = sizeof(*this);
      memory += sizeof (Number) * static_cast<std::size_t>(n_elements);

      return memory;
    }



    // Explicit Instanationation
    template class Vector<float>;
    template class Vector<double>;
  }
}

DEAL_II_NAMESPACE_CLOSE

#endif

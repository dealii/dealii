#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2016 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the deal.II distribution.
//
// ---------------------------------------------------------------------

#include <deal.II/lac/cuda_vector.h>
#include <deal.II/lac/cuda_atomic.h>
#include <deal.II/lac/read_write_vector.h>
#include <deal.II/base/exceptions.h>
#include <deal.II/base/cuda_size.h>
#include <cmath>

#ifdef DEAL_II_WITH_CUDA

DEAL_II_NAMESPACE_OPEN

namespace LinearAlgebra
{
  namespace CUDAWrappers
  {
    using ::dealii::CUDAWrappers::block_size;
    using ::dealii::CUDAWrappers::chunk_size;
    namespace internal
    {
      template <typename Number>
      __global__ void
      vec_scale(Number                                   *val,
                const Number                              a,
                const typename Vector<Number>::size_type  N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*chunk_size);
        for (unsigned int i=0; i<chunk_size; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*block_size;
            if (idx<N)
              val[idx] *= a;
          }
      }



      struct Binop_Addition
      {
        template <typename Number>
        __device__ static inline Number
        operation(const Number a,
                  const Number b)
        {
          return a+b;
        }
      };



      struct Binop_Subtraction
      {
        template <typename Number>
        __device__ static inline Number
        operation(const Number a,
                  const Number b)
        {
          return a-b;
        }
      };



      template <typename Number, typename Binop>
      __global__ void
      vector_bin_op(Number                                   *v1,
                    Number                                   *v2,
                    const typename Vector<Number>::size_type  N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*chunk_size);
        for (unsigned int i=0; i<chunk_size; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*block_size;
            if (idx<N)
              v1[idx] = Binop::operation(v1[idx],v2[idx]);
          }
      }



      template <typename Number>
      struct ElemSum
      {
        __device__ static Number
        reduction_op(const Number a, const Number b)
        {
          return (a + b);
        }

        __device__ static Number
        atomic_op(Number *dst, const Number a)
        {
          return atomicAdd_wrapper(dst, a);
        }

        __device__ static Number
        element_wise_op(const Number a)
        {
          return a;
        }

        __device__ static Number
        null_value()
        {
          return Number();
        }
      };



      template <typename Number>
      struct L1Norm
      {
        __device__ static Number
        reduction_op(const Number a, const Number b)
        {
          return (a + b);
        }

        __device__ static Number
        atomic_op(Number *dst, const Number a)
        {
          return atomicAdd_wrapper(dst, a);
        }

        __device__ static Number
        element_wise_op(const Number a)
        {
          return std::fabs(a);
        }

        __device__ static Number
        null_value()
        {
          return Number();
        }
      };



      template <typename Number>
      struct LInfty
      {
        __device__ static Number
        reduction_op(const Number a, const Number b)
        {
          if  (a > b)
            return a;
          else
            return b;
        }

        __device__ static Number
        atomic_op(Number *dst, const Number a)
        {
          return atomicMax_wrapper(dst, a);
        }

        __device__ static Number
        element_wise_op(const Number a)
        {
          return std::fabs(a);
        }

        __device__ static Number
        null_value()
        {
          return Number();
        }
      };



      template <typename Number, typename Operation>
      __device__ void
      reduce_within_warp(volatile Number                    *result_buffer,
                         typename Vector<Number>::size_type  local_idx)
      {
        if (block_size >= 64)
          result_buffer[local_idx] =
            Operation::reduction_op(result_buffer[local_idx],
                                    result_buffer[local_idx+32]);
        if (block_size >= 32)
          result_buffer[local_idx] =
            Operation::reduction_op(result_buffer[local_idx],
                                    result_buffer[local_idx+16]);
        if (block_size >= 16)
          result_buffer[local_idx] =
            Operation::reduction_op(result_buffer[local_idx],
                                    result_buffer[local_idx+8]);
        if (block_size >= 8)
          result_buffer[local_idx] =
            Operation::reduction_op(result_buffer[local_idx],
                                    result_buffer[local_idx+4]);
        if (block_size >= 4)
          result_buffer[local_idx] =
            Operation::reduction_op(result_buffer[local_idx],
                                    result_buffer[local_idx+2]);
        if (block_size >= 2)
          result_buffer[local_idx] =
            Operation::reduction_op(result_buffer[local_idx],
                                    result_buffer[local_idx+1]);
      }



      template <typename Number, typename Operation>
      __device__ void
      reduce(Number                                   *result,
             Number                                   *result_buffer,
             const typename Vector<Number>::size_type  local_idx,
             const typename Vector<Number>::size_type  global_idx,
             const typename Vector<Number>::size_type  N)
      {
        for (typename Vector<Number>::size_type s=block_size/2; s>32; s=s>>1)
          {
            if (local_idx < s)
              result_buffer[local_idx] = Operation::reduction_op(result_buffer[local_idx],
                                                                 result_buffer[local_idx+s]);
            __syncthreads();
          }

        if (local_idx < 32)
          reduce_within_warp<Number,Operation>(result_buffer, local_idx);

        if (local_idx == 0)
          Operation::atomic_op(result, result_buffer[0]);
      }



      template <typename Number, typename Operation>
      __global__ void
      reduction(Number       *result,
                const Number *v,
                const typename Vector<Number>::size_type N)
      {
        __shared__ Number result_buffer[block_size];

        const typename Vector<Number>::size_type global_idx = threadIdx.x +
                                                              blockIdx.x*(blockDim.x*chunk_size);
        const typename Vector<Number>::size_type local_idx = threadIdx.x;

        if (global_idx<N)
          result_buffer[local_idx] = Operation::element_wise_op(v[global_idx]);
        else
          result_buffer[local_idx] = Operation::null_value();

        __syncthreads();

        reduce<Number,Operation> (result, result_buffer, local_idx, global_idx, N);
      }



      template <typename Number>
      struct DotProduct
      {
        __device__ static Number
        binary_op(const Number a, const Number b)
        {
          return a*b;
        }

        __device__ static Number
        reduction_op(const Number a, const Number b)
        {
          return a+b;
        }

        __device__ static Number
        atomic_op(Number *dst, const Number a)
        {
          return atomicAdd_wrapper(dst, a);
        }

        __device__ static Number
        null_value()
        {
          return Number();
        }
      };



      template <typename Number, typename Operation>
      __global__ void
      double_vector_reduction(Number       *result,
                              Number *v1,
                              Number *v2,
                              const typename Vector<Number>::size_type N)
      {
        __shared__ Number result_buffer[block_size];

        const typename Vector<Number>::size_type global_idx = threadIdx.x +
                                                              blockIdx.x*(blockDim.x*chunk_size);
        const typename Vector<Number>::size_type local_idx = threadIdx.x;

        if (global_idx<N)
          result_buffer[local_idx] = Operation::binary_op(v1[global_idx],v2[global_idx]);
        else
          result_buffer[local_idx] = Operation::null_value();

        for (unsigned int i=1; i<chunk_size; ++i)
          {
            const typename Vector<Number>::size_type idx = global_idx +
                                                           i*block_size;
            if (idx<N)
              result_buffer[local_idx] =
                Operation::reduction_op(result_buffer[local_idx],
                                        Operation::binary_op(v1[idx], v2[idx]));
          }

        __syncthreads();

        reduce<Number,Operation> (result,result_buffer,local_idx,global_idx,N);
      }



      template <typename Number>
      __global__ void
      vec_add(Number       *val,
              const Number  a,
              const typename Vector<Number>::size_type  N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*chunk_size);
        for (unsigned int i=0; i<chunk_size; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*block_size;
            if (idx<N)
              val[idx] += a;
          }
      }



      template <typename Number>
      __global__ void
      add_aV(Number       *val,
             const Number  a,
             Number       *V_val,
             const typename Vector<Number>::size_type  N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*chunk_size);
        for (unsigned int i=0; i<chunk_size; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*block_size;
            if (idx<N)
              val[idx] += a*V_val[idx];
          }
      }



      template <typename Number>
      __global__ void
      add_aVbW(Number       *val,
               const Number  a,
               Number       *V_val,
               const Number  b,
               Number       *W_val,
               const typename Vector<Number>::size_type  N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*chunk_size);
        for (unsigned int i=0; i<chunk_size; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*block_size;
            if (idx<N)
              val[idx] += a*V_val[idx] + b*W_val[idx];
          }
      }



      template <typename Number>
      __global__ void
      sadd(const Number  s,
           Number       *val,
           const Number  a,
           const Number *V_val,
           const typename Vector<Number>::size_type  N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*chunk_size);
        for (unsigned int i=0; i<chunk_size; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*block_size;
            if (idx<N)
              val[idx] = s*val[idx] + a*V_val[idx];
          }
      }



      template <typename Number>
      __global__ void
      scale(Number       *val,
            const Number *V_val,
            const typename Vector<Number>::size_type N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*chunk_size);
        for (unsigned int i=0; i<chunk_size; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*block_size;
            if (idx<N)
              val[idx] *= V_val[idx];
          }
      }



      template <typename Number>
      __global__ void
      equ(Number       *val,
          const Number a,
          const Number *V_val,
          const typename Vector<Number>::size_type N)
      {
        const typename Vector<Number>::size_type idx_base = threadIdx.x +
                                                            blockIdx.x *
                                                            (blockDim.x*chunk_size);
        for (unsigned int i=0; i<chunk_size; ++i)
          {
            const typename Vector<Number>::size_type idx = idx_base +
                                                           i*block_size;
            if (idx<N)
              val[idx] = a * V_val[idx];
          }
      }



      template <typename Number>
      __global__ void
      add_and_dot(Number       *res,
                  Number       *v1,
                  const Number *v2,
                  const Number *v3,
                  const Number  a,
                  const typename Vector<Number>::size_type N)
      {
        __shared__ Number res_buf[block_size];

        const unsigned int global_idx = threadIdx.x + blockIdx.x *
                                        (blockDim.x*chunk_size);
        const unsigned int local_idx = threadIdx.x;
        if (global_idx < N)
          {
            v1[global_idx] += a*v2[global_idx];
            res_buf[local_idx] = v1[global_idx] * Number(numbers::NumberTraits<Number>::conjugate(v3[global_idx]));
          }
        else
          res_buf[local_idx] = 0.;

        for (unsigned int i=1; i<block_size; ++i)
          {
            const unsigned int idx = global_idx + i*block_size;
            if (idx < N)
              {
                v1[idx] += a*v2[idx];
                res_buf[local_idx] += v1[idx]*v3[idx];
              }
          }

        __syncthreads();

        reduce<Number, DotProduct<Number>> (res, res_buf, local_idx,
                                            global_idx, N);
      }
    }



    template <typename Number>
    Vector<Number>::Vector()
      :
      val(nullptr),
      n_elements(0)
    {}



    template <typename Number>
    Vector<Number>::Vector(const Vector<Number> &V)
      :
      n_elements(V.n_elements)
    {
      // Allocate the memory
      hipError_t error_code = hipMalloc(&val, n_elements*sizeof(Number));
      AssertCuda(error_code);
      // Copy the values.
      error_code = hipMemcpy(val, V.val,n_elements*sizeof(Number),
                              hipMemcpyDeviceToDevice);
      AssertCuda(error_code);
    }



    template <typename Number>
    Vector<Number>::Vector(const size_type n)
      :
      val(nullptr),
      n_elements(0)
    {
      reinit(n, false);
    }



    template <typename Number>
    Vector<Number>::~Vector()
    {
      if (val != nullptr)
        {
          hipError_t error_code = hipFree(val);
          AssertCuda(error_code);
          val = nullptr;
          n_elements = 0;
        }
    }



    template <typename Number>
    void
    Vector<Number>::reinit(const size_type n,
                           const bool      omit_zeroing_entries)
    {
      // Resize the underlying array if necessary
      if (n == 0)
        {
          if (val != nullptr)
            {
              hipError_t error_code = hipFree(val);
              AssertCuda(error_code);
              val = nullptr;
            }
        }
      else
        {
          if ((n_elements != n) && (val != nullptr))
            {
              hipError_t error_code = hipFree(val);
              AssertCuda(error_code);
            }

          hipError_t error_code = hipMalloc(&val, n*sizeof(Number));
          AssertCuda(error_code);

          // If necessary set the elements to zero
          if (omit_zeroing_entries == false)
            {
              hipError_t error_code = hipMemset(val, 0,
                                                  n*sizeof(Number));
              AssertCuda(error_code);
            }
        }
      n_elements = n;
    }



    template <typename Number>
    void
    Vector<Number>::reinit(const VectorSpaceVector<Number> &V,
                           const bool omit_zeroing_entries)
    {
      reinit(V.size(), omit_zeroing_entries);
    }



    template <typename Number>
    void
    Vector<Number>::import(const ReadWriteVector<Number> &V,
                           VectorOperation::values operation,
                           std::shared_ptr<const CommunicationPatternBase> )
    {
      if (operation == VectorOperation::insert)
        {
          hipError_t error_code = hipMemcpy(val, V.begin(),
                                              n_elements*sizeof(Number),
                                              hipMemcpyHostToDevice);
          AssertCuda(error_code);
        }
      else
        {
          // Create a temporary vector on the device
          Number *tmp;
          hipError_t error_code = hipMalloc(&tmp, n_elements*sizeof(Number));
          AssertCuda(error_code);

          // Copy the vector from the host to the temporary vector on the device
          error_code = hipMemcpy(tmp, V.begin(), n_elements*sizeof(Number),
                                  hipMemcpyHostToDevice);
          AssertCuda(error_code);

          // Add the two vectors
          const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);

          internal::vector_bin_op<Number,internal::Binop_Addition>
          <<<n_blocks,block_size>>>(val, tmp, n_elements);
          // Check that the kernel was launched correctly
          AssertCuda(hipGetLastError());
          // Check that there was no problem during the execution of the kernel
          AssertCuda(hipDeviceSynchronize());

          // Delete the temporary vector
          error_code = hipFree(tmp);
          AssertCuda(error_code);
        }
    }



    template <typename Number>
    Vector<Number> &
    Vector<Number>::operator= (const Number s)
    {
      Assert(s == Number(), ExcMessage("Onlyt 0 can be assigned to a vector."));
      (void)s;

      hipError_t error_code = hipMemset(val, 0, n_elements*sizeof(Number));
      AssertCuda(error_code);

      return *this;
    }



    template <typename Number>
    Vector<Number> &
    Vector<Number>::operator*= (const Number factor)
    {
      AssertIsFinite(factor);
      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);
      internal::vec_scale<Number> <<<n_blocks,block_size>>>(val,
                                                            factor, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());

      return *this;
    }



    template <typename Number>
    Vector<Number> &
    Vector<Number>::operator/= (const Number factor)
    {
      AssertIsFinite(factor);
      Assert(factor!=Number(0.), ExcZero());
      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);
      internal::vec_scale<Number> <<<n_blocks,block_size>>>(val,
                                                            1./factor, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());

      return *this;
    }



    template <typename Number>
    Vector<Number> &
    Vector<Number>::operator+= (const VectorSpaceVector<Number> &V)
    {
      // Check that casting will work
      Assert(dynamic_cast<const Vector<Number>*>(&V)!=nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If it fails, it throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size()==this->size(),
             ExcMessage("Cannot add two vectors with different numbers of elements"));

      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);

      internal::vector_bin_op<Number,internal::Binop_Addition>
      <<<n_blocks,block_size>>>(val, down_V.val, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());

      return *this;
    }



    template <typename Number>
    Vector<Number> &
    Vector<Number>::operator-= (const VectorSpaceVector<Number> &V)
    {
      // Check that casting will work
      Assert(dynamic_cast<const Vector<Number>*>(&V)!=nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throws an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size()==this->size(),
             ExcMessage("Cannot add two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);

      internal::vector_bin_op<Number,internal::Binop_Subtraction>
      <<<n_blocks,block_size>>>(val, down_V.val, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());

      return *this;
    }



    template <typename Number>
    Number
    Vector<Number>::operator* (const VectorSpaceVector<Number> &V) const
    {
      // Check that casting will work
      Assert(dynamic_cast<const Vector<Number>*>(&V)!=nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throws an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size()==this->size(),
             ExcMessage("Cannot add two vectors with different numbers of elements"));

      Number *result_device;
      hipError_t error_code = hipMalloc(&result_device, n_elements*sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(result_device, Number(), sizeof(Number));

      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);
      internal::double_vector_reduction<Number, internal::DotProduct<Number>>
          <<<dim3(n_blocks,1),dim3(block_size)>>> (result_device, val,
                                                   down_V.val,
                                                   static_cast<unsigned int>(n_elements));

      // Copy the result back to the host
      Number result;
      error_code = hipMemcpy(&result, result_device, sizeof(Number),
                              hipMemcpyDeviceToHost);
      AssertCuda(error_code);
      // Free the memory on the device
      error_code = hipFree(result_device);
      AssertCuda(error_code);

      return result;
    }



    template <typename Number>
    void
    Vector<Number>::add(const Number a)
    {
      AssertIsFinite(a);
      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);
      internal::vec_add<Number> <<<n_blocks,block_size>>>(val, a,
                                                          n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());
    }



    template <typename Number>
    void
    Vector<Number>::add(const Number a, const VectorSpaceVector<Number> &V)
    {
      AssertIsFinite(a);

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number>*>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage("Cannot add two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);
      internal::add_aV<Number> <<<dim3(n_blocks,1),dim3(block_size)>>> (val,
          a, down_V.val, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());
    }



    template <typename Number>
    void
    Vector<Number>::add(const Number a, const VectorSpaceVector<Number> &V,
                        const Number b, const VectorSpaceVector<Number> &W)
    {
      AssertIsFinite(a);
      AssertIsFinite(b);

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number>*>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage("Cannot add two vectors with different numbers of elements."));

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number>*>(&W) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_W = dynamic_cast<const Vector<Number>&>(W);
      Assert(down_W.size() == this->size(),
             ExcMessage("Cannot add two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);
      internal::add_aVbW<Number> <<<dim3(n_blocks,1),dim3(block_size)>>> (val,
          a, down_V.val, b, down_W.val, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());
    }



    template <typename Number>
    void
    Vector<Number>::sadd(const Number s, const Number a,
                         const VectorSpaceVector<Number> &V)
    {
      AssertIsFinite(s);
      AssertIsFinite(a);

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number>*>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage("Cannot add two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);
      internal::sadd<Number> <<<dim3(n_blocks,1),dim3(block_size)>>> (s, val,
          a, down_V.val, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());
    }



    template <typename Number>
    void
    Vector<Number>::scale(const VectorSpaceVector<Number> &scaling_factors)
    {
      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number>*>(&scaling_factors) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_scaling_factors =
        dynamic_cast<const Vector<Number>&>(scaling_factors);
      Assert(down_scaling_factors.size() == this->size(),
             ExcMessage("Cannot scale two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);
      internal::scale<Number> <<<dim3(n_blocks,1),dim3(block_size)>>> (val,
          down_scaling_factors.val, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());
    }



    template <typename Number>
    void
    Vector<Number>::equ(const Number a, const VectorSpaceVector<Number> &V)
    {
      AssertIsFinite(a);

      // Check that casting will work.
      Assert(dynamic_cast<const Vector<Number>*>(&V) != nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V. If fails, throw an exception.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage("Cannot assign two vectors with different numbers of elements."));

      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);
      internal::equ<Number> <<<dim3(n_blocks,1),dim3(block_size)>>> (val, a,
          down_V.val, n_elements);

      // Check that the kernel was launched correctly
      AssertCuda(hipGetLastError());
      // Check that there was no problem during the execution of the kernel
      AssertCuda(hipDeviceSynchronize());
    }



    template <typename Number>
    bool
    Vector<Number>::all_zero() const
    {
      return (linfty_norm() == 0) ? true : false;
    }



    template <typename Number>
    typename Vector<Number>::value_type
    Vector<Number>::mean_value() const
    {
      Number *result_device;
      hipError_t error_code = hipMalloc(&result_device, sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(result_device, Number(), sizeof(Number));

      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);
      internal::reduction<Number, internal::ElemSum<Number>>
                                                          <<<dim3(n_blocks,1),dim3(block_size)>>> (
                                                            result_device, val,
                                                            n_elements);

      // Copy the result back to the host
      Number result;
      error_code = hipMemcpy(&result, result_device, sizeof(Number),
                              hipMemcpyDeviceToHost);
      AssertCuda(error_code);
      // Free the memory on the device
      error_code = hipFree(result_device);
      AssertCuda(error_code);

      return result/static_cast<typename Vector<Number>::value_type>(n_elements);
    }



    template <typename Number>
    typename Vector<Number>::real_type
    Vector<Number>::l1_norm() const
    {
      Number *result_device;
      hipError_t error_code = hipMalloc(&result_device, sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(result_device, Number(), sizeof(Number));

      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);
      internal::reduction<Number, internal::L1Norm<Number>>
                                                         <<<dim3(n_blocks,1),dim3(block_size)>>> (
                                                           result_device, val,
                                                           n_elements);

      // Copy the result back to the host
      Number result;
      error_code = hipMemcpy(&result, result_device, sizeof(Number),
                              hipMemcpyDeviceToHost);
      AssertCuda(error_code);
      // Free the memory on the device
      error_code = hipFree(result_device);
      AssertCuda(error_code);

      return result;
    }



    template <typename Number>
    typename Vector<Number>::real_type
    Vector<Number>::l2_norm() const
    {
      return std::sqrt((*this)*(*this));
    }



    template <typename Number>
    typename Vector<Number>::real_type
    Vector<Number>::linfty_norm() const
    {
      Number *result_device;
      hipError_t error_code = hipMalloc(&result_device, sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(result_device, Number(), sizeof(Number));

      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);
      internal::reduction<Number, internal::LInfty<Number>>
                                                         <<<dim3(n_blocks,1),dim3(block_size)>>> (
                                                           result_device, val,
                                                           n_elements);

      // Copy the result back to the host
      Number result;
      error_code = hipMemcpy(&result, result_device, sizeof(Number),
                              hipMemcpyDeviceToHost);
      AssertCuda(error_code);
      // Free the memory on the device
      error_code = hipFree(result_device);
      AssertCuda(error_code);

      return result;
    }



    template <typename Number>
    Number
    Vector<Number>::add_and_dot(const Number                     a,
                                const VectorSpaceVector<Number> &V,
                                const VectorSpaceVector<Number> &W)
    {
      AssertIsFinite(a);

      // Check that casting will work
      Assert(dynamic_cast<const Vector<Number>*>(&V)!=nullptr,
             ExcVectorTypeNotCompatible());
      Assert(dynamic_cast<const Vector<Number>*>(&W)!=nullptr,
             ExcVectorTypeNotCompatible());

      // Downcast V and W. If it fails, throw an exceptiion.
      const Vector<Number> &down_V = dynamic_cast<const Vector<Number>&>(V);
      Assert(down_V.size() == this->size(),
             ExcMessage("Vector V has the wrong size."));
      const Vector<Number> &down_W = dynamic_cast<const Vector<Number>&>(W);
      Assert(down_W.size() == this->size(),
             ExcMessage("Vector W has the wrong size."));

      Number *res_d;
      hipError_t error_code = hipMalloc(&res_d, sizeof(Number));
      AssertCuda(error_code);
      error_code = hipMemset(res_d, 0., sizeof(Number));
      AssertCuda(error_code);

      const int n_blocks = 1 + (n_elements-1)/(chunk_size*block_size);
      internal::add_and_dot<Number> <<<dim3(n_blocks,1),dim3(block_size)>>>(
        res_d, val, down_V.val, down_W.val, a, n_elements);

      Number res;
      error_code = hipMemcpy(&res, res_d, sizeof(Number), hipMemcpyDeviceToHost);
      AssertCuda(error_code);
      error_code = hipFree(res_d);

      return res;
    }



    template <typename Number>
    void
    Vector<Number>::print(std::ostream       &out,
                          const unsigned int  precision,
                          const bool          scientific,
                          const bool          ) const
    {
      AssertThrow(out, ExcIO());
      std::ios::fmtflags old_flags = out.flags();
      unsigned int old_precision = out.precision (precision);

      out.precision (precision);
      if (scientific)
        out.setf (std::ios::scientific, std::ios::floatfield);
      else
        out.setf (std::ios::fixed, std::ios::floatfield);

      out << "IndexSet: ";
      complete_index_set(n_elements).print(out);
      out << std::endl;

      // Copy the vector to the host
      Number *cpu_val = new Number[n_elements];
      hipError_t error_code = hipMemcpy(cpu_val, val,
                                          n_elements*sizeof(Number),
                                          hipMemcpyHostToDevice);
      AssertCuda(error_code);
      for (unsigned int i=0; i<n_elements; ++i)
        out << cpu_val[i] << std::endl;
      out << std::flush;
      delete [] cpu_val;
      cpu_val = nullptr;


      AssertThrow (out, ExcIO());
      // reset output format
      out.flags (old_flags);
      out.precision(old_precision);
    }



    template <typename Number>
    std::size_t
    Vector<Number>::memory_consumption() const
    {
      std::size_t memory = sizeof(*this);
      memory += sizeof (Number) * static_cast<std::size_t>(n_elements);

      return memory;
    }



    // Explicit Instanationation
    template class Vector<float>;
    template class Vector<double>;
  }
}

DEAL_II_NAMESPACE_CLOSE

#endif

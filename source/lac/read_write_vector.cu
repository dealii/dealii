// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

#include <deal.II/lac/read_write_vector.h>
#include <deal.II/lac/read_write_vector.templates.h>

DEAL_II_NAMESPACE_OPEN


namespace LinearAlgebra
{
  template void
  ReadWriteVector<float>::import(
    const CUDAWrappers::Vector<float> &,
    VectorOperation::values,
    const std::shared_ptr<const Utilities::MPI::CommunicationPatternBase> &);
  template void
  ReadWriteVector<float>::import(
    const distributed::Vector<float, ::dealii::MemorySpace::CUDA> &,
    VectorOperation::values,
    const std::shared_ptr<const Utilities::MPI::CommunicationPatternBase> &);

  template void
  ReadWriteVector<double>::import(
    const CUDAWrappers::Vector<double> &,
    VectorOperation::values,
    const std::shared_ptr<const Utilities::MPI::CommunicationPatternBase> &);
  template void
  ReadWriteVector<double>::import(
    const distributed::Vector<double, ::dealii::MemorySpace::CUDA> &,
    VectorOperation::values,
    const std::shared_ptr<const Utilities::MPI::CommunicationPatternBase> &);
} // namespace LinearAlgebra

DEAL_II_NAMESPACE_CLOSE

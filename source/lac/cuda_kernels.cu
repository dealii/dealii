#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2018 - 2019 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

#include <deal.II/lac/cuda_kernels.templates.h>

DEAL_II_NAMESPACE_OPEN

namespace LinearAlgebra
{
  namespace CUDAWrappers
  {
    namespace kernel
    {
      /////////////////////////////////////////////////////////////////////////
      // Explicit instantiation                                              //
      /////////////////////////////////////////////////////////////////////////

      template __global__ void
      vec_scale<float>(float *, const float a, const size_type);
      template __global__ void
      vector_bin_op<float, Binop_Addition>(float *         v1,
                                           const float *   v2,
                                           const size_type N);
      template __global__ void
      vector_bin_op<float, Binop_Subtraction>(float *         v1,
                                              const float *   v2,
                                              const size_type N);
      template __global__ void
      masked_vector_bin_op<float, Binop_Addition>(const unsigned int *mask,
                                                  float *             v1,
                                                  const float *       v2,
                                                  const size_type     N);
      template __global__ void
      masked_vector_bin_op<float, Binop_Subtraction>(const unsigned int *mask,
                                                     float *             v1,
                                                     const float *       v2,
                                                     const size_type     N);
      template struct ElemSum<float>;
      template struct L1Norm<float>;
      template struct LInfty<float>;
      template __global__ void
      reduction<float, ElemSum<float>>(float *         result,
                                       const float *   v,
                                       const size_type N);
      template __global__ void
      reduction<float, L1Norm<float>>(float *         result,
                                      const float *   v,
                                      const size_type N);
      template __global__ void
      reduction<float, LInfty<float>>(float *         result,
                                      const float *   v,
                                      const size_type N);
      template struct DotProduct<float>;
      template __global__ void
      double_vector_reduction<float, DotProduct<float>>(float *         result,
                                                        const float *   v1,
                                                        const float *   v2,
                                                        const size_type N);
      template __global__ void
      vec_add<float>(float *val, const float, const size_type N);
      template __global__ void
      add_aV<float>(float *         val,
                    const float     a,
                    const float *   V_val,
                    const size_type N);
      template __global__ void
      add_aVbW<float>(float *         val,
                      const float     a,
                      const float *   V_val,
                      const float     b,
                      const float *   W_val,
                      const size_type N);
      template __global__ void
      sadd<float>(const float     s,
                  float *         val,
                  const float     a,
                  const float *   V_val,
                  const size_type N);
      template __global__ void
      sadd<float>(const float     s,
                  float *         val,
                  const float     a,
                  const float *   V_val,
                  const float     b,
                  const float *   W_val,
                  const size_type N);
      template __global__ void
      scale<float>(float *val, const float *V_val, const size_type N);
      template __global__ void
      equ<float>(float *         val,
                 const float     a,
                 const float *   V_val,
                 const size_type N);
      template __global__ void
      equ<float>(float *         val,
                 const float     a,
                 const float *   V_val,
                 const float     b,
                 const float *   W_val,
                 const size_type N);
      template __global__ void
      add_and_dot<float>(float *         res,
                         float *         v1,
                         const float *   v2,
                         const float *   v3,
                         const float     a,
                         const size_type N);
      template __global__ void
      set<float>(float *val, const float s, const size_type N);
      template __global__ void
      set_permutated<float, size_type>(const size_type *indices,
                                       float *          val,
                                       const float *    v,
                                       const size_type  N);
      template __global__ void
      gather<float, size_type>(float *          val,
                               const size_type *indices,
                               const float *    v,
                               const size_type  N);
      template __global__ void
      add_permutated<float>(const size_type *indices,
                            float *          val,
                            const float *    v,
                            const size_type  N);



      template __global__ void
      vec_scale<double>(double *, const double a, const size_type);
      template __global__ void
      vector_bin_op<double, Binop_Addition>(double *        v1,
                                            const double *  v2,
                                            const size_type N);
      template __global__ void
      vector_bin_op<double, Binop_Subtraction>(double *        v1,
                                               const double *  v2,
                                               const size_type N);
      template __global__ void
      masked_vector_bin_op<double, Binop_Addition>(const unsigned int *mask,
                                                   double *            v1,
                                                   const double *      v2,
                                                   const size_type     N);
      template __global__ void
      masked_vector_bin_op<double, Binop_Subtraction>(const unsigned int *mask,
                                                      double *            v1,
                                                      const double *      v2,
                                                      const size_type     N);
      template struct ElemSum<double>;
      template struct L1Norm<double>;
      template struct LInfty<double>;
      template __global__ void
      reduction<double, ElemSum<double>>(double *        result,
                                         const double *  v,
                                         const size_type N);
      template __global__ void
      reduction<double, L1Norm<double>>(double *        result,
                                        const double *  v,
                                        const size_type N);
      template __global__ void
      reduction<double, LInfty<double>>(double *        result,
                                        const double *  v,
                                        const size_type N);
      template struct DotProduct<double>;
      template __global__ void
      double_vector_reduction<double, DotProduct<double>>(double *      result,
                                                          const double *v1,
                                                          const double *v2,
                                                          const size_type N);
      template __global__ void
      vec_add<double>(double *val, const double, const size_type N);
      template __global__ void
      add_aV<double>(double *        val,
                     const double    a,
                     const double *  V_val,
                     const size_type N);
      template __global__ void
      add_aVbW<double>(double *        val,
                       const double    a,
                       const double *  V_val,
                       const double    b,
                       const double *  W_val,
                       const size_type N);
      template __global__ void
      sadd<double>(const double    s,
                   double *        val,
                   const double    a,
                   const double *  V_val,
                   const size_type N);
      template __global__ void
      sadd<double>(const double    s,
                   double *        val,
                   const double    a,
                   const double *  V_val,
                   const double    b,
                   const double *  W_val,
                   const size_type N);
      template __global__ void
      scale<double>(double *val, const double *V_val, const size_type N);
      template __global__ void
      equ<double>(double *        val,
                  const double    a,
                  const double *  V_val,
                  const size_type N);
      template __global__ void
      equ<double>(double *        val,
                  const double    a,
                  const double *  V_val,
                  const double    b,
                  const double *  W_val,
                  const size_type N);
      template __global__ void
      add_and_dot<double>(double *        res,
                          double *        v1,
                          const double *  v2,
                          const double *  v3,
                          const double    a,
                          const size_type N);
      template __global__ void
      set<double>(double *val, const double s, const size_type N);
      template __global__ void
      set_permutated<double, size_type>(const size_type *indices,
                                        double *         val,
                                        const double *   v,
                                        const size_type  N);
      template __global__ void
      gather<double, size_type>(double *         val,
                                const size_type *indices,
                                const double *   v,
                                const size_type  N);
      template __global__ void
      add_permutated<double>(const size_type *indices,
                             double *         val,
                             const double *   v,
                             const size_type  N);
    } // namespace kernel
  }   // namespace CUDAWrappers
} // namespace LinearAlgebra

DEAL_II_NAMESPACE_CLOSE

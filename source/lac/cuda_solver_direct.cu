// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the deal.II distribution.
//
// ---------------------------------------------------------------------

#include <deal.II/lac/cuda_solver_direct.h>

DEAL_II_NAMESPACE_OPEN

namespace CUDAWrappers
{
  namespace internal
  {
    void
    cusparsecsr2dense(hipsparseHandle_t cusparse_handle,
                      const SparseMatrix<float> &matrix,
                      float *dense_matrix_dev)
    {
      auto cusparse_matrix = matrix.get_cusparse_matrix();

      hipsparseStatus_t cusparse_error_code =
        hipsparseScsr2dense(
          cusparse_handle, matrix.m(), matrix.n(),
          std::get<3>(cusparse_matrix), std::get<0>(cusparse_matrix),
          std::get<2>(cusparse_matrix), std::get<1>(cusparse_matrix),
          dense_matrix_dev, matrix.m());
      AssertCusparse(cusparse_error_code);
    }



    void
    cusparsecsr2dense(hipsparseHandle_t cusparse_handle,
                      const SparseMatrix<double> &matrix,
                      double *dense_matrix_dev)
    {
      auto cusparse_matrix = matrix.get_cusparse_matrix();

      hipsparseStatus_t cusparse_error_code =
        hipsparseDcsr2dense(
          cusparse_handle, matrix.m(), matrix.n(),
          std::get<3>(cusparse_matrix), std::get<0>(cusparse_matrix),
          std::get<2>(cusparse_matrix), std::get<1>(cusparse_matrix),
          dense_matrix_dev, matrix.m());
      AssertCusparse(cusparse_error_code);
    }



    void
    cusolverDngetrf_buffer_size(hipsolverHandle_t cusolver_dn_handle, int m,
                                int n, float *dense_matrix_dev,
                                int &workspace_size)
    {
      hipsolverStatus_t cusolver_error_code = hipsolverDnSgetrf_bufferSize(
                                               cusolver_dn_handle, m, n, dense_matrix_dev, m, &workspace_size);
      AssertCusolver(cusolver_error_code);
    }



    void
    cusolverDngetrf_buffer_size(hipsolverHandle_t cusolver_dn_handle, int m,
                                int n, double *dense_matrix_dev,
                                int &workspace_size)
    {
      hipsolverStatus_t cusolver_error_code = hipsolverDnDgetrf_bufferSize(
                                               cusolver_dn_handle, m, n, dense_matrix_dev, m, &workspace_size);
      AssertCusolver(cusolver_error_code);
    }



    void
    cusolverDngetrf(hipsolverHandle_t cusolver_dn_handle, int m, int n,
                    float *dense_matrix_dev, float *workspace_dev,
                    int *pivot_dev, int *info_dev)
    {
      hipsolverStatus_t cusolver_error_code =
        hipsolverDnSgetrf(cusolver_dn_handle, m, n, dense_matrix_dev, m,
                         workspace_dev, pivot_dev, info_dev);
      AssertCusolver(cusolver_error_code);
    }



    void
    cusolverDngetrf(hipsolverHandle_t cusolver_dn_handle, int m, int n,
                    double *dense_matrix_dev, double *workspace_dev,
                    int *pivot_dev, int *info_dev)
    {
      hipsolverStatus_t cusolver_error_code =
        hipsolverDnDgetrf(cusolver_dn_handle, m, n, dense_matrix_dev, m,
                         workspace_dev, pivot_dev, info_dev);
      AssertCusolver(cusolver_error_code);
    }



    void
    cusolverDngetrs(hipsolverHandle_t cusolver_dn_handle, int m,
                    float *dense_matrix_dev, int *pivot_dev, float *b,
                    int *info_dev)
    {
      const int n_rhs = 1;
      hipsolverStatus_t cusolver_error_code =
        hipsolverDnSgetrs(cusolver_dn_handle, HIPBLAS_OP_N, m, n_rhs,
                         dense_matrix_dev, m, pivot_dev, b, m, info_dev);
      AssertCusolver(cusolver_error_code);
    }



    void
    cusolverDngetrs(hipsolverHandle_t cusolver_dn_handle, int m,
                    double *dense_matrix_dev, int *pivot_dev, double *b,
                    int *info_dev)
    {
      const int n_rhs = 1;
      hipsolverStatus_t cusolver_error_code =
        hipsolverDnDgetrs(cusolver_dn_handle, HIPBLAS_OP_N, m, n_rhs,
                         dense_matrix_dev, m, pivot_dev, b, m, info_dev);
      AssertCusolver(cusolver_error_code);
    }



    void
    cusolverSpcsrlsvluHost(hipsolverSpHandle_t cusolver_sp_handle,
                           const unsigned int n_rows, const unsigned int nnz,
                           hipsparseMatDescr_t descr, const float *val_host,
                           const int *row_ptr_host, const int *column_index_host,
                           const float *b_host, float *x_host)
    {
      int singularity = 0;
      hipsolverStatus_t cusolver_error_code = cusolverSpScsrlsvluHost(
                                               cusolver_sp_handle, n_rows, nnz, descr, val_host, row_ptr_host,
                                               column_index_host, b_host, 0., 1, x_host, &singularity);
      AssertCusolver(cusolver_error_code);
      Assert(singularity == -1, ExcMessage("Coarse matrix is singular"));
    }



    void
    cusolverSpcsrlsvluHost(hipsolverSpHandle_t cusolver_sp_handle,
                           const unsigned int n_rows, unsigned int nnz,
                           hipsparseMatDescr_t descr, const double *val_host,
                           const int *row_ptr_host, const int *column_index_host,
                           const double *b_host, double *x_host)
    {
      int singularity = 0;
      hipsolverStatus_t cusolver_error_code = cusolverSpDcsrlsvluHost(
                                               cusolver_sp_handle, n_rows, nnz, descr, val_host, row_ptr_host,
                                               column_index_host, b_host, 0., 1, x_host, &singularity);
      AssertCusolver(cusolver_error_code);
      Assert(singularity == -1, ExcMessage("Coarse matrix is singular"));
    }



    void
    cholesky_factorization(hipsolverSpHandle_t cusolver_sp_handle,
                           const SparseMatrix<float> &matrix,
                           const float *b, float *x)
    {
      auto cusparse_matrix = matrix.get_cusparse_matrix();
      int singularity = 0;

      hipsolverStatus_t cusolver_error_code = hipsolverSpScsrlsvchol(
                                               cusolver_sp_handle, matrix.m(), matrix.n_nonzero_elements(),
                                               std::get<3>(cusparse_matrix), std::get<0>(cusparse_matrix),
                                               std::get<2>(cusparse_matrix), std::get<1>(cusparse_matrix), b, 0., 0, x,
                                               &singularity);
      AssertCusolver(cusolver_error_code);
      Assert(singularity == -1, ExcMessage("Coarse matrix is not SPD"));
    }



    void
    cholesky_factorization(hipsolverSpHandle_t cusolver_sp_handle,
                           const SparseMatrix<double> &matrix,
                           const double *b, double *x)
    {
      auto cusparse_matrix = matrix.get_cusparse_matrix();
      int singularity = 0;

      hipsolverStatus_t cusolver_error_code = hipsolverSpDcsrlsvchol(
                                               cusolver_sp_handle, matrix.m(), matrix.n_nonzero_elements(),
                                               std::get<3>(cusparse_matrix), std::get<0>(cusparse_matrix),
                                               std::get<2>(cusparse_matrix), std::get<1>(cusparse_matrix), b, 0., 0, x,
                                               &singularity);
      AssertCusolver(cusolver_error_code);
      Assert(singularity == -1, ExcMessage("Coarse matrix is not SPD"));
    }



    template <typename Number>
    void
    lu_factorization(hipsparseHandle_t cusparse_handle,
                     hipsolverHandle_t cusolver_dn_handle,
                     const SparseMatrix<Number> &matrix,
                     const Number *b_dev, Number *x_dev)
    {
      // Change the format of the matrix from sparse to dense
      unsigned int const m = matrix.m();
      unsigned int const n = matrix.n();
      Assert(m == n, ExcMessage("The matrix is not square"));
      Number *dense_matrix_dev;
      Utilities::CUDA::malloc(dense_matrix_dev, m * n);

      // Change the format of matrix to dense
      internal::cusparsecsr2dense(cusparse_handle, matrix, dense_matrix_dev);

      // Create the working space
      int workspace_size = 0;
      internal::cusolverDngetrf_buffer_size(cusolver_dn_handle, m, n,
                                            dense_matrix_dev, workspace_size);
      Assert(workspace_size > 0, ExcMessage("No workspace was allocated"));
      Number *workspace_dev;
      Utilities::CUDA::malloc(workspace_dev, workspace_size);

      // LU factorization
      int *pivot_dev;
      Utilities::CUDA::malloc(pivot_dev, m);
      int *info_dev;
      Utilities::CUDA::malloc(info_dev, 1);

      internal::cusolverDngetrf(cusolver_dn_handle, m, n, dense_matrix_dev,
                                workspace_dev, pivot_dev, info_dev);

#ifdef DEBUG
      int info = 0;
      hipError_t cuda_error_code_debug =
        hipMemcpy(&info, info_dev, sizeof(int), hipMemcpyDeviceToHost);
      AssertCuda(cuda_error_code_debug);
      Assert(info == 0, ExcMessage("There was a problem during the LU factorization"));
#endif

      // Solve Ax = b
      hipError_t cuda_error_code = hipMemcpy(x_dev, b_dev, m * sizeof(Number),
                                               hipMemcpyDeviceToDevice);
      AssertCuda(cuda_error_code);
      internal::cusolverDngetrs(cusolver_dn_handle, m, dense_matrix_dev, pivot_dev,
                                x_dev, info_dev);
#ifdef DEBUG
      cuda_error_code =
        hipMemcpy(&info, info_dev, sizeof(int), hipMemcpyDeviceToHost);
      AssertCuda(cuda_error_code);
      Assert(info == 0, ExcMessage("There was a problem during the LU solve"));
#endif

      // Free the memory allocated
      Utilities::CUDA::free(dense_matrix_dev);
      Utilities::CUDA::free(workspace_dev);
      Utilities::CUDA::free(pivot_dev);
      Utilities::CUDA::free(info_dev);
    }



    template <typename Number>
    void
    lu_factorization(hipsolverSpHandle_t cusolver_sp_handle,
                     const SparseMatrix<Number> &matrix,
                     const Number *b_dev, Number *x_dev)
    {
      // cuSOLVER does not support LU factorization of sparse matrix on the device,
      // so we need to move everything to the host first and then back to the host.
      const unsigned int nnz = matrix.n_nonzero_elements();
      const unsigned int n_rows = matrix.m();
      std::vector<Number> val_host(nnz);
      std::vector<int> column_index_host(nnz);
      std::vector<int> row_ptr_host(n_rows + 1);
      auto cusparse_matrix = matrix.get_cusparse_matrix();
      Utilities::CUDA::copy_to_host(std::get<0>(cusparse_matrix), val_host);
      Utilities::CUDA::copy_to_host(std::get<1>(cusparse_matrix), column_index_host);
      Utilities::CUDA::copy_to_host(std::get<2>(cusparse_matrix), row_ptr_host);
      std::vector<Number> b_host(n_rows);
      Utilities::CUDA::copy_to_host(b_dev, b_host);
      std::vector<Number> x_host(n_rows);
      Utilities::CUDA::copy_to_host(x_dev, x_host);

      internal::cusolverSpcsrlsvluHost(
        cusolver_sp_handle, n_rows, nnz, std::get<3>(cusparse_matrix), val_host.data(),
        row_ptr_host.data(), column_index_host.data(), b_host.data(),
        x_host.data());

      // Move the solution back to the device
      Utilities::CUDA::copy_to_dev(x_host, x_dev);
    }
  }



  template <typename Number>
  SolverDirect<Number>::AdditionalData::
  AdditionalData(const std::string &solver_type)
    :
    solver_type(solver_type)
  {}



  template <typename Number>
  SolverDirect<Number>::SolverDirect(const Utilities::CUDA::Handle &handle,
                                     SolverControl  &cn,
                                     const AdditionalData &data)
    :
    cuda_handle(handle),
    solver_control(cn),
    additional_data(data.solver_type)
  {}



  template <typename Number>
  SolverControl &
  SolverDirect<Number>::control() const
  {
    return solver_control;
  }



  template <typename Number>
  void
  SolverDirect<Number>::solve(const SparseMatrix<Number> &A,
                              LinearAlgebra::CUDAWrappers::Vector<Number> &x,
                              const LinearAlgebra::CUDAWrappers::Vector<Number> &b)
  {
    if (additional_data.solver_type == "Cholesky")
      internal::cholesky_factorization(cuda_handle.cusolver_sp_handle, A,
                                       b.get_values(), x.get_values());
    else if (additional_data.solver_type == "LU_dense")
      internal::lu_factorization(cuda_handle.cusparse_handle,
                                 cuda_handle.cusolver_dn_handle, A,
                                 b.get_values(), x.get_values());
    else if (additional_data.solver_type == "LU_host")
      internal::lu_factorization(cuda_handle.cusolver_sp_handle, A,
                                 b.get_values(), x.get_values());
    else
      AssertThrow(false, ExcMessage("The provided solver name " +
                                    additional_data.solver_type + " is invalid."));

    // Force the SolverControl object to report convergence
    solver_control.check(0, 0);
  }


  // Explicit Instanationation
  template class SolverDirect<float>;
  template class SolverDirect<double>;
}

DEAL_II_NAMESPACE_CLOSE

#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2018 - 2020 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

#include <deal.II/base/cuda_size.h>
#include <deal.II/base/exceptions.h>

#include <deal.II/lac/cuda_atomic.h>
#include <deal.II/lac/cuda_sparse_matrix.h>

#ifdef DEAL_II_WITH_CUDA

#  include <hipsparse.h>

DEAL_II_NAMESPACE_OPEN

namespace CUDAWrappers
{
  namespace internal
  {
    template <typename Number>
    __global__ void
    scale(Number *                                       val,
          const Number                                   a,
          const typename SparseMatrix<Number>::size_type N)
    {
      const typename SparseMatrix<Number>::size_type idx =
        threadIdx.x + blockIdx.x * blockDim.x;
      if (idx < N)
        val[idx] *= a;
    }



    void
    csrmv(hipsparseHandle_t         handle,
          bool                     transpose,
          int                      m,
          int                      n,
          int                      nnz,
          const hipsparseMatDescr_t descr,
          const float *            A_val_dev,
          const int *              A_row_ptr_dev,
          const int *              A_column_index_dev,
          const float *            x,
          bool                     add,
          float *                  y)
    {
      float               alpha = 1.;
      float               beta  = add ? 1. : 0.;
      hipsparseOperation_t cusparse_operation =
        transpose ? HIPSPARSE_OPERATION_TRANSPOSE :
                    HIPSPARSE_OPERATION_NON_TRANSPOSE;

      // This function performs y = alpha*op(A)*x + beta*y
      hipsparseStatus_t error_code = hipsparseScsrmv(handle,
                                                   cusparse_operation,
                                                   m,
                                                   n,
                                                   nnz,
                                                   &alpha,
                                                   descr,
                                                   A_val_dev,
                                                   A_row_ptr_dev,
                                                   A_column_index_dev,
                                                   x,
                                                   &beta,
                                                   y);
      AssertCusparse(error_code);
    }



    void
    csrmv(hipsparseHandle_t         handle,
          bool                     transpose,
          int                      m,
          int                      n,
          int                      nnz,
          const hipsparseMatDescr_t descr,
          const double *           A_val_dev,
          const int *              A_row_ptr_dev,
          const int *              A_column_index_dev,
          const double *           x,
          bool                     add,
          double *                 y)
    {
      double              alpha = 1.;
      double              beta  = add ? 1. : 0.;
      hipsparseOperation_t cusparse_operation =
        transpose ? HIPSPARSE_OPERATION_TRANSPOSE :
                    HIPSPARSE_OPERATION_NON_TRANSPOSE;

      // This function performs y = alpha*op(A)*x + beta*y
      hipsparseStatus_t error_code = hipsparseDcsrmv(handle,
                                                   cusparse_operation,
                                                   m,
                                                   n,
                                                   nnz,
                                                   &alpha,
                                                   descr,
                                                   A_val_dev,
                                                   A_row_ptr_dev,
                                                   A_column_index_dev,
                                                   x,
                                                   &beta,
                                                   y);
      AssertCusparse(error_code);
    }



    template <typename Number>
    __global__ void
    l1_norm(const typename SparseMatrix<Number>::size_type n_rows,
            const Number *                                 val_dev,
            const int *                                    column_index_dev,
            const int *                                    row_ptr_dev,
            Number *                                       sums)
    {
      const typename SparseMatrix<Number>::size_type row =
        threadIdx.x + blockIdx.x * blockDim.x;

      if (row < n_rows)
        {
          for (int j = row_ptr_dev[row]; j < row_ptr_dev[row + 1]; ++j)
            atomicAdd(&sums[column_index_dev[j]], abs(val_dev[j]));
        }
    }



    template <typename Number>
    __global__ void
    linfty_norm(const typename SparseMatrix<Number>::size_type n_rows,
                const Number *                                 val_dev,
                const int *                                    column_index_dev,
                const int *                                    row_ptr_dev,
                Number *                                       sums)
    {
      const typename SparseMatrix<Number>::size_type row =
        threadIdx.x + blockIdx.x * blockDim.x;

      if (row < n_rows)
        {
          sums[row] = (Number)0.;
          for (int j = row_ptr_dev[row]; j < row_ptr_dev[row + 1]; ++j)
            sums[row] += abs(val_dev[j]);
        }
    }
  } // namespace internal



  template <typename Number>
  SparseMatrix<Number>::SparseMatrix()
    : nnz(0)
    , n_rows(0)
    , val_dev(nullptr, Utilities::CUDA::delete_device_data<Number>)
    , column_index_dev(nullptr, Utilities::CUDA::delete_device_data<int>)
    , row_ptr_dev(nullptr, Utilities::CUDA::delete_device_data<int>)
    , descr(nullptr)
  {}



  template <typename Number>
  SparseMatrix<Number>::SparseMatrix(
    Utilities::CUDA::Handle &             handle,
    const ::dealii::SparseMatrix<Number> &sparse_matrix_host)
    : val_dev(nullptr, Utilities::CUDA::delete_device_data<Number>)
    , column_index_dev(nullptr, Utilities::CUDA::delete_device_data<int>)
    , row_ptr_dev(nullptr, Utilities::CUDA::delete_device_data<int>)
    , descr(nullptr)
  {
    reinit(handle, sparse_matrix_host);
  }



  template <typename Number>
  SparseMatrix<Number>::SparseMatrix(CUDAWrappers::SparseMatrix<Number> &&other)
    : cusparse_handle(other.cusparse_handle)
    , nnz(other.nnz)
    , n_rows(other.n_rows)
    , n_cols(other.n_cols)
    , val_dev(std::move(other.val_dev))
    , column_index_dev(std::move(other.column_index_dev))
    , row_ptr_dev(std::move(other.row_ptr_dev))
    , descr(other.descr)
  {
    other.nnz    = 0;
    other.n_rows = 0;
    other.n_cols = 0;
    other.descr  = nullptr;
  }



  template <typename Number>
  SparseMatrix<Number>::~SparseMatrix<Number>()
  {
    if (descr != nullptr)
      {
        const hipsparseStatus_t cusparse_error_code =
          hipsparseDestroyMatDescr(descr);
        AssertNothrowCusparse(cusparse_error_code);
        descr = nullptr;
      }

    nnz    = 0;
    n_rows = 0;
  }



  template <typename Number>
  SparseMatrix<Number> &
  SparseMatrix<Number>::operator=(SparseMatrix<Number> &&other)
  {
    cusparse_handle  = other.cusparse_handle;
    nnz              = other.nnz;
    n_rows           = other.n_rows;
    n_cols           = other.n_cols;
    val_dev          = std::move(other.val_dev);
    column_index_dev = std::move(other.column_index_dev);
    row_ptr_dev      = std::move(other.row_ptr_dev);
    descr            = other.descr;

    other.nnz    = 0;
    other.n_rows = 0;
    other.n_cols = 0;
    other.descr  = nullptr;

    return *this;
  }



  template <typename Number>
  void
  SparseMatrix<Number>::reinit(
    Utilities::CUDA::Handle &             handle,
    const ::dealii::SparseMatrix<Number> &sparse_matrix_host)
  {
    cusparse_handle                  = handle.cusparse_handle;
    nnz                              = sparse_matrix_host.n_nonzero_elements();
    n_rows                           = sparse_matrix_host.m();
    n_cols                           = sparse_matrix_host.n();
    unsigned int const  row_ptr_size = n_rows + 1;
    std::vector<Number> val;
    val.reserve(nnz);
    std::vector<int> column_index;
    column_index.reserve(nnz);
    std::vector<int> row_ptr(row_ptr_size, 0);

    // dealii::SparseMatrix stores the diagonal first in each row so we need to
    // do some reordering
    for (int row = 0; row < n_rows; ++row)
      {
        auto         p_end   = sparse_matrix_host.end(row);
        unsigned int counter = 0;
        for (auto p = sparse_matrix_host.begin(row); p != p_end; ++p)
          {
            val.emplace_back(p->value());
            column_index.emplace_back(p->column());
            ++counter;
          }
        row_ptr[row + 1] = row_ptr[row] + counter;

        // Sort the elements in the row
        unsigned int const offset     = row_ptr[row];
        int const          diag_index = column_index[offset];
        Number             diag_elem  = sparse_matrix_host.diag_element(row);
        unsigned int       pos        = 1;
        while ((column_index[offset + pos] < row) && (pos < counter))
          {
            val[offset + pos - 1]          = val[offset + pos];
            column_index[offset + pos - 1] = column_index[offset + pos];
            ++pos;
          }
        val[offset + pos - 1]          = diag_elem;
        column_index[offset + pos - 1] = diag_index;
      }

    // Copy the elements to the gpu
    val_dev.reset(Utilities::CUDA::allocate_device_data<Number>(nnz));
    hipError_t error_code = hipMemcpy(val_dev.get(),
                                        val.data(),
                                        nnz * sizeof(Number),
                                        hipMemcpyHostToDevice);
    AssertCuda(error_code);

    // Copy the column indices to the gpu
    column_index_dev.reset(Utilities::CUDA::allocate_device_data<int>(nnz));
    AssertCuda(error_code);
    error_code = hipMemcpy(column_index_dev.get(),
                            column_index.data(),
                            nnz * sizeof(int),
                            hipMemcpyHostToDevice);
    AssertCuda(error_code);

    // Copy the row pointer to the gpu
    row_ptr_dev.reset(Utilities::CUDA::allocate_device_data<int>(row_ptr_size));
    AssertCuda(error_code);
    error_code = hipMemcpy(row_ptr_dev.get(),
                            row_ptr.data(),
                            row_ptr_size * sizeof(int),
                            hipMemcpyHostToDevice);
    AssertCuda(error_code);

    // Create the matrix descriptor
    hipsparseStatus_t cusparse_error_code = hipsparseCreateMatDescr(&descr);
    AssertCusparse(cusparse_error_code);
    cusparse_error_code =
      hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    AssertCusparse(cusparse_error_code);
    cusparse_error_code =
      hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    AssertCusparse(cusparse_error_code);
  }



  template <typename Number>
  SparseMatrix<Number> &
  SparseMatrix<Number>::operator*=(const Number factor)
  {
    AssertIsFinite(factor);
    const int n_blocks = 1 + (nnz - 1) / block_size;
    internal::scale<Number>
      <<<n_blocks, block_size>>>(val_dev.get(), factor, nnz);
    AssertCudaKernel();

    return *this;
  }



  template <typename Number>
  SparseMatrix<Number> &
  SparseMatrix<Number>::operator/=(const Number factor)
  {
    AssertIsFinite(factor);
    Assert(factor != Number(0.), ExcZero());
    const int n_blocks = 1 + (nnz - 1) / block_size;
    internal::scale<Number>
      <<<n_blocks, block_size>>>(val_dev.get(), 1. / factor, nnz);
    AssertCudaKernel();

    return *this;
  }



  template <typename Number>
  void
  SparseMatrix<Number>::vmult(
    LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
    const LinearAlgebra::CUDAWrappers::Vector<Number> &src) const
  {
    internal::csrmv(cusparse_handle,
                    false,
                    n_rows,
                    n_cols,
                    nnz,
                    descr,
                    val_dev.get(),
                    row_ptr_dev.get(),
                    column_index_dev.get(),
                    src.get_values(),
                    false,
                    dst.get_values());
  }



  template <typename Number>
  void
  SparseMatrix<Number>::Tvmult(
    LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
    const LinearAlgebra::CUDAWrappers::Vector<Number> &src) const
  {
    internal::csrmv(cusparse_handle,
                    true,
                    n_rows,
                    n_cols,
                    nnz,
                    descr,
                    val_dev.get(),
                    row_ptr_dev.get(),
                    column_index_dev.get(),
                    src.get_values(),
                    false,
                    dst.get_values());
  }



  template <typename Number>
  void
  SparseMatrix<Number>::vmult_add(
    LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
    const LinearAlgebra::CUDAWrappers::Vector<Number> &src) const
  {
    internal::csrmv(cusparse_handle,
                    false,
                    n_rows,
                    n_cols,
                    nnz,
                    descr,
                    val_dev.get(),
                    row_ptr_dev.get(),
                    column_index_dev.get(),
                    src.get_values(),
                    true,
                    dst.get_values());
  }



  template <typename Number>
  void
  SparseMatrix<Number>::Tvmult_add(
    LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
    const LinearAlgebra::CUDAWrappers::Vector<Number> &src) const
  {
    internal::csrmv(cusparse_handle,
                    true,
                    n_rows,
                    n_cols,
                    nnz,
                    descr,
                    val_dev.get(),
                    row_ptr_dev.get(),
                    column_index_dev.get(),
                    src.get_values(),
                    true,
                    dst.get_values());
  }



  template <typename Number>
  Number
  SparseMatrix<Number>::matrix_norm_square(
    const LinearAlgebra::CUDAWrappers::Vector<Number> &v) const
  {
    LinearAlgebra::CUDAWrappers::Vector<Number> tmp = v;
    vmult(tmp, v);

    return v * tmp;
  }



  template <typename Number>
  Number
  SparseMatrix<Number>::matrix_scalar_product(
    const LinearAlgebra::CUDAWrappers::Vector<Number> &u,
    const LinearAlgebra::CUDAWrappers::Vector<Number> &v) const
  {
    LinearAlgebra::CUDAWrappers::Vector<Number> tmp = v;
    vmult(tmp, v);

    return u * tmp;
  }



  template <typename Number>
  Number
  SparseMatrix<Number>::residual(
    LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
    const LinearAlgebra::CUDAWrappers::Vector<Number> &x,
    const LinearAlgebra::CUDAWrappers::Vector<Number> &b) const
  {
    vmult(dst, x);
    dst.sadd(-1., 1., b);

    return dst.l2_norm();
  }



  template <typename Number>
  Number
  SparseMatrix<Number>::l1_norm() const
  {
    LinearAlgebra::CUDAWrappers::Vector<real_type> column_sums(n_cols);
    const int n_blocks = 1 + (nnz - 1) / block_size;
    internal::l1_norm<Number>
      <<<n_blocks, block_size>>>(n_rows,
                                 val_dev.get(),
                                 column_index_dev.get(),
                                 row_ptr_dev.get(),
                                 column_sums.get_values());
    AssertCudaKernel();

    return column_sums.linfty_norm();
  }



  template <typename Number>
  Number
  SparseMatrix<Number>::linfty_norm() const
  {
    LinearAlgebra::CUDAWrappers::Vector<real_type> row_sums(n_rows);
    const int n_blocks = 1 + (nnz - 1) / block_size;
    internal::linfty_norm<Number>
      <<<n_blocks, block_size>>>(n_rows,
                                 val_dev.get(),
                                 column_index_dev.get(),
                                 row_ptr_dev.get(),
                                 row_sums.get_values());
    AssertCudaKernel();

    return row_sums.linfty_norm();
  }



  template <typename Number>
  Number
  SparseMatrix<Number>::frobenius_norm() const
  {
    LinearAlgebra::CUDAWrappers::Vector<real_type> matrix_values(nnz);
    hipError_t cuda_error = hipMemcpy(matrix_values.get_values(),
                                        val_dev.get(),
                                        nnz * sizeof(Number),
                                        hipMemcpyDeviceToDevice);

    return matrix_values.l2_norm();
  }



  template <typename Number>
  std::tuple<Number *, int *, int *, hipsparseMatDescr_t>
  SparseMatrix<Number>::get_cusparse_matrix() const
  {
    return std::make_tuple(val_dev.get(),
                           column_index_dev.get(),
                           row_ptr_dev.get(),
                           descr);
  }



  template class SparseMatrix<float>;
  template class SparseMatrix<double>;
} // namespace CUDAWrappers
DEAL_II_NAMESPACE_CLOSE

#endif

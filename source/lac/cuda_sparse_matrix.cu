#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (C) 2018 - 2020 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------

#include <deal.II/base/cuda_size.h>
#include <deal.II/base/exceptions.h>

#include <deal.II/lac/cuda_atomic.h>
#include <deal.II/lac/cuda_sparse_matrix.h>

#ifdef DEAL_II_WITH_CUDA

#  include <hipsparse.h>

DEAL_II_NAMESPACE_OPEN

namespace CUDAWrappers
{
  namespace internal
  {
    template <typename Number>
    __global__ void
    scale(Number *                                       val,
          const Number                                   a,
          const typename SparseMatrix<Number>::size_type N)
    {
      const typename SparseMatrix<Number>::size_type idx =
        threadIdx.x + blockIdx.x * blockDim.x;
      if (idx < N)
        val[idx] *= a;
    }



    void
    create_sp_mat_descr(int                   m,
                        int                   n,
                        int                   nnz,
                        const float *         A_val_dev,
                        const int *           A_row_ptr_dev,
                        const int *           A_column_index_dev,
                        hipsparseSpMatDescr_t &sp_descr)
    {
      hipsparseStatus_t error_code = hipsparseCreateCsr(
        &sp_descr,
        m,
        n,
        nnz,
        reinterpret_cast<void *>(const_cast<int *>(A_row_ptr_dev)),
        reinterpret_cast<void *>(const_cast<int *>(A_column_index_dev)),
        reinterpret_cast<void *>(const_cast<float *>(A_val_dev)),
        HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO,
        HIP_R_32F);
      AssertCusparse(error_code);
    }



    void
    create_sp_mat_descr(int                   m,
                        int                   n,
                        int                   nnz,
                        const double *        A_val_dev,
                        const int *           A_row_ptr_dev,
                        const int *           A_column_index_dev,
                        hipsparseSpMatDescr_t &sp_descr)
    {
      hipsparseStatus_t error_code = hipsparseCreateCsr(
        &sp_descr,
        m,
        n,
        nnz,
        reinterpret_cast<void *>(const_cast<int *>(A_row_ptr_dev)),
        reinterpret_cast<void *>(const_cast<int *>(A_column_index_dev)),
        reinterpret_cast<void *>(const_cast<double *>(A_val_dev)),
        HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO,
        HIP_R_64F);
      AssertCusparse(error_code);
    }



    void
    csrmv(hipsparseHandle_t           handle,
          bool                       transpose,
          int                        m,
          int                        n,
          const hipsparseSpMatDescr_t sp_descr,
          const float *              x,
          bool                       add,
          float *                    y)
    {
      float               alpha = 1.;
      float               beta  = add ? 1. : 0.;
      hipsparseOperation_t cusparse_operation =
        transpose ? HIPSPARSE_OPERATION_TRANSPOSE :
                    HIPSPARSE_OPERATION_NON_TRANSPOSE;

      // Move the data to cuSPARSE data type
      hipsparseDnVecDescr_t x_cuvec;
      hipsparseStatus_t     error_code =
        hipsparseCreateDnVec(&x_cuvec,
                            n,
                            reinterpret_cast<void *>(const_cast<float *>(x)),
                            HIP_R_32F);
      AssertCusparse(error_code);

      hipsparseDnVecDescr_t y_cuvec;
      error_code =
        hipsparseCreateDnVec(&y_cuvec,
                            m,
                            reinterpret_cast<void *>(const_cast<float *>(y)),
                            HIP_R_32F);
      AssertCusparse(error_code);

      // This function performs y = alpha*op(A)*x + beta*y
      size_t buffer_size = 0;
      error_code         = hipsparseSpMV_bufferSize(handle,
                                           cusparse_operation,
                                           &alpha,
                                           sp_descr,
                                           x_cuvec,
                                           &beta,
                                           y_cuvec,
                                           HIP_R_32F,
                                           HIPSPARSE_MV_ALG_DEFAULT,
                                           &buffer_size);

      void *      buffer          = nullptr;
      hipError_t cuda_error_code = hipMalloc(&buffer, buffer_size);
      AssertCuda(cuda_error_code);

      // execute SpMV
      error_code = hipsparseSpMV(handle,
                                cusparse_operation,
                                &alpha,
                                sp_descr,
                                x_cuvec,
                                &beta,
                                y_cuvec,
                                HIP_R_32F,
                                HIPSPARSE_MV_ALG_DEFAULT,
                                buffer);
      AssertCusparse(error_code);

      cuda_error_code = hipFree(buffer);
      AssertCuda(cuda_error_code);
      error_code = hipsparseDestroyDnVec(x_cuvec);
      AssertCusparse(error_code);
      error_code = hipsparseDestroyDnVec(y_cuvec);
      AssertCusparse(error_code);
    }



    void
    csrmv(hipsparseHandle_t           handle,
          bool                       transpose,
          int                        m,
          int                        n,
          const hipsparseSpMatDescr_t sp_descr,
          const double *             x,
          bool                       add,
          double *                   y)
    {
      double              alpha = 1.;
      double              beta  = add ? 1. : 0.;
      hipsparseOperation_t cusparse_operation =
        transpose ? HIPSPARSE_OPERATION_TRANSPOSE :
                    HIPSPARSE_OPERATION_NON_TRANSPOSE;

      // Move the data to cuSPARSE data type
      hipsparseDnVecDescr_t x_cuvec;
      hipsparseStatus_t     error_code =
        hipsparseCreateDnVec(&x_cuvec,
                            n,
                            reinterpret_cast<void *>(const_cast<double *>(x)),
                            HIP_R_64F);
      AssertCusparse(error_code);

      hipsparseDnVecDescr_t y_cuvec;
      error_code =
        hipsparseCreateDnVec(&y_cuvec,
                            m,
                            reinterpret_cast<void *>(const_cast<double *>(y)),
                            HIP_R_64F);
      AssertCusparse(error_code);

      // This function performs y = alpha*op(A)*x + beta*y
      size_t buffer_size = 0;
      error_code         = hipsparseSpMV_bufferSize(handle,
                                           cusparse_operation,
                                           &alpha,
                                           sp_descr,
                                           x_cuvec,
                                           &beta,
                                           y_cuvec,
                                           HIP_R_64F,
                                           HIPSPARSE_MV_ALG_DEFAULT,
                                           &buffer_size);

      void *      buffer          = nullptr;
      hipError_t cuda_error_code = hipMalloc(&buffer, buffer_size);
      AssertCuda(cuda_error_code);

      // execute SpMV
      error_code = hipsparseSpMV(handle,
                                cusparse_operation,
                                &alpha,
                                sp_descr,
                                x_cuvec,
                                &beta,
                                y_cuvec,
                                HIP_R_64F,
                                HIPSPARSE_MV_ALG_DEFAULT,
                                buffer);
      AssertCusparse(error_code);

      cuda_error_code = hipFree(buffer);
      AssertCuda(cuda_error_code);
      error_code = hipsparseDestroyDnVec(x_cuvec);
      AssertCusparse(error_code);
      error_code = hipsparseDestroyDnVec(y_cuvec);
      AssertCusparse(error_code);
    }



    template <typename Number>
    __global__ void
    l1_norm(const typename SparseMatrix<Number>::size_type n_rows,
            const Number *                                 val_dev,
            const int *                                    column_index_dev,
            const int *                                    row_ptr_dev,
            Number *                                       sums)
    {
      const typename SparseMatrix<Number>::size_type row =
        threadIdx.x + blockIdx.x * blockDim.x;

      if (row < n_rows)
        {
          for (int j = row_ptr_dev[row]; j < row_ptr_dev[row + 1]; ++j)
            atomicAdd(&sums[column_index_dev[j]], abs(val_dev[j]));
        }
    }



    template <typename Number>
    __global__ void
    linfty_norm(const typename SparseMatrix<Number>::size_type n_rows,
                const Number *                                 val_dev,
                const int *                                    column_index_dev,
                const int *                                    row_ptr_dev,
                Number *                                       sums)
    {
      const typename SparseMatrix<Number>::size_type row =
        threadIdx.x + blockIdx.x * blockDim.x;

      if (row < n_rows)
        {
          sums[row] = (Number)0.;
          for (int j = row_ptr_dev[row]; j < row_ptr_dev[row + 1]; ++j)
            sums[row] += abs(val_dev[j]);
        }
    }
  } // namespace internal



  template <typename Number>
  SparseMatrix<Number>::SparseMatrix()
    : nnz(0)
    , n_rows(0)
    , val_dev(nullptr, Utilities::CUDA::delete_device_data<Number>)
    , column_index_dev(nullptr, Utilities::CUDA::delete_device_data<int>)
    , row_ptr_dev(nullptr, Utilities::CUDA::delete_device_data<int>)
    , descr(nullptr)
    , sp_descr(nullptr)
  {}



  template <typename Number>
  SparseMatrix<Number>::SparseMatrix(
    Utilities::CUDA::Handle &             handle,
    const ::dealii::SparseMatrix<Number> &sparse_matrix_host)
    : val_dev(nullptr, Utilities::CUDA::delete_device_data<Number>)
    , column_index_dev(nullptr, Utilities::CUDA::delete_device_data<int>)
    , row_ptr_dev(nullptr, Utilities::CUDA::delete_device_data<int>)
    , descr(nullptr)
    , sp_descr(nullptr)
  {
    reinit(handle, sparse_matrix_host);
  }



  template <typename Number>
  SparseMatrix<Number>::SparseMatrix(CUDAWrappers::SparseMatrix<Number> &&other)
    : cusparse_handle(other.cusparse_handle)
    , nnz(other.nnz)
    , n_rows(other.n_rows)
    , n_cols(other.n_cols)
    , val_dev(std::move(other.val_dev))
    , column_index_dev(std::move(other.column_index_dev))
    , row_ptr_dev(std::move(other.row_ptr_dev))
    , descr(other.descr)
    , sp_descr(other.sp_descr)
  {
    other.nnz      = 0;
    other.n_rows   = 0;
    other.n_cols   = 0;
    other.descr    = nullptr;
    other.sp_descr = nullptr;
  }



  template <typename Number>
  SparseMatrix<Number>::~SparseMatrix<Number>()
  {
    if (descr != nullptr)
      {
        const hipsparseStatus_t cusparse_error_code =
          hipsparseDestroyMatDescr(descr);
        AssertNothrowCusparse(cusparse_error_code);
        descr = nullptr;
      }

    if (sp_descr != nullptr)
      {
        const hipsparseStatus_t cusparse_error_code =
          hipsparseDestroySpMat(sp_descr);
        AssertNothrowCusparse(cusparse_error_code);
        sp_descr = nullptr;
      }

    nnz    = 0;
    n_rows = 0;
  }



  template <typename Number>
  SparseMatrix<Number> &
  SparseMatrix<Number>::operator=(SparseMatrix<Number> &&other)
  {
    cusparse_handle  = other.cusparse_handle;
    nnz              = other.nnz;
    n_rows           = other.n_rows;
    n_cols           = other.n_cols;
    val_dev          = std::move(other.val_dev);
    column_index_dev = std::move(other.column_index_dev);
    row_ptr_dev      = std::move(other.row_ptr_dev);
    descr            = other.descr;
    sp_descr         = other.sp_descr;

    other.nnz      = 0;
    other.n_rows   = 0;
    other.n_cols   = 0;
    other.descr    = nullptr;
    other.sp_descr = nullptr;

    return *this;
  }



  template <typename Number>
  void
  SparseMatrix<Number>::reinit(
    Utilities::CUDA::Handle &             handle,
    const ::dealii::SparseMatrix<Number> &sparse_matrix_host)
  {
    cusparse_handle                  = handle.cusparse_handle;
    nnz                              = sparse_matrix_host.n_nonzero_elements();
    n_rows                           = sparse_matrix_host.m();
    n_cols                           = sparse_matrix_host.n();
    unsigned int const  row_ptr_size = n_rows + 1;
    std::vector<Number> val;
    val.reserve(nnz);
    std::vector<int> column_index;
    column_index.reserve(nnz);
    std::vector<int> row_ptr(row_ptr_size, 0);

    // dealii::SparseMatrix stores the diagonal first in each row so we need to
    // do some reordering
    for (int row = 0; row < n_rows; ++row)
      {
        auto         p_end   = sparse_matrix_host.end(row);
        unsigned int counter = 0;
        for (auto p = sparse_matrix_host.begin(row); p != p_end; ++p)
          {
            val.emplace_back(p->value());
            column_index.emplace_back(p->column());
            ++counter;
          }
        row_ptr[row + 1] = row_ptr[row] + counter;

        // Sort the elements in the row
        unsigned int const offset     = row_ptr[row];
        int const          diag_index = column_index[offset];
        Number             diag_elem  = sparse_matrix_host.diag_element(row);
        unsigned int       pos        = 1;
        while ((column_index[offset + pos] < row) && (pos < counter))
          {
            val[offset + pos - 1]          = val[offset + pos];
            column_index[offset + pos - 1] = column_index[offset + pos];
            ++pos;
          }
        val[offset + pos - 1]          = diag_elem;
        column_index[offset + pos - 1] = diag_index;
      }

    // Copy the elements to the gpu
    val_dev.reset(Utilities::CUDA::allocate_device_data<Number>(nnz));
    hipError_t error_code = hipMemcpy(val_dev.get(),
                                        val.data(),
                                        nnz * sizeof(Number),
                                        hipMemcpyHostToDevice);
    AssertCuda(error_code);

    // Copy the column indices to the gpu
    column_index_dev.reset(Utilities::CUDA::allocate_device_data<int>(nnz));
    AssertCuda(error_code);
    error_code = hipMemcpy(column_index_dev.get(),
                            column_index.data(),
                            nnz * sizeof(int),
                            hipMemcpyHostToDevice);
    AssertCuda(error_code);

    // Copy the row pointer to the gpu
    row_ptr_dev.reset(Utilities::CUDA::allocate_device_data<int>(row_ptr_size));
    AssertCuda(error_code);
    error_code = hipMemcpy(row_ptr_dev.get(),
                            row_ptr.data(),
                            row_ptr_size * sizeof(int),
                            hipMemcpyHostToDevice);
    AssertCuda(error_code);

    // Create the matrix descriptor
    hipsparseStatus_t cusparse_error_code = hipsparseCreateMatDescr(&descr);
    AssertCusparse(cusparse_error_code);
    cusparse_error_code =
      hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    AssertCusparse(cusparse_error_code);
    cusparse_error_code =
      hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    AssertCusparse(cusparse_error_code);

    // Create the sparse matrix descriptor
    internal::create_sp_mat_descr(n_rows,
                                  n_cols,
                                  nnz,
                                  val_dev.get(),
                                  row_ptr_dev.get(),
                                  column_index_dev.get(),
                                  sp_descr);
  }



  template <typename Number>
  SparseMatrix<Number> &
  SparseMatrix<Number>::operator*=(const Number factor)
  {
    AssertIsFinite(factor);
    const int n_blocks = 1 + (nnz - 1) / block_size;
    internal::scale<Number>
      <<<n_blocks, block_size>>>(val_dev.get(), factor, nnz);
    AssertCudaKernel();

    return *this;
  }



  template <typename Number>
  SparseMatrix<Number> &
  SparseMatrix<Number>::operator/=(const Number factor)
  {
    AssertIsFinite(factor);
    Assert(factor != Number(0.), ExcZero());
    const int n_blocks = 1 + (nnz - 1) / block_size;
    internal::scale<Number>
      <<<n_blocks, block_size>>>(val_dev.get(), 1. / factor, nnz);
    AssertCudaKernel();

    return *this;
  }



  template <typename Number>
  void
  SparseMatrix<Number>::vmult(
    LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
    const LinearAlgebra::CUDAWrappers::Vector<Number> &src) const
  {
    internal::csrmv(cusparse_handle,
                    false,
                    n_rows,
                    n_cols,
                    sp_descr,
                    src.get_values(),
                    false,
                    dst.get_values());
  }



  template <typename Number>
  void
  SparseMatrix<Number>::Tvmult(
    LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
    const LinearAlgebra::CUDAWrappers::Vector<Number> &src) const
  {
    internal::csrmv(cusparse_handle,
                    true,
                    n_rows,
                    n_cols,
                    sp_descr,
                    src.get_values(),
                    false,
                    dst.get_values());
  }



  template <typename Number>
  void
  SparseMatrix<Number>::vmult_add(
    LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
    const LinearAlgebra::CUDAWrappers::Vector<Number> &src) const
  {
    internal::csrmv(cusparse_handle,
                    false,
                    n_rows,
                    n_cols,
                    sp_descr,
                    src.get_values(),
                    true,
                    dst.get_values());
  }



  template <typename Number>
  void
  SparseMatrix<Number>::Tvmult_add(
    LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
    const LinearAlgebra::CUDAWrappers::Vector<Number> &src) const
  {
    internal::csrmv(cusparse_handle,
                    true,
                    n_rows,
                    n_cols,
                    sp_descr,
                    src.get_values(),
                    true,
                    dst.get_values());
  }



  template <typename Number>
  Number
  SparseMatrix<Number>::matrix_norm_square(
    const LinearAlgebra::CUDAWrappers::Vector<Number> &v) const
  {
    LinearAlgebra::CUDAWrappers::Vector<Number> tmp = v;
    vmult(tmp, v);

    return v * tmp;
  }



  template <typename Number>
  Number
  SparseMatrix<Number>::matrix_scalar_product(
    const LinearAlgebra::CUDAWrappers::Vector<Number> &u,
    const LinearAlgebra::CUDAWrappers::Vector<Number> &v) const
  {
    LinearAlgebra::CUDAWrappers::Vector<Number> tmp = v;
    vmult(tmp, v);

    return u * tmp;
  }



  template <typename Number>
  Number
  SparseMatrix<Number>::residual(
    LinearAlgebra::CUDAWrappers::Vector<Number> &      dst,
    const LinearAlgebra::CUDAWrappers::Vector<Number> &x,
    const LinearAlgebra::CUDAWrappers::Vector<Number> &b) const
  {
    vmult(dst, x);
    dst.sadd(-1., 1., b);

    return dst.l2_norm();
  }



  template <typename Number>
  Number
  SparseMatrix<Number>::l1_norm() const
  {
    LinearAlgebra::CUDAWrappers::Vector<real_type> column_sums(n_cols);
    const int n_blocks = 1 + (nnz - 1) / block_size;
    internal::l1_norm<Number>
      <<<n_blocks, block_size>>>(n_rows,
                                 val_dev.get(),
                                 column_index_dev.get(),
                                 row_ptr_dev.get(),
                                 column_sums.get_values());
    AssertCudaKernel();

    return column_sums.linfty_norm();
  }



  template <typename Number>
  Number
  SparseMatrix<Number>::linfty_norm() const
  {
    LinearAlgebra::CUDAWrappers::Vector<real_type> row_sums(n_rows);
    const int n_blocks = 1 + (nnz - 1) / block_size;
    internal::linfty_norm<Number>
      <<<n_blocks, block_size>>>(n_rows,
                                 val_dev.get(),
                                 column_index_dev.get(),
                                 row_ptr_dev.get(),
                                 row_sums.get_values());
    AssertCudaKernel();

    return row_sums.linfty_norm();
  }



  template <typename Number>
  Number
  SparseMatrix<Number>::frobenius_norm() const
  {
    LinearAlgebra::CUDAWrappers::Vector<real_type> matrix_values(nnz);
    hipError_t cuda_error = hipMemcpy(matrix_values.get_values(),
                                        val_dev.get(),
                                        nnz * sizeof(Number),
                                        hipMemcpyDeviceToDevice);

    return matrix_values.l2_norm();
  }



  template <typename Number>
  std::tuple<Number *, int *, int *, hipsparseMatDescr_t, hipsparseSpMatDescr_t>
  SparseMatrix<Number>::get_cusparse_matrix() const
  {
    return std::make_tuple(val_dev.get(),
                           column_index_dev.get(),
                           row_ptr_dev.get(),
                           descr,
                           sp_descr);
  }



  template class SparseMatrix<float>;
  template class SparseMatrix<double>;
} // namespace CUDAWrappers
DEAL_II_NAMESPACE_CLOSE

#endif

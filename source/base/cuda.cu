// ---------------------------------------------------------------------
//
// Copyright (C) 2018 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the deal.II distribution.
//
// ---------------------------------------------------------------------

#include <deal.II/base/hip/hip_runtime.h>

#include <deal.II/base/exceptions.h>
#include <deal.II/lac/cuda_vector.h>
#include <deal.II/lac/vector_memory.h>

DEAL_II_NAMESPACE_OPEN

namespace Utilities
{
  namespace CUDA
  {
    Handle::Handle()
    {
      hipsolverStatus_t cusolver_error_code
        = hipsolverDnCreate(&cusolver_dn_handle);
      AssertCusolver(cusolver_error_code);

      cusolver_error_code = hipsolverSpCreate(&cusolver_sp_handle);
      AssertCusolver(cusolver_error_code);

      hipsparseStatus_t cusparse_error_code = hipsparseCreate(&cusparse_handle);
      AssertCusparse(cusparse_error_code);
    }

    Handle::~Handle()
    {
      dealii::GrowingVectorMemory<
        LinearAlgebra::CUDAWrappers::Vector<float>>::release_unused_memory();
      dealii::GrowingVectorMemory<
        LinearAlgebra::CUDAWrappers::Vector<double>>::release_unused_memory();

      hipsolverStatus_t cusolver_error_code
        = hipsolverDnDestroy(cusolver_dn_handle);
      AssertCusolver(cusolver_error_code);

      cusolver_error_code = hipsolverSpDestroy(cusolver_sp_handle);
      AssertCusolver(cusolver_error_code);

      hipsparseStatus_t cusparse_error_code = hipsparseDestroy(cusparse_handle);
      AssertCusparse(cusparse_error_code);
    }
  } // namespace CUDA
} // namespace Utilities

DEAL_II_NAMESPACE_CLOSE
